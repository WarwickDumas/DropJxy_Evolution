
#include "cuda_struct.h"

__host__ bool Call(hipError_t cudaStatus, char str[])
{
	if (cudaStatus == hipSuccess) return false;
	printf("Error: %s\nReturned %d : %s\n",
		str, cudaStatus, hipGetErrorString(cudaStatus));
	printf("Anykey.\n");	getch();
	return true;
}


cuSyst::cuSyst(){
	bInvoked = false;
	bInvokedHost = false;
}

int cuSyst::Invoke()
{
	 Nverts = NUMVERTICES;
	 Ntris = NUMTRIANGLES; // FFxtubes.h
	 Nminor = Nverts + Ntris;

	if (bInvoked == false) {

		if (
			   (!CallMAC(hipMalloc((void**)&p_info, Nminor * sizeof(structural))))

			&& (!CallMAC(hipMalloc((void**)&p_izTri_vert, Nverts*MAXNEIGH_d * sizeof(long))))
			&& (!CallMAC(hipMalloc((void**)&p_izNeigh_vert, Nverts*MAXNEIGH_d * sizeof(long))))
			&& (!CallMAC(hipMalloc((void**)&p_szPBC_vert, Nverts*MAXNEIGH_d * sizeof(char))))
			
			&& (!CallMAC(hipMalloc((void**)&p_Indexneigh_triminor, Ntris*6 * sizeof(long))))
			&& (!CallMAC(hipMalloc((void**)&p_szPBC_triminor, Ntris * 6 * sizeof(char))))
			&& (!CallMAC(hipMalloc((void**)&p_tri_corner_index, Ntris * sizeof(LONG3))))
			&& (!CallMAC(hipMalloc((void**)&p_tri_periodic_corner_flags, Ntris * sizeof(CHAR4))))
			&& (!CallMAC(hipMalloc((void**)&p_who_am_I_to_corner, Ntris * sizeof(LONG3))))

			&& (!CallMAC(hipMalloc((void**)&p_n_major, Nverts * sizeof(nvals))))
			&& (!CallMAC(hipMalloc((void**)&p_n_minor, Nminor * sizeof(nvals))))
			&& (!CallMAC(hipMalloc((void**)&p_T_minor, Nminor * sizeof(T3))))

			&& (!CallMAC(hipMalloc((void**)&p_AAdot, Nminor * sizeof(AAdot))))
			
			&& (!CallMAC(hipMalloc((void**)&p_v_n, Nminor * sizeof(f64_vec3))))
			&& (!CallMAC(hipMalloc((void**)&p_vie, Nminor * sizeof(v4))))
			&& (!CallMAC(hipMalloc((void**)&p_B, Nminor * sizeof(f64_vec3))))

			&& (!CallMAC(hipMalloc((void**)&p_Lap_Az, Nminor * sizeof(f64))))
			&& (!CallMAC(hipMalloc((void**)&p_v_overall_minor, Nminor * sizeof(f64_vec2))))
			
			&& (!CallMAC(hipMalloc((void**)&p_MomAdditionRate_ion, Nminor * sizeof(f64_vec3))))
			&& (!CallMAC(hipMalloc((void**)&p_MomAdditionRate_elec, Nminor * sizeof(f64_vec3))))
			&& (!CallMAC(hipMalloc((void**)&p_MomAdditionRate_neut, Nminor * sizeof(f64_vec3))))
			
			&& (!CallMAC(hipMalloc((void**)&p_AreaMinor, Nminor * sizeof(f64))))
			&& (!CallMAC(hipMalloc((void**)&p_AreaMajor, Nverts * sizeof(f64))))
			)
		{
			bInvoked = true;
			//Zero();
			printf("Dimensioned for MAXNEIGH_d = %d\n", MAXNEIGH_d);
			return 0;
		}
		else {
			printf("There was an error in dimensioning Systdata object.\n");
			getch();	getch();
			return 1;
		};
	}
	else {
		if (Nverts != NUMVERTICES) { printf("cuSyst Error - Nverts %d != N %d\n", Nverts, NUMVERTICES); getch(); }
		return 2;
	};
}
int cuSyst::InvokeHost()
{
	Nverts = NUMVERTICES;
	Ntris = NUMTRIANGLES;
	Nminor = Nverts + Ntris;
	p_info = ( structural * )malloc(Nminor* sizeof(structural));
		
	p_izTri_vert = ( long *)malloc(Nverts*MAXNEIGH_d * sizeof(long));
	p_izNeigh_vert = (long * )malloc(Nverts*MAXNEIGH_d * sizeof(long));
	p_szPBC_vert = (char * )malloc(Nverts*MAXNEIGH_d * sizeof(char));

	p_Indexneigh_triminor = (long * )malloc(Ntris * 6 * sizeof(long));
	p_szPBC_triminor = (char * )malloc(Ntris * 6 * sizeof(char));
	p_tri_corner_index = ( LONG3 *)malloc(Ntris * sizeof(LONG3));
	p_tri_periodic_corner_flags = (CHAR4 *)malloc(Ntris * sizeof(CHAR4));
	p_who_am_I_to_corner = (LONG3 * )malloc(Ntris * sizeof(LONG3));

	p_n_major = (nvals * )malloc(Nverts * sizeof(nvals));
	p_n_minor = (nvals * )malloc(Nminor * sizeof(nvals));
	p_T_minor = (T3 * )malloc(Nminor * sizeof(T3));

	p_AAdot = ( AAdot *)malloc(Nminor * sizeof(AAdot));

	p_v_n = ( f64_vec3 *)malloc(Nminor * sizeof(f64_vec3));
	p_vie = (v4 * )malloc(Nminor * sizeof(v4));
	p_B = ( f64_vec3 *)malloc(Nminor * sizeof(f64_vec3));

	p_Lap_Az = (f64 * )malloc(Nminor * sizeof(f64));
	p_v_overall_minor = (f64_vec2 *)malloc(Nminor * sizeof(f64_vec2));

	p_MomAdditionRate_ion = (f64_vec3 * )malloc(Nminor * sizeof(f64_vec3));
	p_MomAdditionRate_elec = (f64_vec3 *)malloc(Nminor * sizeof(f64_vec3));
	p_MomAdditionRate_neut = (f64_vec3 *)malloc(Nminor * sizeof(f64_vec3));

	p_AreaMinor = (f64 * )malloc(Nminor * sizeof(f64));
	p_AreaMajor = (f64 * )malloc(Nverts * sizeof(f64));

	if (p_AreaMajor == 0) {
		printf("failed to invokeHost the cusyst.\n");
		getch();
		return 1;
	}
	else {
		bInvokedHost = true;
		return 0;
	};
}
cuSyst::~cuSyst(){
	if (bInvoked)
	{

		hipFree(p_info);
		hipFree(p_izTri_vert);
		hipFree(p_izNeigh_vert);
		hipFree(p_szPBC_vert);
		hipFree(p_Indexneigh_triminor);
		hipFree(p_szPBC_triminor);
		hipFree(p_tri_corner_index);
		hipFree(p_tri_periodic_corner_flags)
		hipFree(p_who_am_I_to_corner);
		hipFree(p_n_major);
		hipFree(p_n_minor);
		hipFree(p_T_minor);
		hipFree(p_AAdot);
		hipFree(p_v_n);
		hipFree(p_vie);
		hipFree(p_B);
		hipFree(p_Lap_Az);
		hipFree(p_v_overall_minor);
		hipFree(p_MomAdditionRate_ion);
		hipFree(p_MomAdditionRate_elec);
		hipFree(p_MomAdditionRate_neut);
		hipFree(p_AreaMinor);
		hipFree(p_AreaMajor);

	}
	if (bInvokedHost) {

free(p_info);
free(p_izTri_vert);
free(p_izNeigh_vert);
free(p_szPBC_vert);
free(p_Indexneigh_triminor);
free(p_szPBC_triminor);
free(p_tri_corner_index);
free(p_tri_periodic_corner_flags);
free(p_who_am_I_to_corner);
free(p_n_major);
free(p_n_minor);
free(p_T_minor);
free(p_AAdot);
free(p_v_n);
free(p_vie);
free(p_B);
free(p_Lap_Az);
free(p_v_overall_minor);
free(p_MomAdditionRate_ion);
free(p_MomAdditionRate_elec);
free(p_MomAdditionRate_neut);
free(p_AreaMinor);
free(p_AreaMajor);

	};
}

void cuSyst::SendToHost(cuSyst & Xhost)
{
	// We are going to need a host-allocated cuSyst in order to
	// do the populating basically.
	if ((!CallMAC(hipMemcpy(Xhost.p_info, p_info, Nminor * sizeof(structural), hipMemcpyDeviceToHost)))

		&& (!CallMAC(hipMemcpy(Xhost.p_izTri_vert, p_izTri_vert, Nverts*MAXNEIGH_d * sizeof(long), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_izNeigh_vert, p_izNeigh_vert, Nverts*MAXNEIGH_d * sizeof(long), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_szPBC_vert, p_szPBC_vert, Nverts*MAXNEIGH_d * sizeof(char), hipMemcpyDeviceToHost)))

		&& (!CallMAC(hipMemcpy(Xhost.p_Indexneigh_triminor, p_Indexneigh_triminor, Ntris * 6 * sizeof(long), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_szPBC_triminor, p_szPBC_triminor, Ntris * 6 * sizeof(char), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_tri_corner_index, p_tri_corner_index, Ntris * sizeof(LONG3), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_who_am_I_to_corner, p_who_am_I_to_corner, Ntris * sizeof(LONG3), hipMemcpyDeviceToHost)))

		&& (!CallMAC(hipMemcpy(Xhost.p_n_major, p_n_major, Nverts * sizeof(nvals), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_n_minor, p_n_minor, Nminor * sizeof(nvals), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_T_minor, p_T_minor, Nminor * sizeof(T3), hipMemcpyDeviceToHost)))

		&& (!CallMAC(hipMemcpy(Xhost.p_AAdot, p_AAdot, Nminor * sizeof(AAdot), hipMemcpyDeviceToHost)))

		&& (!CallMAC(hipMemcpy(Xhost.p_v_n, p_v_n, Nminor * sizeof(f64_vec3), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_vie, p_vie, Nminor * sizeof(v4), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_B, p_B, Nminor * sizeof(f64_vec3), hipMemcpyDeviceToHost)))

		&& (!CallMAC(hipMemcpy(Xhost.p_Lap_Az, p_Lap_Az, Nminor * sizeof(f64), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_v_overall_minor, p_v_overall_minor, Nminor * sizeof(f64_vec2), hipMemcpyDeviceToHost)))

		&& (!CallMAC(hipMemcpy(Xhost.p_MomAdditionRate_ion, p_MomAdditionRate_ion, Nminor * sizeof(f64_vec3), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_MomAdditionRate_elec, p_MomAdditionRate_elec, Nminor * sizeof(f64_vec3), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_MomAdditionRate_neut, p_MomAdditionRate_neut, Nminor * sizeof(f64_vec3), hipMemcpyDeviceToHost)))

		&& (!CallMAC(hipMemcpy(Xhost.p_AreaMinor, p_AreaMinor, Nminor * sizeof(f64), hipMemcpyDeviceToHost)))
		&& (!CallMAC(hipMemcpy(Xhost.p_AreaMajor, p_AreaMajor, Nverts * sizeof(f64), hipMemcpyDeviceToHost)))
		)
	{
		// success - do nothing
	}
	else {
		printf("hipMemcpy error");
		getch();
	}
	Call(hipDeviceSynchronize(), "hipDeviceSynchronize cuSyst::SendToHost");

}
void cuSyst::SendToDevice(cuSyst & Xdevice)
{
	// We are going to need a host-allocated cuSyst in order to
	// do the populating basically.
	if (
		   (!CallMAC(hipMemcpy(Xdevice.p_info, p_info, Nminor * sizeof(structural), hipMemcpyHostToDevice)))

		&& (!CallMAC(hipMemcpy(Xdevice.p_izTri_vert, p_izTri_vert, Nverts*MAXNEIGH_d * sizeof(long), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_izNeigh_vert, p_izNeigh_vert, Nverts*MAXNEIGH_d * sizeof(long), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_szPBC_vert, p_szPBC_vert, Nverts*MAXNEIGH_d * sizeof(char), hipMemcpyHostToDevice)))

		&& (!CallMAC(hipMemcpy(Xdevice.p_Indexneigh_triminor, p_Indexneigh_triminor, Ntris * 6 * sizeof(long), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_szPBC_triminor, p_szPBC_triminor, Ntris * 6 * sizeof(char), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_tri_corner_index, p_tri_corner_index, Ntris * sizeof(LONG3), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_who_am_I_to_corner, p_who_am_I_to_corner, Ntris * sizeof(LONG3), hipMemcpyHostToDevice)))

		&& (!CallMAC(hipMemcpy(Xdevice.p_n_major, p_n_major, Nverts * sizeof(nvals), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_n_minor, p_n_minor, Nminor * sizeof(nvals), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_T_minor, p_T_minor, Nminor * sizeof(T3), hipMemcpyHostToDevice)))

		&& (!CallMAC(hipMemcpy(Xdevice.p_AAdot, p_AAdot, Nminor * sizeof(AAdot), hipMemcpyHostToDevice)))

		&& (!CallMAC(hipMemcpy(Xdevice.p_v_n, p_v_n, Nminor * sizeof(f64_vec3), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_vie, p_vie, Nminor * sizeof(v4), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_B, p_B, Nminor * sizeof(f64_vec3), hipMemcpyHostToDevice)))

		&& (!CallMAC(hipMemcpy(Xdevice.p_Lap_Az, p_Lap_Az, Nminor * sizeof(f64), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_v_overall_minor, p_v_overall_minor, Nminor * sizeof(f64_vec2), hipMemcpyHostToDevice)))

		&& (!CallMAC(hipMemcpy(Xdevice.p_MomAdditionRate_ion, p_MomAdditionRate_ion, Nminor * sizeof(f64_vec3), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_MomAdditionRate_elec, p_MomAdditionRate_elec, Nminor * sizeof(f64_vec3), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_MomAdditionRate_neut, p_MomAdditionRate_neut, Nminor * sizeof(f64_vec3), hipMemcpyHostToDevice)))

		&& (!CallMAC(hipMemcpy(Xdevice.p_AreaMinor, p_AreaMinor, Nminor * sizeof(f64), hipMemcpyHostToDevice)))
		&& (!CallMAC(hipMemcpy(Xdevice.p_AreaMajor, p_AreaMajor, Nverts * sizeof(f64), hipMemcpyHostToDevice)))
		)
	{

	}
	else {
		printf("SendToDevice error"); getch();
	}
	Call(hipDeviceSynchronize(), "hipDeviceSynchronize cuSyst::SendToHost");
}

void cuSyst::PopulateFromTriMesh(TriMesh * pX)
{
	// AsSUMES THIS cuSyst has been allocated on the host.
	if ((Nverts != pX->numVertices) ||
		(Ntris != pX->numTriangles))
	{
		printf("ERROR (nVerts %d != pX->numVertices %d) || (nTris != pX->numTriangles)\n",
			Nverts, pX->numVertices);
		getch();
		return;
	}

	plasma_data data;
	long iMinor;
	for (iMinor = 0; iMinor < NMINOR; iMinor++)
	{
		memcpy(&data, &(pX->pData[iMinor]), sizeof(plasma_data));
		p_n_minor[iMinor].n = data.n;
		p_n_minor[iMinor].n_n = data.n_n;
		if (iMinor > BEGINNING_OF_CENTRAL) {
			p_n_major[iMinor - BEGINNING_OF_CENTRAL].n = data.n;
			p_n_major[iMinor - BEGINNING_OF_CENTRAL].n_n = data.n_n;
		}
		p_T_minor[iMinor].Tn = data.Tn;
		p_T_minor[iMinor].Ti = data.Ti;
		p_T_minor[iMinor].Te = data.Te;
		p_AAdot[iMinor].Az = data.Az;
		p_AAdot[iMinor].Azdot = data.Azdot;
		p_v_n[iMinor] = data.v_n;
		p_vie[iMinor].vxy = data.vxy;
		p_vie[iMinor].vez = data.vez;
		p_vie[iMinor].viz = data.viz;
		p_B[iMinor] = pX->p_B[iMinor];
		p_AreaMinor[iMinor] = pX->p_AreaMinor[iMinor];
	}
	
	// AreaMajor??? pVertex->AreaCell?
	Vertex * pVertex;
	pVertex = pX->X;
	long izTri[MAXNEIGH],izNeigh[MAXNEIGH];
	long tri_len;
	long iVertex;
	structural info;
	for (iVertex = 0; iVertex < Nverts; iVertex++)
	{
		tri_len = pVertex->GetTriIndexArray(izTri);
		info.neigh_len = tri_len;
		memset(izTri+tri_len, 0, sizeof(long)*(MAXNEIGH-tri_len));
		memcpy(p_izTri_vert + iVertex*MAXNEIGH, izTri, sizeof(long)*MAXNEIGH);

		tri_len = pVertex->GetNeighIndexArray(izNeigh);
		memset(izNeigh + tri_len, 0, sizeof(long)*(MAXNEIGH - tri_len));
		memcpy(p_izNeigh_vert + iVertex*MAXNEIGH,izNeigh, sizeof(long)*MAXNEIGH);
		
		// PBC list????
		info.pos = pVertex->pos;
		p_info_minor[iVertex + BEGINNING_OF_CENTRAL] = info;
		++pVertex;
	};
	
	long iTri;
	// Triangle structural?
	Triangle * pTri = pX->T;
	for (iTri = 0; iTri < Ntris; iTri++)
	{
		LONG3 tri_corner_index;
		CHAR3 tri_periodic_corner_flags;
		CHAR3 who_am_I_to_corner;

		tri_corner_index.i1 = pTri->cornerptr[0] - pX->T;
		tri_corner_index.i2 = pTri->cornerptr[1] - pX->T;
		tri_corner_index.i3 = pTri->cornerptr[2] - pX->T;
		p_tri_corner_index[iTri] = tri_corner_index;
		
		tri_len = pTri->cornerptr[0]->GetTriIndexArray(izTri);
		for (i = 0; i < tri_len; i++)
		{
			if (izTri[i] == iTri) who_am_I_to_corner.c1 = i;
		}
		tri_len = pTri->cornerptr[1]->GetTriIndexArray(izTri);
		for (i = 0; i < tri_len; i++)
		{
			if (izTri[i] == iTri) who_am_I_to_corner.c2 = i;
		}
		tri_len = pTri->cornerptr[2]->GetTriIndexArray(izTri);
		for (i = 0; i < tri_len; i++)
		{
			if (izTri[i] == iTri) who_am_I_to_corner.c3 = i;
		}

		memset(&tri_periodic_corner_flags, 0, sizeof(CHAR3));
		if (pTri->periodic != 0) {
			if (pTri->cornerptr[0]->pos.x > 0.0) tri_corner_index.c1 = ROTATE_ME_ANTICLOCKWISE;
			if (pTri->cornerptr[1]->pos.x > 0.0) tri_corner_index.c2 = ROTATE_ME_ANTICLOCKWISE;
			if (pTri->cornerptr[2]->pos.x > 0.0) tri_corner_index.c3 = ROTATE_ME_ANTICLOCKWISE;
		}
		p_tri_periodic_corner_flags[iTri] = tri_periodic_corner_flags;
		p_who_am_I_to_corner[iTri] = who_am_I_to_corner;
		++pTri;
	};


}

void cuSyst::PopulateTriMesh(TriMesh * pX)
{
	// AsSUMES THIS cuSyst has been allocated on the host.

	

}
                             