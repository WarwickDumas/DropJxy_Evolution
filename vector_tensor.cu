#include "hip/hip_runtime.h"
#ifndef VECTOR_TENSOR_H
#define VECTOR_TENSOR_H

#include "FFxtubes.h"
#include <conio.h>
#include <stdio.h>

// will want to do #include type.h
// for #define f64, qd_or_d

#define f64 double
#define __HIPCC__

int const MAX_TRIS_PER_VERTEX = 12;  
#ifdef __HIPCC__

#define QUALIFIERS __host__ __device__ __forceinline__ 
#define QUALS __host__ __device__ inline
// __host__ == cpu
// __global__ == kernel
// __device__ == call from kernel

#else

#define QUALIFIERS inline
#define QUALS   inline

#endif

struct Vector2
{
    double x, y;  
	
	QUALIFIERS Vector2 (){}

	QUALIFIERS Vector2 (double newx,double newy)
	{
		x = newx; y = newy;
	}

	f64 QUALIFIERS dot(const Vector2 &v) const
	{
		return x*v.x+y*v.y;
	}


	// NVCC will not put up with forward declaration of friend.
	//friend Vector2 operator* (const f64 h,const Vector2 &v);
	//friend Vector2 operator* (const Vector2 &v,const f64 h);
	
	// Have to try naked definition in cpp file of prefix multiply instead.
	// But that's no good for MSVS : the definition/declaration will get 
	// parsed 0 or several times. Better declare just outside class instead.
 
	Vector2 QUALIFIERS operator -() {
		return Vector2(-x,-y); 
	}
	
	Vector2 QUALIFIERS operator -(const Vector2 &v) const
	{
		Vector2 result;
		result.x = x - v.x;
		result.y = y - v.y;
		return result;
	}
	Vector2 QUALIFIERS operator +(const Vector2 &v) const
	{
		Vector2 result;
		result.x = x + v.x;
		result.y = y + v.y;
		return result;
	}

	Vector2 QUALIFIERS operator* (const f64 hh) const
	{
		Vector2 result;
		result.x = hh*x; result.y = hh*y;
		return result;
	}	

	Vector2 QUALIFIERS operator /(const f64 h) const
	{
		Vector2 result;
		result.x = x/h;
		result.y = y/h;
		return result;
	}
	void QUALIFIERS operator += (const Vector2 &v) 
	{
		x += v.x;
		y += v.y;
	}
	void QUALIFIERS operator -= (const Vector2 &v)
	{
		x -= v.x;
		y -= v.y;
		// Don't think we should try to use return *this in NVCC. 
	}
	void QUALIFIERS operator *= (const f64 alpha)
	{
		x *= alpha;
		y *= alpha;
	}
	void QUALIFIERS operator /= (const f64 alpha)
	{
		x /= alpha;
		y /= alpha;
	}
	f64 QUALIFIERS modulus() const
	{
		f64 modsq = x*x + y*y;
		if (modsq <= 0.0) return 0.0;
		return sqrt(x*x+y*y);
	}

	void QUALIFIERS Normalise()
	{
		f64 r = modulus();
		x /= r; y /= r;
	}
	
	void QUALIFIERS project_to_ins(Vector2 & result) const
	{
		// If it never gets called, hopefully NVCC won't care what is in that.
		f64 factor = DEVICE_RADIUS_INSULATOR_OUTER/sqrt(x*x+y*y);
		result.x = x*factor; result.y = y*factor;
	}

	void QUALIFIERS project_to_radius
				(Vector2 & result, f64 radius)
	{
		f64 factor = radius/sqrt(x*x+y*y);
		result.x = x*factor; result.y = y*factor;
	}
};

Vector2 QUALS operator* (const f64 hh,const Vector2 &v)
{
	return Vector2(hh*v.x,hh*v.y);
}

// NOTE: For MSVC, declaring members inline means that 
// they have to be defined in the header - and
// that seems to be okay.
// So we probably should do that with everything here.

struct Vector3
{
	f64 x,y,z;

	QUALIFIERS Vector3() {};
	QUALIFIERS Vector3(f64 a, f64 b, f64 c)
	{
		x = a; y = b; z = c;
	}

	Vector3 QUALIFIERS operator- () const
	{
		return Vector3 (-x,-y,-z);
	}

	Vector3 QUALIFIERS cross(const Vector3 & v) const
	{
		return Vector3(y*v.z-z*v.y,
			           z*v.x-x*v.z,
					   x*v.y-y*v.x);
	}
	
	Vector3 QUALIFIERS operator +(const Vector3 &v) const
	{
		return Vector3(x+v.x,y+v.y,z+v.z);
	}

	Vector3 QUALIFIERS operator -(const Vector3 &v) const
	{
		return Vector3(x-v.x,y-v.y,z-v.z);
	}

	
	Vector3 QUALIFIERS operator* (const f64 hh) const
	{
		return Vector3(hh*x,hh*y,hh*z);
	}
	Vector3 QUALIFIERS operator/ (const f64 hh) const 
	{
		return Vector3(x/hh,y/hh,z/hh);
	}
	
	void QUALIFIERS operator +=(const Vector3 &v) 
	{
		x += v.x; y += v.y; z += v.z;
	}
	void QUALIFIERS operator -=(const Vector3 &v) 
	{
		x -= v.x; y -= v.y; z -= v.z;
	}
	void QUALIFIERS operator *=(const f64 xx) 
	{
		x *= xx; y *= xx; z *= xx;
	}
	void QUALIFIERS operator /=(const f64 xx) 
	{
		x /= xx; y /= xx; z /= xx;
	}

	bool QUALIFIERS operator != (const Vector3 &v) const 
	{
		return ((v.x != x) || (v.y != y) || (v.z != z));
	}

	f64 QUALIFIERS dotxy(const Vector3 &v) const
	{
		return x*v.x+y*v.y;
	}
	f64 QUALIFIERS dotxy(const Vector2 &v) const
	{
		return x*v.x+y*v.y;
	}
	f64 QUALIFIERS dot(const Vector2 &v) const
	{
		return x*v.x+y*v.y;
	}
	f64 QUALIFIERS dot(const Vector3 &v) const
	{
		return x*v.x+y*v.y+z*v.z;
	}

	Vector2 QUALIFIERS xypart() const
	{
		Vector2 u;
		u.x = x; 
		u.y = y;
		return u;
	}

	f64 QUALIFIERS modulusxy()
	{
		return sqrt(x*x+y*y);
	}
	
	f64 QUALIFIERS modulus()
	{
		f64 modsq = x*x + y*y + z*z;
		if (modsq <= 0.0) return 0.0;
		return sqrt(x*x+y*y+z*z);
	}

	void Reflect_radially(Vector2 & centre);
	void ZeroRadially(Vector2 & centre);
};

Vector3 QUALS operator* (const f64 hh,const Vector3 &v)
{
	return Vector3(hh*v.x,hh*v.y,hh*v.z);
}


f64 QUALS dotxy(const Vector2 & v1, const Vector3 & v2)
{
	return v1.x*v2.x+v1.y*v2.y;
}
f64 QUALS dotxy(const Vector3 & v1, const Vector2 & v2)
{
	return v1.x*v2.x+v1.y*v2.y;
}

struct Tensor2
{
	f64 xx, xy, yx, yy;
	QUALIFIERS Tensor2() {}
	QUALIFIERS Tensor2(f64 x_x, f64 x_y, f64 y_x, f64 y_y)
	{
		xx = x_x; xy = x_y; yx = y_x; yy = y_y;
	}
	QUALIFIERS ~Tensor2() {}
	QUALIFIERS Tensor2 operator +(const Tensor2 &X) const
	{
		return Tensor2(
			xx + X.xx,
			xy + X.xy,
			yx + X.yx,
			yy + X.yy);
	}

	QUALIFIERS Tensor2 operator *(const Tensor2 &X) const
	{
		Tensor2 result;
		// did a test: X is the one on the right.
		result.xx = xx*X.xx + xy*X.yx;
		result.xy = xx*X.xy + xy*X.yy;
		result.yx = yx*X.xx + yy*X.yx;
		result.yy = yx*X.xy + yy*X.yy;
		return result;
	}

	QUALIFIERS Tensor2 operator *(const f64 hh) const
	{
		return Tensor2 (hh*xx,hh*xy,hh*yx, hh*yy);
	}

	QUALIFIERS Tensor2 operator -(const Tensor2 &X) const
	{
		return Tensor2(
			xx - X.xx, xy-X.xy, yx-X.yx, yy-X.yy
			);
	}
	
	QUALIFIERS void Inverse(Tensor2 & result) const
	{
		f64 overdet = 1.0/(xx*yy-xy*yx);
		result.xx = yy*overdet;
		result.xy = -xy*overdet;
		result.yx = -yx*overdet;
		result.yy = xx*overdet;
	};

	QUALIFIERS Vector2 operator *(const Vector2 &v) const
	{
		return Vector2(xx*v.x+xy*v.y,yx*v.x+yy*v.y);
	}
	
	QUALIFIERS void operator +=(const Tensor2 &X)
	{
		xx += X.xx; xy += X.xy;
		yx += X.yx; yy += X.yy;
	}

	QUALIFIERS void operator *=(const f64 hh) 
	{
		xx *= hh; xy *= hh;
		yx *= hh; yy *= hh;
	}
};
QUALIFIERS Tensor2 operator *(const f64 hh, const Tensor2 &X) 
{
	return Tensor2(hh*X.xx,hh*X.xy,hh*X.yx,hh*X.yy);
}

extern Tensor2 ID2x2;
extern Tensor2 zero2x2;

struct Tensor3
{
	f64 xx,xy,xz,yx,yy,yz,zx,zy,zz;
	QUALIFIERS Tensor3() {};

	QUALIFIERS Tensor3(f64 x_x, f64 x_y, f64 x_z, 
		          f64 y_x, f64 y_y, f64 y_z,
				  f64 z_x, f64 z_y, f64 z_z) 
	{
		xx = x_x; xy = x_y; xz = x_z; 
		yx = y_x; yy = y_y; yz = y_z; 
		zx = z_x; zy = z_y; zz = z_z;
	}

	QUALIFIERS void MakeCross (const Vector3 om)
	{
		xx = 0.0;
		xy = -om.z;
		xz = om.y;
		yx = om.z;
		yy = 0.0;
		yz = -om.x;
		zx = -om.y;
		zy = om.x;
		zz = 0.0;
	}
	
	QUALIFIERS Tensor3 Inverse()
	{
		Tensor3 result;
		f64 det =	  xx*(yy*zz-yz*zy)
					+ xy*(zx*yz-yx*zz)
					+ xz*(yx*zy-yy*zx);
		
		// Fill in matrix of minor determinants; 
		// transposed with applied cofactors (signs)
		
		result.xx = yy*zz-yz*zy;
		result.yx = zx*yz-yx*zz; 
		result.zx = yx*zy-yy*zx;
		result.xy = zy*xz-xy*zz;
		result.yy = xx*zz-xz*zx;
		result.zy = zx*xy-xx*zy;
		result.xz = xy*yz-xz*yy;
		result.yz = yx*xz-xx*yz;
		result.zz = xx*yy-yx*xy;

		if (det != 0.0) {
			result = result / det;
		} else {
			printf("\n\nMATRIX INVERSE FAILED. Det==0\n\n\n");
			memset(&result, 0, sizeof(Tensor3));
			result.xx = 1.0; result.yy = 1.0; result.zz = 1.0;
		}
		return result; // inline so return object doesn't matter
	};

	QUALIFIERS void Inverse(Tensor3 & result)
	{
		f64 det = (xx*(yy*zz - yz*zy)
			+ xy*(zx*yz - yx*zz)
			+ xz*(yx*zy - yy*zx));

		if (det == 0.0) {
			printf("\n\nMATRIX INVERSE FAILED II. Det == 0\n\n\n");
			return;
		}
		f64 over =	1.0/det;
		
		// Fill in matrix of minor determinants; 
		// transposed with applied cofactors (signs)
		
		result.xx = (yy*zz-yz*zy)*over;
		result.yx = (zx*yz-yx*zz)*over; 
		result.zx = (yx*zy-yy*zx)*over;
		result.xy = (zy*xz-xy*zz)*over;
		result.yy = (xx*zz-xz*zx)*over;
		result.zy = (zx*xy-xx*zy)*over;
		result.xz = (xy*yz-xz*yy)*over;
		result.yz = (yx*xz-xx*yz)*over;
		result.zz = (xx*yy-yx*xy)*over;

		//return result; // inline so return object doesn't matter
	};

QUALIFIERS Tensor2 xy2x2part () const
	{
		Tensor2 res;
		res.xx = xx;
		res.xy = xy;
		res.yx = yx;
		res.yy = yy;
		return res;
	}

QUALIFIERS Tensor3 operator- () const
	{
		Tensor3 res;
		res.xx = -xx; res.xy = -xy; res.xz = -xz;
		res.yx = -yx; res.yy = -yy; res.yz = -yz;
		res.zx = -zx; res.zy = -zy; res.zz = -zz;
		return res;
	}

QUALIFIERS Vector3 operator* (const Vector3 &v) const
	{
		Vector3 res;
		res.x = xx*v.x + xy*v.y + xz*v.z;
		res.y = yx*v.x + yy*v.y + yz*v.z;
		res.z = zx*v.x + zy*v.y + zz*v.z;
		return res;
	}
	
	
QUALIFIERS Tensor3 operator* (const f64 hh) const
	{ 
		return Tensor3(
			hh*xx, hh*xy, hh*xz,
			hh*yx, hh*yy, hh*yz,
			hh*zx, hh*zy, hh*zz);
	};
		
QUALIFIERS Tensor3 operator/ (const f64 r) const
	{
		Tensor3 result;
		// did a test: X is the one on the right.
		result.xx = xx/r;
		result.xy = xy/r;
		result.xz = xz/r;
		result.yx = yx/r;
		result.yy = yy/r;
		result.yz = yz/r;
		result.zx = zx/r;
		result.zy = zy/r;
		result.zz = zz/r;
		return result;
	}

QUALIFIERS Tensor3 operator +(const Tensor3 &v) const
	{
		Tensor3 result;
		result.xx = xx + v.xx;
		result.xy = xy + v.xy;
		result.xz = xz + v.xz;
		result.yx = yx + v.yx;
		result.yy = yy + v.yy;
		result.yz = yz + v.yz;
		result.zx = zx + v.zx;
		result.zy = zy + v.zy;
		result.zz = zz + v.zz;
		return result;
	}

QUALIFIERS Tensor3 operator -(const Tensor3 &v) const
	{
		Tensor3 result;
		result.xx = xx - v.xx;
		result.xy = xy - v.xy;
		result.xz = xz - v.xz;
		result.yx = yx - v.yx;
		result.yy = yy - v.yy;
		result.yz = yz - v.yz;
		result.zx = zx - v.zx;
		result.zy = zy - v.zy;
		result.zz = zz - v.zz;
		return result;
	}
	
QUALIFIERS Tensor3 operator *(const Tensor3 &X) const
	{
		Tensor3 result;
		result.xx = xx*X.xx + xy*X.yx + xz*X.zx;
		result.xy = xx*X.xy + xy*X.yy + xz*X.zy;
		result.xz = xx*X.xz + xy*X.yz + xz*X.zz;
		result.yx = yx*X.xx + yy*X.yx + yz*X.zx;
		result.yy = yx*X.xy + yy*X.yy + yz*X.zy;
		result.yz = yx*X.xz + yy*X.yz + yz*X.zz;
		result.zx = zx*X.xx + zy*X.yx + zz*X.zx;
		result.zy = zx*X.xy + zy*X.yy + zz*X.zy;
		result.zz = zx*X.xz + zy*X.yz + zz*X.zz;
		return result;
	}
	
QUALIFIERS Tensor3 operator +=(const Tensor3 &X) 
	{
		xx += X.xx;
		xy += X.xy;
		xz += X.xz;
		yx += X.yx;
		yy += X.yy;
		yz += X.yz;
		zx += X.zx;
		zy += X.zy;
		zz += X.zz;
		return *this;
	}
	QUALIFIERS Tensor3 operator -=(const Tensor3 &X) 
	{
		xx -= X.xx;
		xy -= X.xy;
		xz -= X.xz;
		yx -= X.yx;
		yy -= X.yy;
		yz -= X.yz;
		zx -= X.zx;
		zy -= X.zy;
		zz -= X.zz;
		return *this;
	}
	
	void Make3DRotationAboutAxis(Vector3 w, f64 t);
	void spitout(void);
};
QUALIFIERS Tensor3 operator* (const f64 hh,const Tensor3 &X)
	{		
		Tensor3 result;
		result.xx = hh*X.xx;
		result.xy = hh*X.xy;
		result.xz = hh*X.xz;
		result.yx = hh*X.yx;
		result.yy = hh*X.yy;
		result.yz = hh*X.yz;
		result.zx = hh*X.zx;
		result.zy = hh*X.zy;
		result.zz = hh*X.zz;
		return result;
	}

// Not clear to me : do we want the following for NVCC to be here?
// It actually makes sense to keep "matrix" here!

struct Matrix3
{
	f64 a[3][3];

	QUALIFIERS void Inverse(Matrix3 & result)
	{
		// find+replace on the above

		f64 det =	  a[0][0]*(a[1][1]*a[2][2]-a[1][2]*a[2][1])
					+ a[0][1]*(a[2][0]*a[1][2]-a[1][0]*a[2][2])
					+ a[0][2]*(a[1][0]*a[2][1]-a[1][1]*a[2][0]);

		// Fill in matrix of minor determinants; 
		// transposed with applied cofactors (signs)
	
		result.a[0][0] = a[1][1]*a[2][2]-a[1][2]*a[2][1];
		result.a[1][0] = a[2][0]*a[1][2]-a[1][0]*a[2][2]; 
		result.a[2][0] = a[1][0]*a[2][1]-a[1][1]*a[2][0];
		result.a[0][1] = a[2][1]*a[0][2]-a[0][1]*a[2][2];
		result.a[1][1] = a[0][0]*a[2][2]-a[0][2]*a[2][0];
		result.a[2][1] = a[2][0]*a[0][1]-a[0][0]*a[2][1];
		result.a[0][2] = a[0][1]*a[1][2]-a[0][2]*a[1][1];
		result.a[1][2] = a[1][0]*a[0][2]-a[0][0]*a[1][2];
		result.a[2][2] = a[0][0]*a[1][1]-a[1][0]*a[0][1];

		//f64 * ptr = (f64 *)(result.a);
		for (int i = 0; i < 3; i++)
		for (int j = 0; j < 3; j++)
		{
			result.a[i][j] /= det; // 99% sure static array elems are contiguous but hey.
		}
	
	};

	QUALIFIERS void multiply(f64 RHS[3], f64 output[3])
	{
		output[0] = a[0][0]*RHS[0] + a[0][1]*RHS[1] + a[0][2]*RHS[2];
		output[1] = a[1][0]*RHS[0] + a[1][1]*RHS[1] + a[1][2]*RHS[2];
		output[2] = a[2][0]*RHS[0] + a[2][1]*RHS[1] + a[2][2]*RHS[2];
	};
	
};

extern Tensor3 ID3x3;
extern Tensor3 zero3x3;

struct f64_tens3mag {
	f64 bx, by, bz, Px, Py, Pz, Hx, Hy, Hz;
};
struct f64_vec3mag {
	f64 b, P, H;
};
Vector3 QUALS Make3(const Vector2 & v, const f64 scalar)
{
	Vector3 result;
	result.x = v.x;
	result.y = v.y;
	result.z = scalar;
	return result;
};

// Never used? :
struct Symmetric3
{
	f64 xx,yy,zz,xy,xz,yz;
	QUALIFIERS Symmetric3() {};
	QUALIFIERS Symmetric3(f64 x_x, f64 x_y, f64 y_y, f64 x_z, f64 y_z, f64 z_z) ;
	
	Vector3 QUALIFIERS operator* (const Vector3 &v) const;
};


// Never used? :
struct Symmetric2
{
	f64 xx,yy,xy;
};


#define f64 double
#define f64_vec2 Vector2
#define f64_vec3 Vector3
#define f64_tens2 Tensor2
#define f64_tens3 Tensor3
//#define u32 unsigned long


//struct vertinfo
//{
//	long flag;
//	long numTris;
//	f64_vec2 pos; 
//	long iTriIndex[MAX_TRIS_PER_VERTEX]; // 10 x 8
//};
////
//struct structural
//{
//	u32 u32corner[3];                    
//	u32 u32neigh[3];                     // 8x3
//	int iDomain_flag, iPeriodic;       
//	f64_vec2 edge_normal[3];             // 8x6
//	f64_vec2 gradT;                        
//	f64 weight[3]; // weights used for averaging at corners. :/ from CPU
//	f64_vec2 pos; // centroid
//	f64 area;                            // 8x5+8x3
//	// size ~ 24 + 8 + 48 + 40 + 24 
//	// 144 bytes or so
//	f64_vec2 coeff[3]; // for each triangle, the coefficient by which nT_cell creates pressure at vertex.
//	
//	// Demoralised from having to do vertex aggregation so let's bung this in here.
//	// In the next version we can get rid of it.
//};

struct species_f64
{
	f64 neut, ion, elec;
};

struct species_vec2
{
	f64_vec2 neut, ion, elec;
};

struct species_vec3
{
	f64_vec3 neut, ion, elec;
};

struct f64_vec4
{
	f64 x[4];
};
#endif


#undef QUALIFIERS