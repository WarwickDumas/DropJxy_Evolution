#include "hip/hip_runtime.h"
#define real double
#define f64 double

#define HISTORY										4
 
#include <stdlib.h>
#include <stdio.h>
#include "lapacke.h"

/* Auxiliary routines prototypes */
extern void print_matrix(char* desc, lapack_int m, lapack_int n, double* a, lapack_int lda);
extern void print_int_vector(char* desc, lapack_int n, lapack_int* a);


#include "headers.h"
#include "hip/hip_runtime.h"
#include ""

#include <time.h>
#include <stdio.h>
#include <windows.h>
#include "resource.h"
#include "flags.h"
#include "FFxtubes.h"
//#include "cppconst.h"
#include "cuda_struct.h"
#include "constant.h"
#include "d3d.h"    
#include <d3dx9.h> 
#include <dxerr.h>
#include <commdlg.h>    // probably used by avi_utils
#include "surfacegraph_tri.h"
#include "avi_utils.cpp"     // for making .avi
#include "kernel.h"

//=======================================================
// Declarations of functions:

void RefreshGraphs(TriMesh & X, const int iGraphsFlag);
LRESULT CALLBACK	WndProc(HWND, UINT, WPARAM, LPARAM);
INT_PTR CALLBACK	About(HWND, UINT, WPARAM, LPARAM);
INT_PTR CALLBACK	SetupBox(HWND, UINT, WPARAM, LPARAM);
extern f64 GetEzShape__(f64 r);
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

extern f64 * temp_array_host;
extern OhmsCoeffs * p_OhmsCoeffs_host;
extern f64 * p_graphdata1_host,* p_graphdata2_host,* p_graphdata3_host, *p_graphdata4_host, *p_graphdata5_host, *p_graphdata6_host;
extern f64 * p_Tgraph_host[9];
extern f64 * p_accelgraph_host[12];

// Global variables:
// =================
//extern f64_vec3 * p_B_host;
extern f64 EzStrength_;
extern cuSyst cuSyst1, cuSyst2, cuSyst3;
extern D3D Direct3D;
extern f64 * p_temphost1, *p_temphost2,
*p_temphost3, *p_temphost4, *p_temphost5, *p_temphost6;

extern __device__ f64 * p_LapCoeffself;
extern __device__ f64 * p_temp1;
extern __device__ long * p_longtemp;
extern __device__ f64 * p_Az, *p_LapAz;


float xzscale;

bool bCullNone = false;
bool bGlobalsave = false;
int GlobalSwitchBox = 0;
int iGlobalScratch;

real GlobalHeightScale;

int GlobalSpeciesToGraph = SPECIES_ION;
int GlobalWhichLabels = 0;
bool GlobalRenderLabels = false;
int GlobalColoursPlanView = 0;
bool GlobalBothSystemsInUse;

bool GlobalCutaway = true;

unsigned int cw; // control word for floating point hardware exception hiding

TriMesh * pX, *pXnew;
TriMesh X1, X2, X3, X4;
cuSyst cuSyst_host, cuSyst_host2;

D3DXVECTOR3 GlobalEye, GlobalLookat, GlobalPlanEye, GlobalPlanEye2, GlobalPlanLookat,
GlobalPlanLookat2, GlobalEye2, GlobalLookat2;

D3DXVECTOR3 newEye;
D3DXVECTOR3 newLookat;

IDirect3DSurface9* p_backbuffer_surface;

long steps_remaining, GlobalStepsCounter, steps_remaining_CPU;
real evaltime, h;

extern real GlobalIzElasticity;
FILE * massfile, *maxfile;

// Global Variables:
HINSTANCE hInst;   // current instance
				   // window vars:
HWND hWnd;
WNDCLASSEX wcex;
TCHAR szTitle[1024];					// The title bar text
TCHAR szWindowClass[1024];			// the main window class name

char Functionalfilename[1024];
int GlobalGraphSetting[7];
surfacegraph Graph[7]; // why was it 5? // 5th one can be whole thing.

float Historic_max[512][HISTORY]; // if max is falling, use historic maximum for graph.
float Historic_min[512][HISTORY];
int Historic_powermax[512];
int Historic_powermin[512]; // just store previous value only.

bool flaglist[NMINOR];

bool boolGlobalHistory, GlobalboolDisplayMeshWireframe;

// avi file -oriented variables
int const NUMAVI = 7;
HAVI hAvi[NUMAVI + 1]; // does it work without OHMSLAW? //  OHMSLAW,
int const GraphFlags[NUMAVI] = { SPECIES_ION, OVERALL, JZAZBXYEZ, ONE_D, IONIZEGRAPH,
				DTGRAPH, ACCELGRAPHS};
char szAvi[NUMAVI][128] = { "Elec","Total","JzAzBxy","Test", "Ionize", "dT", "Accel"};

AVICOMPRESSOPTIONS opts;
int counter;
HBITMAP surfbit, dib;
HDC surfdc, dibdc;
LPVOID lpvBits;
BITMAPINFO bitmapinfo;

f64 graphdata[12][10000]; 
f64 graph_r[10000];
int numgraphs = 4;
int num_graph_data_points = 10000;
f64 maximum[12];

extern TriMesh * pTriMesh;


char * report_time(int action)
{
	/* action = 0: reset ; action = 1: report */
	/* requires timebuffy to be defined as char[255] globally */

	static char timebuffer[255];
	static clock_t start;
	double timenow;
	long ops;

	if (action == 0)
	{
		start = clock();
	}
	else
	{
		timenow = ((double)(clock() - start) / (double)CLOCKS_PER_SEC);
		ops = (long)(clock() - start);
		/* create a null-terminated string */
		sprintf(timebuffer, "%6.4f sec.", timenow);
	};
	return &(timebuffer[0]);
};

f64 GetTriangleArea(f64_vec2 pos0, f64_vec2 pos1, f64_vec2 pos2)
{
	f64 area = 0.5*((pos0.x + pos1.x)*(pos1.y - pos0.y) + (pos1.x + pos2.x)*(pos2.y - pos1.y)
		+ (pos0.x + pos2.x)*(pos0.y - pos2.y));
	return fabs(area);
}

void TriMesh::CalculateTotalGraphingData()
{
	/*long iVertex;
	Vertex * pVertex = X;
	for (iVertex = 0; iVertex < numVertices; iVertex++)
	{
	if ((pVertex->flags == DOMAIN_VERTEX) || (pVertex->flags == OUTERMOST))
	{
	pVertex->n = (pVertex->Neut.mass + pVertex->Ion.mass) / pVertex->AreaCell;
	pVertex->v = (m_n*pVertex->Neut.mom + m_ion * pVertex->Ion.mom + m_e * pVertex->Elec.mom) /
	(m_n*pVertex->Neut.mass + m_ion * pVertex->Ion.mass + m_e * pVertex->Elec.mass);
	pVertex->T = (pVertex->Neut.heat + pVertex->Ion.heat + pVertex->Elec.heat) /
	(pVertex->Neut.mass + pVertex->Ion.mass + pVertex->Elec.mass);
	pVertex->Temp.x = pVertex->Ion.mass / (pVertex->Neut.mass + pVertex->Ion.mass);
	};
	++pVertex;
	}*/
}

void TriMesh::Setup_J()
{
	/*long iVertex;
	Vertex * pVertex = X;
	for (iVertex = 0; iVertex < numVertices; iVertex++)
	{
	if ((pVertex->flags == DOMAIN_VERTEX) || (pVertex->flags == OUTERMOST))
	{
	pVertex->Temp = q * (pVertex->Ion.mom - pVertex->Elec.mom) / pVertex->AreaCell;
	}
	else {
	memset(&(pVertex->Temp), 0, sizeof(Vector3));
	}
	++pVertex;
	}*/
}

void surfacegraph::DrawSurface(const char * szname,
	const int heightflag,
	const real * var_ptr_0,
	const int colourflag,
	const real * var_ptr_c,
	const bool bDisplayInner,
	const int code, // graph code, to pass to called routines - sometimes useful
	const TriMesh * pX // for passing to SetDataWithColour and Render
					   // and for working out offsets
)
{
	// replaced CreateSurfaceGraphs.
	// I think this is about the right balance.

	char buff[256];
	real * temprealptr = (real *)(pX->pData);
	long offset = var_ptr_0 - temprealptr;
	long offset_c = var_ptr_c - temprealptr;

	// Does shader always go with colour type?? yes I think.
	switch (colourflag) {
	case VELOCITY_COLOUR:
		this->mhTech = mFX->GetTechniqueByName("VelociTech");
		break;
	case SEGUE_COLOUR:
		this->mhTech = mFX->GetTechniqueByName("SegueTech");
		break;
	case CURRENT_COLOUR:
		this->mhTech = mFX->GetTechniqueByName("XYZTech");
		break;
	case AZSEGUE_COLOUR:
		mhTech = mFX->GetTechniqueByName("AzSegueTech");
		break;
	case IONISE_COLOUR:
		mhTech = mFX->GetTechniqueByName("IoniseTech");
		break;
	case PPN_COLOUR:
		mhTech = mFX->GetTechniqueByName("ProportionTech"); // 1 = blue
		break;
	};

	// Usual settings:
	//if (GlobalGraphSetting[i] != GRAPH_NONE) {

	this->boolDisplayShadow = true;
	this->boolDisplayMainMesh = true;
	this->boolDisplayMeshWireframe = GlobalboolDisplayMeshWireframe;
	this->boolClearZBufferBeforeWireframe = false;
	// Or try setting true and CULL_CCW to see if this stops it showing "the back of the wireframe"
	this->SetEyeAndLookat(GlobalEye, GlobalLookat);
	this->boolDisplayScales = true;
	this->boolDisplayInnerMesh = bDisplayInner;

	// work out whether to display key button:
	if (((colourflag == FLAG_VELOCITY_COLOUR) || (colourflag == FLAG_CURRENT_COLOUR))
		&& (bDisplayInner == 0))
	{
		this->boolDisplayKeyButton = true;
	}
	else {
		this->boolDisplayKeyButton = false;
	};
	//int const FLAG_COLOUR_MESH = 0;
	//int const FLAG_SEGUE_COLOUR = 1;
	//int const FLAG_VELOCITY_COLOUR = 2;
	//int const FLAG_CURRENT_COLOUR = 3;
	//int const FLAG_AZSEGUE_COLOUR = 4;
	//int const FLAG_IONISE_COLOUR = 5;

	this->SetDataWithColour(*pX,
		colourflag, heightflag, // apparently it's that way round
		offset, offset_c,
		code);

	printf("DrawSurface code %d : calling Render:\n", code);
	if (this->bDisplayTimestamp) {
		sprintf(buff, "%6.2f ns", evaltime*1.0e9);
		this->Render(szname, false, pX, buff);
	}
	else {
		this->Render(szname, false, pX);
	};

}


void Create1DGraphingData(TriMesh * pX, bool bTdata = false, bool bAcceldata = false)
{
	// Takes p_temphost3,4,5,6 and turns them into graphdata[iGraph=0,1,2,3][]

	Vertex * pVertex, * pVert2;
	f64_vec2 pos, pos0, pos1, pos2;
	f64 dist0, dist1, dist2, wt0, wt1, wt2, wttotal, y0, y1, y2;
	int iGraph, asdf, iWhich, iCorner, tri_len, i;
	bool has_more, has_less, has_grad;
	Triangle * pTri;
	long izTri[MAXNEIGH];

	long VertexIndexArray[10000];

	num_graph_data_points = pX->GetVertsRightOfCutawayLine_Sorted(VertexIndexArray, graph_r, true);
	
	printf("Xebeques furious\n");

	memset(maximum, 0, sizeof(f64) * 12);

	// Method used in Render routine looks quite reasonable: find tri that crosses cutaway,
	// use some kind of interp on tri. But we need to use values from p_temphost array not a graph position.

	for (asdf = 0; asdf < num_graph_data_points; asdf++)
	{
		if (asdf % 10 == 0) printf("<");
		
		pVertex = pX->X + VertexIndexArray[asdf];

		// We want the tri directly to the left of it, through which (-1,0) passes.
		// 1.Get these vertex indices
		// which tri contains a point which is further and a point less far?

		real rr = pVertex->pos.x*pVertex->pos.x + pVertex->pos.y*pVertex->pos.y;
		iWhich = -1;
		tri_len = pVertex->GetTriIndexArray(izTri);
		for (i = 0; i < tri_len; i++)
		{
			pTri = pX->T + izTri[i];
			has_more = false; has_less = false; has_grad = false;
			for (iCorner = 0; iCorner < 3; iCorner++)
			{
				pVert2 = pTri->cornerptr[iCorner];
				if (pVert2 != pVertex)
				{
					if (pVert2->pos.x*pVert2->pos.x + pVert2->pos.y*pVert2->pos.y > rr)
					{
						has_more = true;
					}
					else {
						has_less = true;
					};
				};
				if (pVert2->pos.x / pVert2->pos.y < pVertex->pos.x / pVertex->pos.y)
					has_grad = true;
			};

			if (has_more && has_less && has_grad)
			{
				iWhich = i;
			}
		};

		if (iWhich == -1) {// give up, do nothing} 
			printf("gave up. %d \n", VertexIndexArray[asdf]);
			graphdata[0][asdf] = 0.0;
			graphdata[1][asdf] = 0.0;
			graphdata[2][asdf] = 0.0;
			graphdata[3][asdf] = 0.0;
		} else {
			pTri = pX->T + izTri[iWhich];
			while ((pTri->u8domain_flag != DOMAIN_TRIANGLE) && (iWhich >= 0)) {
				pTri = pX->T + izTri[iWhich];
				iWhich--;
			};
			iWhich++;

			// we are needing to adjust graph_r and interp graphdata

			pos.y = pVertex->pos.y;
			pos.x = pVertex->pos.x*CUTAWAYANGLE; // can leave graph_r undisturbed

			pos0 = pTri->cornerptr[0]->pos;
			pos1 = pTri->cornerptr[1]->pos;
			pos2 = pTri->cornerptr[2]->pos;

			dist0 = sqrt((pos0 - pos).dot(pos0 - pos));
			dist1 = sqrt((pos1 - pos).dot(pos1 - pos));
			dist2 = sqrt((pos2 - pos).dot(pos2 - pos));

			wt0 = 1.0f / dist0;
			wt1 = 1.0f / dist1;
			wt2 = 1.0f / dist2;
			wttotal = wt0 + wt1 + wt2;
			wt0 /= wttotal;
			wt1 /= wttotal;
			wt2 /= wttotal;
			// Not a great way it has to be said.

			if ((bTdata == false) && (bAcceldata == false)) {
				y0 = p_temphost3[(pTri->cornerptr[0] - pX->X) + BEGINNING_OF_CENTRAL];
				y1 = p_temphost3[(pTri->cornerptr[1] - pX->X) + BEGINNING_OF_CENTRAL];
				y2 = p_temphost3[(pTri->cornerptr[2] - pX->X) + BEGINNING_OF_CENTRAL];
				graphdata[0][asdf] = wt0*y0 + wt1*y1 + wt2*y2;
				if (fabs(graphdata[0][asdf]) > maximum[0]) maximum[0] = fabs(graphdata[0][asdf]);

				if (numgraphs > 1) {
					y0 = p_temphost4[(pTri->cornerptr[0] - pX->X) + BEGINNING_OF_CENTRAL];
					y1 = p_temphost4[(pTri->cornerptr[1] - pX->X) + BEGINNING_OF_CENTRAL];
					y2 = p_temphost4[(pTri->cornerptr[2] - pX->X) + BEGINNING_OF_CENTRAL];
					graphdata[1][asdf] = wt0*y0 + wt1*y1 + wt2*y2;
					if (fabs(graphdata[1][asdf]) > maximum[1]) maximum[1] = fabs(graphdata[1][asdf]);
				};

				if (numgraphs > 2) {
					y0 = p_temphost5[(pTri->cornerptr[0] - pX->X) + BEGINNING_OF_CENTRAL];
					y1 = p_temphost5[(pTri->cornerptr[1] - pX->X) + BEGINNING_OF_CENTRAL];
					y2 = p_temphost5[(pTri->cornerptr[2] - pX->X) + BEGINNING_OF_CENTRAL];
					graphdata[2][asdf] = wt0*y0 + wt1*y1 + wt2*y2;
					if (fabs(graphdata[2][asdf]) > maximum[2]) maximum[2] = fabs(graphdata[2][asdf]);
				};
				if (numgraphs > 3) {
					y0 = p_temphost6[(pTri->cornerptr[0] - pX->X) + BEGINNING_OF_CENTRAL];
					y1 = p_temphost6[(pTri->cornerptr[1] - pX->X) + BEGINNING_OF_CENTRAL];
					y2 = p_temphost6[(pTri->cornerptr[2] - pX->X) + BEGINNING_OF_CENTRAL];
					graphdata[3][asdf] = wt0*y0 + wt1*y1 + wt2*y2;
					if (fabs(graphdata[3][asdf]) > maximum[3]) maximum[3] = fabs(graphdata[3][asdf]);
				}
			} else {
				// go through from 0 = conduction to 5 = dTe/dt itself
				// we have missed out compressive...

				if (bTdata) {
					for (int j = 0; j < 8; j++)
					{
						y0 = p_Tgraph_host[j][(pTri->cornerptr[0] - pX->X)];
						y1 = p_Tgraph_host[j][(pTri->cornerptr[1] - pX->X)];
						y2 = p_Tgraph_host[j][(pTri->cornerptr[2] - pX->X)];
						graphdata[j][asdf] = wt0*y0 + wt1*y1 + wt2*y2;
						if (fabs(graphdata[j][asdf]) > maximum[j]) maximum[j] = fabs(graphdata[j][asdf]);
					}
				} else {
					int j;
					j = 1; // total
					y0 = p_accelgraph_host[j][(pTri->cornerptr[0] - pX->X)];
					y1 = p_accelgraph_host[j][(pTri->cornerptr[1] - pX->X)];
					y2 = p_accelgraph_host[j][(pTri->cornerptr[2] - pX->X)];
					graphdata[0][asdf] = wt0*y0 + wt1*y1 + wt2*y2;

					
					j = 3; // vxB
					y0 = p_accelgraph_host[j][(pTri->cornerptr[0] - pX->X)];
					y1 = p_accelgraph_host[j][(pTri->cornerptr[1] - pX->X)];
					y2 = p_accelgraph_host[j][(pTri->cornerptr[2] - pX->X)];
					graphdata[1][asdf] = wt0*y0 + wt1*y1 + wt2*y2;
					j = 5; // pressure
					y0 = p_accelgraph_host[j][(pTri->cornerptr[0] - pX->X)];
					y1 = p_accelgraph_host[j][(pTri->cornerptr[1] - pX->X)];
					y2 = p_accelgraph_host[j][(pTri->cornerptr[2] - pX->X)];
					graphdata[2][asdf] = wt0*y0 + wt1*y1 + wt2*y2;
					j = 6; // neutral soak
					y0 = p_accelgraph_host[j][(pTri->cornerptr[0] - pX->X)];
					y1 = p_accelgraph_host[j][(pTri->cornerptr[1] - pX->X)];
					y2 = p_accelgraph_host[j][(pTri->cornerptr[2] - pX->X)];
					graphdata[3][asdf] = wt0*y0 + wt1*y1 + wt2*y2;
										
					j = 8; // viscosity
					y0 = p_accelgraph_host[j][(pTri->cornerptr[0] - pX->X)];
					y1 = p_accelgraph_host[j][(pTri->cornerptr[1] - pX->X)];
					y2 = p_accelgraph_host[j][(pTri->cornerptr[2] - pX->X)];
					graphdata[4][asdf] = wt0*y0 + wt1*y1 + wt2*y2;

					j = 9; // ionization
					y0 = p_accelgraph_host[j][(pTri->cornerptr[0] - pX->X)];
					y1 = p_accelgraph_host[j][(pTri->cornerptr[1] - pX->X)];
					y2 = p_accelgraph_host[j][(pTri->cornerptr[2] - pX->X)];
					graphdata[5][asdf] = wt0*y0 + wt1*y1 + wt2*y2;
					
					j = 10; // advection
					y0 = p_accelgraph_host[j][(pTri->cornerptr[0] - pX->X)];
					y1 = p_accelgraph_host[j][(pTri->cornerptr[1] - pX->X)];
					y2 = p_accelgraph_host[j][(pTri->cornerptr[2] - pX->X)];
					graphdata[6][asdf] = wt0*y0 + wt1*y1 + wt2*y2;

					// works if comment here
					
				//	printf("%d ", asdf);
					for (int j = 0; j < 7; j++)
					{
				//		printf("%d", j);
						if (fabs(graphdata[j][asdf]) > maximum[0]) {
							maximum[0] = fabs(graphdata[j][asdf]);
				//			printf("maximum %1.9E\n", maximum[0]);
						}
					}

					// does it work if comment here? no
					
					j = 11; // grad_y Az
					y0 = p_accelgraph_host[j][(pTri->cornerptr[0] - pX->X)];
					y1 = p_accelgraph_host[j][(pTri->cornerptr[1] - pX->X)];
					y2 = p_accelgraph_host[j][(pTri->cornerptr[2] - pX->X)];
					graphdata[7][asdf] = wt0*y0 + wt1*y1 + wt2*y2;
					if (fabs(graphdata[7][asdf]) > maximum[7]) maximum[7] = fabs(graphdata[7][asdf]);
					
				}
			}
		}; // found triangle		
	}; // asdf	
	if ((bTdata == false) && (bAcceldata == false)) {		
		maximum[3] = max(maximum[3], maximum[2]);
		maximum[2] = maximum[3];
	} else {
		// for dT graphs, let maximum be overall
		if (bTdata) {
			for (int j = 1; j <= 6; j++)
				maximum[j] = max(maximum[j], maximum[j - 1]);
			for (int j = 5; j >= 0; j--)
				maximum[j] = maximum[j + 1];
		} else {
			printf("here:\n");

			for (int j = 1; j < 7; j++)
				maximum[j] = maximum[0];
		}
	}
}


void RefreshGraphs(TriMesh & X, // only not const because of such as Reset_vertex_nvT
	const int iGraphsFlag)
{
	D3DXMATRIXA16 matWorld;
	Vertex * pVertex;
	long iVertex;
	plasma_data * pdata;
	int offset_v, offset_T;
	char buff[256];
	sprintf(buff, "%5.2f ns", evaltime*1.0e9);
	f64 overc;
	char buffer[256];
	overc = 1.0 / c_;
	float x, y, z;
	float zeroplane = 0.0f;
	int i;
	int iGraph;
	char graphname[4][128] = { "Azdot","Azdotdot","Lap Az","-4pi/c Jz" };
	char Tgraphname[9][128] = { "conduction","ionization","viscosity","frictional","interspecies","dTe/dt total","compressive" ,
	"DnT","undefined"};
	char accelgraphname[9][128] = { "dvy/dt total", "v x B", "pressure", "neutral soak","viscosity", "ionization", "advection","grad_y Az"};

	float const MAXX = 11.0f;
	float const MAXY = 6.0f;

	vertex1 linedata[10000];
	vertex1 linedata2[12];
	
	switch (iGraphsFlag) {
		
	case ONE_D:

		// We are going to have to think about using LineTo the way it is done in RenderGraphs
		// let's start by rendering in the x-y plane and we can let the present camera look at it
		printf("\n\nGot to here: ONE_D\n\n");
		
		// Create data:
		Create1DGraphingData(&X);
		
		Graph[6].SetEyeAndLookat(newEye, newLookat); // sets matView not matProj
		printf("Eye %f %f %f\n", newEye.x, newEye.y, newEye.z);
		Direct3D.pd3dDevice->SetViewport(&(Graph[6].vp));

		D3DXMatrixIdentity(&matWorld);
		//D3DXMatrixIdentity(&Graph[6].matProj); // ???????????????
		Direct3D.pd3dDevice->SetTransform(D3DTS_WORLD, &matWorld);
		Direct3D.pd3dDevice->SetTransform(D3DTS_VIEW, &(Graph[6].matView));
		Direct3D.pd3dDevice->SetTransform(D3DTS_PROJECTION, &(Graph[6].matProj));
		
		Direct3D.pd3dDevice->Clear(0, NULL, D3DCLEAR_TARGET | D3DCLEAR_ZBUFFER,
			D3DCOLOR_XRGB(250, 255, 250), 1.0f, 0);

		if (SUCCEEDED(Direct3D.pd3dDevice->BeginScene()))
		{			
//			x = (float)(-sin(HALFANGLE)*DEVICE_RADIUS_INSULATOR_OUTER)*xzscale;
//			z = (float)(cos(HALFANGLE)*DEVICE_RADIUS_INSULATOR_OUTER)*xzscale;
//			y = zeroplane;
//			linedata[0].x = x; linedata[0].y = y; linedata[0].z = z;
//			x = (float)(-sin(HALFANGLE)*DOMAIN_OUTER_RADIUS)*xzscale;
//			z = (float)(cos(HALFANGLE)*DOMAIN_OUTER_RADIUS)*xzscale;
//			linedata[1].x = x; linedata[1].y = y; linedata[1].z = z;
//			for (i = 0; i < 12; i++)
//				linedata[i].colour = 0xff000000;
//			Direct3D.pd3dDevice->SetFVF(point_fvf);
//			Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, 1, linedata, sizeof(vertex1));
//
//			x = (float)(sin(HALFANGLE)*DEVICE_RADIUS_INSULATOR_OUTER)*xzscale;
//			z = (float)(cos(HALFANGLE)*DEVICE_RADIUS_INSULATOR_OUTER)*xzscale;
//			y = zeroplane;
//			linedata[0].x = x; linedata[0].y = y; linedata[0].z = z;
//			x = (float)(sin(HALFANGLE)*DOMAIN_OUTER_RADIUS)*xzscale;
//			z = (float)(cos(HALFANGLE)*DOMAIN_OUTER_RADIUS)*xzscale;
//			linedata[1].x = x; linedata[1].y = y; linedata[1].z = z;
//			for (i = 0; i < 12; i++)
//				linedata[i].colour = 0;
//			Direct3D.pd3dDevice->SetFVF(point_fvf);
//			Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, 1, linedata, sizeof(vertex1));
			Direct3D.pd3dDevice->SetFVF(point_fvf);

			real theta = -HALFANGLE;
			real r = 3.44;

			linedata[0].x = -MAXX;
			linedata[0].z = 3.44*xzscale;
			linedata[0].y = 0.0f;
			linedata[0].colour = 0xff888888; // grey
			
			linedata[1].x = -linedata[0].x;
			linedata[1].y = 0.0f;
			linedata[1].z = linedata[0].z;
			linedata[1].colour = linedata[0].colour;

			Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, 1, linedata, sizeof(vertex1));
			
			for (iGraph = 0; iGraph < numgraphs; iGraph++)
			{			
				linedata[0].x = -MAXX;
				linedata[0].z = 3.44*xzscale;
				linedata[0].y = MAXY + 4.0f-0.9f*(float)iGraph;
				int asdf = 0;
				linedata[asdf].colour = 0xff000000;
				if (iGraph == 1) linedata[asdf].colour = 0xff0022ff;
				if (iGraph == 2) linedata[asdf].colour = 0xffff0055;
				if (iGraph == 3) linedata[asdf].colour = 0xff22ff00;
				linedata[1].x = linedata[0].x + 1.0f;
				linedata[1].y = linedata[0].y;
				linedata[1].z = linedata[0].z;
				linedata[1].colour = linedata[0].colour;
				Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, 1, linedata, sizeof(vertex1));
				
				Graph[6].RenderLabel2(graphname[iGraph], linedata[1].x + 1.0f, linedata[1].y-0.3f, linedata[1].z,0);

				for (asdf = 0; asdf < num_graph_data_points; asdf++)
				{
					linedata[asdf].x = (float)(MAXX - 2.0*MAXX*((graph_r[asdf] - INNER_A_BOUNDARY) /
						(DOMAIN_OUTER_RADIUS - INNER_A_BOUNDARY)));
					
					// map 0 to 0.0f, maximum[iGraph] to MAXY and -maximum[iGraph] to MINY
					// Decide on graph scales maximum[] in preceding bit of code
					linedata[asdf].y = (float)( MAXY*graphdata[iGraph][asdf] / maximum[iGraph]);					
					linedata[asdf].z = 3.44f*xzscale;					
					linedata[asdf].colour = 0xff000000;
					if (iGraph == 1) linedata[asdf].colour = 0xff0022ff;
					if (iGraph == 2) linedata[asdf].colour = 0xffff0055;
					if (iGraph == 3) linedata[asdf].colour = 0xff22ff00;
				};
				Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, num_graph_data_points-1, linedata, sizeof(vertex1));

				sprintf(buffer, "%2.2E", maximum[iGraph]);
				Graph[6].RenderLabel2(buffer,  // text
					MAXX*0.66f+1.2f*(float)iGraph,
					MAXY,
					linedata[0].z, 0, linedata[0].colour);
				sprintf(buffer, "0.0");
				Graph[6].RenderLabel2(buffer,  // text
					MAXX*0.66f+1.2f*(float)iGraph,
					0.0f,
					linedata[0].z, 0, linedata[0].colour);
				sprintf(buffer, "-%2.2E", maximum[iGraph]);
				Graph[6].RenderLabel2(buffer,  // text
					MAXX*0.66f + 1.2f*(float)iGraph,
					-MAXY,
					linedata[0].z, 0, linedata[0].colour);
			};
			

			// Vertical lines:
			for (int i = 0; i < 9; i++)
			{
				x = 0.16*(-r*xzscale + 2.0*r*xzscale*(((real)i) / 8.0));
				z = 3.44*xzscale;// (float)(cos(HALFANGLE)*DEVICE_RADIUS_INSULATOR_OUTER)*xzscale;
				
				linedata[0].x = x; linedata[0].z = z;
				linedata[1].x = x; linedata[1].z = z;
				linedata[0].colour = 0xff220011;
				linedata[1].colour = 0xff220011;
				linedata[0].y = -6.8f;// GRAPHIC_MIN_Y - 1.0f;  
				linedata[1].y = ((i == 0) || (i == 8)) ? 6.0f:0.0f;// GRAPHIC_MAX_Y + 2.5f;

				Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, 1, linedata, sizeof(vertex1));

				sprintf(buffer, "%5.2f", INNER_A_BOUNDARY + (1.0-((real)i) / 8.0)*(DOMAIN_OUTER_RADIUS-INNER_A_BOUNDARY));
				Graph[6].RenderLabel2(buffer,  // text
					linedata[0].x,
					-7.6f,
					linedata[0].z, 0);

			};
			//DXChk(mFX->SetValue(mhEyePos, &Eye, sizeof(D3DXVECTOR3)));

			linedata[0].x = -0.16*r*xzscale;
			linedata[0].y = 0.0f;
			linedata[0].z = 3.44*xzscale;
			linedata[0].colour = 0xff000000; // 

			linedata[1].x = 0.16*r*xzscale;
			linedata[1].y = 0.0f;
			linedata[1].z = linedata[0].z;
			linedata[1].colour = linedata[0].colour;

			Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, 1, linedata, sizeof(vertex1));
			
			Direct3D.pd3dDevice->EndScene();
		} else {
			printf("BeginScene failed!\n\n");
			getch();
		}

		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = p_temphost3[iVertex+BEGINNING_OF_CENTRAL];
			++pVertex;
			++pdata;
		}
		Graph[4].DrawSurface("Azdot",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			true,
			GRAPH_AZDOT, &X);

		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = p_temphost4[iVertex + BEGINNING_OF_CENTRAL];
			++pVertex;
			++pdata;
		}
		Graph[1].DrawSurface("Azdotdot",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			true,
			GRAPH_AZDOT, &X);

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = p_temphost5[iVertex + BEGINNING_OF_CENTRAL];
			++pdata;
		}
		Graph[3].DrawSurface("Lap Az",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			true,
			GRAPH_LAPAZ, &X);

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if ((pVertex->flags == DOMAIN_VERTEX) || (pVertex->flags == OUTERMOST))
			{
				pdata->temp.x = p_temphost6[iVertex + BEGINNING_OF_CENTRAL];
			}
			else {
				pdata->temp.x = 0.0;
			}
			++pdata;
		}
		Graph[5].DrawSurface("Jz",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			true,
			GRAPH_JZ, &X);
		
		break;

	case DTGRAPH:

		// We are going to have to think about using LineTo the way it is done in RenderGraphs
		// let's start by rendering in the x-y plane and we can let the present camera look at it
		printf("\n\nRefreshGraphs: DTGRAPHS\n\n");

		// Create data:
		Create1DGraphingData(&X, true);

		Graph[6].SetEyeAndLookat(newEye, newLookat); // sets matView not matProj
		printf("Eye %f %f %f\n", newEye.x, newEye.y, newEye.z);
		Direct3D.pd3dDevice->SetViewport(&(Graph[6].vp));

		D3DXMatrixIdentity(&matWorld);
		//D3DXMatrixIdentity(&Graph[6].matProj); // ???????????????
		Direct3D.pd3dDevice->SetTransform(D3DTS_WORLD, &matWorld);
		Direct3D.pd3dDevice->SetTransform(D3DTS_VIEW, &(Graph[6].matView));
		Direct3D.pd3dDevice->SetTransform(D3DTS_PROJECTION, &(Graph[6].matProj));

		Direct3D.pd3dDevice->Clear(0, NULL, D3DCLEAR_TARGET | D3DCLEAR_ZBUFFER,
			D3DCOLOR_XRGB(250, 255, 250), 1.0f, 0);

		if (SUCCEEDED(Direct3D.pd3dDevice->BeginScene()))
		{
			
			Direct3D.pd3dDevice->SetFVF(point_fvf);

			real theta = -HALFANGLE;
			real r = 3.44;

			linedata[0].x = -MAXX;
			linedata[0].z = 3.44*xzscale;
			linedata[0].y = 0.0f;
			linedata[0].colour = 0xff888888; // grey

			linedata[1].x = -linedata[0].x;
			linedata[1].y = 0.0f;
			linedata[1].z = linedata[0].z;
			linedata[1].colour = linedata[0].colour;

			Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, 1, linedata, sizeof(vertex1));

			D3DCOLOR colour;
			for (iGraph = 0; iGraph < 7; iGraph++)
			{
				colour = 0xffffaa00; // conduction: orange red
				if (iGraph == 1) colour = 0xff0000ff; // ionization: royal blue
				if (iGraph == 2) colour = 0xff009999; // viscosity: aqua
				if (iGraph == 3) colour = 0xffd500ff; // resistive: heliotrope
				if (iGraph == 4) colour = 0xff00ff00; // soak: green
				if (iGraph == 5) colour = 0xff000000; // total
				if (iGraph == 6) colour = 0xff906545; // compressive: brown


				linedata[0].x = -MAXX;
				linedata[0].z = 3.44*xzscale;
				linedata[0].y = MAXY + 4.0f - 0.9f*(float)iGraph;
				int asdf = 0;
				linedata[asdf].colour = colour;
				linedata[1].x = linedata[0].x + 1.0f;
				linedata[1].y = linedata[0].y;
				linedata[1].z = linedata[0].z;
				linedata[1].colour = linedata[0].colour;
				Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, 1, linedata, sizeof(vertex1));

				Graph[6].RenderLabel2(Tgraphname[iGraph], linedata[1].x + 1.0f, linedata[1].y - 0.3f, linedata[1].z, 0);

				for (asdf = 0; asdf < num_graph_data_points; asdf++)
				{
					linedata[asdf].x = (float)(MAXX - 2.0*MAXX*((graph_r[asdf] - INNER_A_BOUNDARY) /
						(DOMAIN_OUTER_RADIUS - INNER_A_BOUNDARY)));

					// map 0 to 0.0f, maximum[iGraph] to MAXY and -maximum[iGraph] to MINY
					// Decide on graph scales maximum[] in preceding bit of code
					linedata[asdf].y = (float)(MAXY*graphdata[iGraph][asdf] / maximum[iGraph]);
					linedata[asdf].z = 3.44f*xzscale;
					linedata[asdf].colour = colour; // conduction: orange red		

					if (asdf == 200) printf("linedata[200].y %1.9E  | ", linedata[asdf].y);

				};
				Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, num_graph_data_points - 1, linedata, sizeof(vertex1));

				sprintf(buffer, "%2.2E", maximum[iGraph]);
				Graph[6].RenderLabel2(buffer,  // text
					MAXX*0.4f + 1.6f*(float)iGraph,
					MAXY,
					linedata[0].z, 0, linedata[0].colour);
				sprintf(buffer, "0.0");
				Graph[6].RenderLabel2(buffer,  // text
					MAXX*0.4f + 1.6f*(float)iGraph,
					0.0f,
					linedata[0].z, 0, linedata[0].colour);
				sprintf(buffer, "-%2.2E", maximum[iGraph]);
				Graph[6].RenderLabel2(buffer,  // text
					MAXX*0.4f + 1.6f*(float)iGraph,
					-MAXY,
					linedata[0].z, 0, linedata[0].colour);
			};


			// Vertical lines:
			for (int i = 0; i < 9; i++)
			{
				x = 0.16*(-r*xzscale + 2.0*r*xzscale*(((real)i) / 8.0));
				z = 3.44*xzscale;// (float)(cos(HALFANGLE)*DEVICE_RADIUS_INSULATOR_OUTER)*xzscale;

				linedata[0].x = x; linedata[0].z = z;
				linedata[1].x = x; linedata[1].z = z;
				linedata[0].colour = 0xff220011;
				linedata[1].colour = 0xff220011;
				linedata[0].y = -6.8f;// GRAPHIC_MIN_Y - 1.0f;  
				linedata[1].y = ((i == 0) || (i == 8)) ? 6.0f : 0.0f;// GRAPHIC_MAX_Y + 2.5f;

				Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, 1, linedata, sizeof(vertex1));

				sprintf(buffer, "%5.2f", INNER_A_BOUNDARY + (1.0 - ((real)i) / 8.0)*(DOMAIN_OUTER_RADIUS - INNER_A_BOUNDARY));
				Graph[6].RenderLabel2(buffer,  // text
					linedata[0].x,
					-7.6f,
					linedata[0].z, 0);

			};
			//DXChk(mFX->SetValue(mhEyePos, &Eye, sizeof(D3DXVECTOR3)));

			linedata[0].x = -0.16*r*xzscale;
			linedata[0].y = 0.0f;
			linedata[0].z = 3.44*xzscale;
			linedata[0].colour = 0xff000000; // 

			linedata[1].x = 0.16*r*xzscale;
			linedata[1].y = 0.0f;
			linedata[1].z = linedata[0].z;
			linedata[1].colour = linedata[0].colour;

			Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, 1, linedata, sizeof(vertex1));

			Direct3D.pd3dDevice->EndScene();
		}
		else {
			printf("BeginScene failed!\n\n");
			getch();
		}

		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = p_Tgraph_host[5][iVertex];
			++pVertex;
			++pdata;
		}
		Graph[4].DrawSurface("dTe/dt",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			SEGUE_COLOUR, (real *)(&(X.pData[0].Te)),
			false,
			GRAPH_DTE, &X);

		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = p_Tgraph_host[7][iVertex];
			++pVertex;
			++pdata;
		}
		Graph[1].DrawSurface("d/dt nTe",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false,
			GRAPH_DNT, &X);

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = p_Tgraph_host[5][iVertex];
			++pdata;
		}
		Graph[3].DrawSurface("n",
			DATA_HEIGHT, (real *)(&(X.pData[0].n)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false,
			GRAPH_ION_N, &X);

		Graph[5].DrawSurface("Te",
			DATA_HEIGHT, (real *)(&(X.pData[0].Te)),
			SEGUE_COLOUR, (real *)(&(X.pData[0].Te)),
			false,
			GRAPH_ELEC_T, &X);

		break;

	case ACCELGRAPHS:

		// We are going to have to think about using LineTo the way it is done in RenderGraphs
		// let's start by rendering in the x-y plane and we can let the present camera look at it
		printf("\n\nRefreshGraphs: ACCELGRAPHS\n\n");

		// Create data:
		Create1DGraphingData(&X, false, true);
		
		Graph[6].SetEyeAndLookat(newEye, newLookat); // sets matView not matProj
		printf("Eye %f %f %f\n", newEye.x, newEye.y, newEye.z);
		Direct3D.pd3dDevice->SetViewport(&(Graph[6].vp));

		D3DXMatrixIdentity(&matWorld);
		//D3DXMatrixIdentity(&Graph[6].matProj); // ???????????????
		Direct3D.pd3dDevice->SetTransform(D3DTS_WORLD, &matWorld);
		Direct3D.pd3dDevice->SetTransform(D3DTS_VIEW, &(Graph[6].matView));
		Direct3D.pd3dDevice->SetTransform(D3DTS_PROJECTION, &(Graph[6].matProj));

		Direct3D.pd3dDevice->Clear(0, NULL, D3DCLEAR_TARGET | D3DCLEAR_ZBUFFER,
			D3DCOLOR_XRGB(250, 255, 250), 1.0f, 0);

		if (SUCCEEDED(Direct3D.pd3dDevice->BeginScene()))
		{
			Direct3D.pd3dDevice->SetFVF(point_fvf);

			real theta = -HALFANGLE;
			real r = 3.44;

			linedata[0].x = -MAXX;
			linedata[0].z = 3.44*xzscale;
			linedata[0].y = 0.0f;
			linedata[0].colour = 0xff888888; // grey

			linedata[1].x = -linedata[0].x;
			linedata[1].y = 0.0f;
			linedata[1].z = linedata[0].z;
			linedata[1].colour = linedata[0].colour;

			Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, 1, linedata, sizeof(vertex1));
			
			D3DCOLOR colour;
			for (iGraph = 0; iGraph < 8; iGraph++)
			{
				colour = 0xff000000; // total: 
				if (iGraph == 1) colour = 0xffd500ff; // vxB: heliotrope
				if (iGraph == 2) colour = 0xffff2200; // pressure: red
				if (iGraph == 3) colour = 0xff00ff33; // soak:   use green
				if (iGraph == 4) colour = 0xff009999; // viscosity: aqua
				if (iGraph == 5) colour = 0xff0000ff; // ionization : royal blue
				if (iGraph == 6) colour = 0xff906545; // advection :  brown
				if (iGraph == 7) colour = 0xffeecd00; // grady_Az : olive?

				linedata[0].x = -MAXX;
				linedata[0].z = 3.44*xzscale;
				linedata[0].y = MAXY + 4.0f - 0.9f*(float)iGraph;
				int asdf = 0;
				linedata[asdf].colour = colour;
				linedata[1].x = linedata[0].x + 1.0f;
				linedata[1].y = linedata[0].y;
				linedata[1].z = linedata[0].z;
				linedata[1].colour = linedata[0].colour;
				Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, 1, linedata, sizeof(vertex1));

				Graph[6].RenderLabel2(accelgraphname[iGraph], linedata[1].x + 1.0f, linedata[1].y - 0.3f, linedata[1].z, 0);

				for (asdf = 0; asdf < num_graph_data_points; asdf++)
				{
					linedata[asdf].x = (float)(MAXX - 2.0*MAXX*((graph_r[asdf] - INNER_A_BOUNDARY) /
						(DOMAIN_OUTER_RADIUS - INNER_A_BOUNDARY)));

					// map 0 to 0.0f, maximum[iGraph] to MAXY and -maximum[iGraph] to MINY
					// Decide on graph scales maximum[] in preceding bit of code
					linedata[asdf].y = (float)(MAXY*graphdata[iGraph][asdf] / maximum[iGraph]);
					linedata[asdf].z = 3.44f*xzscale;
					linedata[asdf].colour = colour; // conduction: orange red		

				//	if (asdf == 200) printf("linedata[200].y %1.9E  | ", linedata[asdf].y);

				};
				Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, num_graph_data_points - 1, linedata, sizeof(vertex1));

				sprintf(buffer, "%2.2E", maximum[iGraph]);
				Graph[6].RenderLabel2(buffer,  // text
					MAXX*0.4f + ((iGraph == 7)?1.6f:0.0f),
					MAXY,
					linedata[0].z, 0, 0xff000000);
				sprintf(buffer, "0.0");
				Graph[6].RenderLabel2(buffer,  // text
					MAXX*0.4f + ((iGraph == 7) ? 1.6f : 0.0f),
					0.0f,
					linedata[0].z, 0, 0xff000000);
				sprintf(buffer, "-%2.2E", maximum[iGraph]);
				Graph[6].RenderLabel2(buffer,  // text
					MAXX*0.4f + ((iGraph == 7) ? 1.6f : 0.0f),
					-MAXY,
					linedata[0].z, 0, 0xff000000);
			};
			
			// Vertical lines:
			for (int i = 0; i < 9; i++)
			{
				x = 0.16*(-r*xzscale + 2.0*r*xzscale*(((real)i) / 8.0));
				z = 3.44*xzscale;// (float)(cos(HALFANGLE)*DEVICE_RADIUS_INSULATOR_OUTER)*xzscale;

				linedata[0].x = x; linedata[0].z = z;
				linedata[1].x = x; linedata[1].z = z;
				linedata[0].colour = 0xff220011;
				linedata[1].colour = 0xff220011;
				linedata[0].y = -6.8f;// GRAPHIC_MIN_Y - 1.0f;  
				linedata[1].y = ((i == 0) || (i == 8)) ? 6.0f : 0.0f;// GRAPHIC_MAX_Y + 2.5f;

				Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, 1, linedata, sizeof(vertex1));

				sprintf(buffer, "%5.2f", INNER_A_BOUNDARY + (1.0 - ((real)i) / 8.0)*(DOMAIN_OUTER_RADIUS - INNER_A_BOUNDARY));
				Graph[6].RenderLabel2(buffer,  // text
					linedata[0].x,
					-7.6f,
					linedata[0].z, 0);

			};
			//DXChk(mFX->SetValue(mhEyePos, &Eye, sizeof(D3DXVECTOR3)));

			linedata[0].x = -0.16*r*xzscale;
			linedata[0].y = 0.0f;
			linedata[0].z = 3.44*xzscale;
			linedata[0].colour = 0xff000000; // 

			linedata[1].x = 0.16*r*xzscale;
			linedata[1].y = 0.0f;
			linedata[1].z = linedata[0].z;
			linedata[1].colour = linedata[0].colour;

			Direct3D.pd3dDevice->DrawPrimitiveUP(D3DPT_LINESTRIP, 1, linedata, sizeof(vertex1));

			Direct3D.pd3dDevice->EndScene();
		}
		else {
			printf("BeginScene failed!\n\n");
			getch();
		}

		printf("here3 ... \n");

		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = p_accelgraph_host[0][iVertex];
			pdata->temp.y = p_accelgraph_host[1][iVertex];
			++pVertex;
			++pdata;
		}
		Graph[4].DrawSurface("dvxy/dt",
			VELOCITY_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false,
			GRAPH_AXY, &X);

		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = p_accelgraph_host[2][iVertex];
			pdata->temp.y = p_accelgraph_host[3][iVertex];
			++pVertex;
			++pdata;
		}
		Graph[1].DrawSurface("axy : v x B",
			VELOCITY_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false,
			GRAPH_AXY2, &X);

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = p_accelgraph_host[4][iVertex];
			pdata->temp.y = p_accelgraph_host[5][iVertex];
			++pdata;
		}
		Graph[3].DrawSurface("axy : pressure",
			VELOCITY_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false,
			GRAPH_AXY3, &X);

		Graph[5].DrawSurface("vxy",
			VELOCITY_HEIGHT, (real *)(&(X.pData[0].vxy)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].vxy)),
			false,
			GRAPH_ION_V, &X);
		
		break;

		/*
		case JXY_RHO_EXY_GRADPHI_AXYDOTOC_AXY:

		X.Setup_J(); // the others can already exist.

		Graph[4].bDisplayTimestamp = true;

		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		if (pVertex->flags == DOMAIN_VERTEX) {
		pVertex->temp2.x = q * (pVertex->Ion.mass - pVertex->Elec.mass) / pVertex->AreaCell;
		}
		else {
		pVertex->temp2.x = 0.0;
		};
		pVertex->Adot /= c;
		++pVertex;
		}

		Graph[0].DrawSurface("Exy[statV/cm]",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].E)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].E)),
		true,
		GRAPH_EXY, &X);

		Graph[1].DrawSurface("Adotxy/c[statV/cm]",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].Adot.x)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].Adot.x)),
		true,
		GRAPH_ADOTXY, &X);

		Graph[2].DrawSurface("Jxy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].Temp.x)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].Temp.x)),
		false, // no inner mesh display.
		GRAPH_JXY, &X);

		Graph[3].DrawSurface("phidot[statV/s]",
		DATA_HEIGHT, (real *)(&(X.X[0].phidot)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].phidot)),
		true,
		GRAPH_PHIDOT, &X);

		Graph[4].DrawSurface("rho",
		DATA_HEIGHT, (real *)(&(X.X[0].temp2.x)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].temp2.x)),
		false, // no inner mesh display.
		GRAPH_RHO, &X);

		Graph[5].DrawSurface("phi[statV]",
		DATA_HEIGHT, (real *)(&(X.X[0].phi)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].phi)),
		true,
		GRAPH_PHI, &X);

		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		pVertex->Adot *= c;
		++pVertex;
		}
		break;
		case JZ_AZ_BXY_EZ_ADOTZOC_NVZ:

		X.Reset_vertex_nvT(SPECIES_ELEC);

		X.Setup_J(); // the others can already exist.

		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		pVertex->Adot /= c;
		++pVertex;
		}
		Graph[4].bDisplayTimestamp = true;

		Graph[0].DrawSurface("Ez[statV/cm]",
		DATA_HEIGHT, (real *)(&(X.X[0].E.z)),
		FLAG_SEGUE_COLOUR, (real *)(&(X.X[0].E.z)),
		false, // ??
		GRAPH_EZ, &X);

		Graph[1].DrawSurface("Az",
		DATA_HEIGHT, (real *)(&(X.X[0].A.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].A.z)),
		true,
		GRAPH_AZ, &X);
		Graph[2].DrawSurface("Jz",
		DATA_HEIGHT, (real *)(&(X.X[0].Temp.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].Temp.z)),
		false, // no inner mesh display.
		GRAPH_JZ, &X);
		Graph[3].DrawSurface("Bxy[Gs]",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].B)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].B)),
		true, // no inner mesh display: ??
		GRAPH_BXY, &X);

		Graph[4].DrawSurface("Adotz/c [statV/cm]",
		DATA_HEIGHT, (real *)(&(X.X[0].Adot.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].Adot.z)),
		true,
		GRAPH_AZ, &X);

		Graph[5].colourmax = Graph[2].colourmax;
		Graph[5].DrawSurface("Elec n",
		DATA_HEIGHT, (real *)(&(X.X[0].n)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].Temp.z)),
		false, // no inner mesh display
		GRAPH_ELEC_N, &X);

		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		pVertex->Adot *= c;
		++pVertex;
		}

		break;
		case SPECIES_ELECTRON2:

		X.Reset_vertex_nvT(SPECIES_ELEC);

		Graph[0].DrawSurface("Elec n [/cc]",
		DATA_HEIGHT, (real *)(&(X.X[0].n)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].v)),
		false, // no inner mesh display
		GRAPH_ELEC_N, &X);

		Graph[1].DrawSurface("v_e_xy[cm/s]",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].v)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].v)),
		false, // no inner mesh display
		GRAPH_ELEC_V, &X);

		Graph[3].DrawSurface("v_e_z[cm/s]",
		DATA_HEIGHT, (real *)(&(X.X[0].v.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].v.z)),
		false, // no inner mesh display.
		GRAPH_VEZ, &X);

		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		if (pVertex->flags == DOMAIN_VERTEX) {
		pVertex->temp2.x = (pVertex->Ion.mass - pVertex->Elec.mass) / pVertex->AreaCell;
		}
		else {
		pVertex->temp2.x = 0.0;
		};
		++pVertex;
		}
		Graph[2].bDisplayTimestamp = false;
		Graph[2].DrawSurface("n_i-n_e",
		DATA_HEIGHT, (real *)(&(X.X[0].temp2.x)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].temp2.x)),
		false, // no inner mesh display.
		GRAPH_NINE, &X);

		Graph[5].TickRescaling = 1.0 / kB;
		Graph[5].DrawSurface("Elec T [eV]",
		DATA_HEIGHT, (real *)(&(X.X[0].T)),
		SEGUE_COLOUR, (real *)(&(X.X[0].T)),
		false, // no inner mesh display
		GRAPH_ELEC_T, &X);
		Graph[5].TickRescaling = 1.0;

		offset_v = (real *)(&(X.X[0].v)) - (real *)(&(X.X[0]));
		offset_T = (real *)(&(X.X[0].T)) - (real *)(&(X.X[0]));

		Graph[4].SetEyePlan(GlobalPlanEye);
		Graph[4].boolDisplayMeshWireframe = true;
		Graph[4].boolClearZBufferBeforeWireframe = true;
		Graph[4].boolDisplayMainMesh = true;
		Graph[4].boolDisplayInnerMesh = false;
		Graph[4].boolDisplayScales = false;
		Graph[4].boolDisplayShadow = false;
		Graph[4].mhTech = Graph[4].mFX->GetTechniqueByName("VelociTech");
		Graph[4].colourmax = Graph[0].colourmax; // match colours
		Graph[4].SetDataWithColour(X, FLAG_VELOCITY_COLOUR, FLAG_FLAT_MESH, offset_v, offset_v,
		GRAPH_FLAT_WIRE_MESH);
		Graph[4].Render(buff, GlobalRenderLabels, &X);

		break;
		*/

case IONIZEGRAPH:
	printf("\n\nRefreshGraphs: IONIZEGRAPHS\n\n");

	// When we come to speed up graphs, make it so we can
	// just pass an array of f64. !!!!
	// Investigate graphs half an hour: what's up with the rest?

	// Move table, start running.
	// Can we bring back cutaway any how? 
	// Wanted acceleration graphs. 
	// Want to do a big run. 

	pVertex = X.X;
	pdata = X.pData + BEGINNING_OF_CENTRAL;
	for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
	{
		pdata->temp.x = p_graphdata1_host[iVertex];
		pdata->temp.y = p_graphdata2_host[iVertex]; // dn/dt /n
		++pVertex;
		++pdata;
	}
	Graph[0].DrawSurface("dn/dt",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		SEGUE_COLOUR, (real *)(&(X.pData[0].Te)),
		false,
		GRAPH_DNDT, &X);

	Graph[1].DrawSurface("dn/dt / n",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.y)),
		SEGUE_COLOUR, (real *)(&(X.pData[0].Te)),
		false,
		GRAPH_DNDT_OVER_n, &X);
	

	pVertex = X.X;
	pdata = X.pData + BEGINNING_OF_CENTRAL;
	for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
	{
		pdata->temp.x = p_graphdata3_host[iVertex]; // log10 n
		++pVertex;
		++pdata;
	}
	Graph[3].DrawSurface("log10(n)",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
		false,
		GRAPH_LOG10N, &X);

	pVertex = X.X;
	pdata = X.pData + BEGINNING_OF_CENTRAL;
	for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
	{
		pdata->temp.x = p_graphdata4_host[iVertex]; // dTe/dt
		pdata->temp.y = p_graphdata6_host[iVertex]; // n/nn
		++pVertex;
		++pdata;
	} 
	Graph[2].DrawSurface("dTe/dt[ionization]",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		SEGUE_COLOUR, (real *)(&(X.pData[0].Te)),
		false,
		GRAPH_DTEDT, &X);

	Graph[4].DrawSurface("n_e / n_total",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.y)),
		IONISE_COLOUR, (real *)(&(X.pData[0].temp.y)),
		false,
		GRAPH_FRACTION, &X);

	pVertex = X.X;
	pdata = X.pData + BEGINNING_OF_CENTRAL;
	for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
	{
		pdata->temp.x = p_graphdata5_host[iVertex]; // dvez/dt
		//if (pdata->vez != 0.0f) {
		//	pdata->temp.y = pdata->temp.x / (pdata->vez);
		//} else {
		//	pdata->temp.y = 0.0;
		//}
		++pVertex;
		++pdata;
	}
	printf("got to here 1");
	Graph[5].DrawSurface("accel ez[ionization]",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		AZSEGUE_COLOUR, (real *)(&(X.pData[0].vez)),
		false,
		GRAPH_AEZ1, &X);
	// Do we need another shader? Or can we reset limits?
	// see what scale is like.
	printf("got to here 2");

	break;




case OVERALL:
	printf("\n\nRefreshGraphs: OVERALL\n\n");

		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if ((pVertex->flags == DOMAIN_VERTEX) || (pVertex->flags == OUTERMOST))
			{
				pdata->temp.x = pdata->n + pdata->n_n;
				pdata->temp.y = pdata->n / (1.0 + pdata->temp.x);
			} else {
				pdata->temp.x = 0.0;
				pdata->temp.y = 0.0;
			}
			++pVertex;
			++pdata;
		}
		
		Graph[0].DrawSurface("n_n + n_ion",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			IONISE_COLOUR, (real *)(&(X.pData[0].temp.y)),
			false,
			GRAPH_TOTAL_N, &X);
		
		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if ((pVertex->flags == DOMAIN_VERTEX) || (pVertex->flags == OUTERMOST))
			{
				pdata->temp.x = (m_neutral_*pdata->n_n*pdata->v_n.x
					+ (m_ion_ + m_e_) * pdata->n*pdata->vxy.x) /
					(m_neutral_*pdata->n_n + (m_ion_ + m_e_)*pdata->n);
				pdata->temp.y = (m_neutral_*pdata->n_n*pdata->v_n.y
					+ (m_ion_ + m_e_) * pdata->n*pdata->vxy.y) /
					(m_neutral_*pdata->n_n + (m_ion_ + m_e_)*pdata->n);
			} else {
				pdata->temp.x = 0.0; pdata->temp.y = 0.0;
			}
			++pVertex;
			++pdata;
		}
		Graph[1].DrawSurface("sum[n_s v_s m_s]/sum[n_s m_s]",
			VELOCITY_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false, // no inner mesh display
			GRAPH_TOTAL_V, &X);

		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if ((pVertex->flags == DOMAIN_VERTEX) || (pVertex->flags == OUTERMOST))
			{
				pdata->temp.x = (pdata->n_n*pdata->Tn
					+ pdata->n*(pdata->Ti + pdata->Te)) /
					(pdata->n_n + pdata->n + pdata->n);
			} else {
				pdata->temp.x = 0.0; pdata->temp.y = 0.0;
			}
			++pVertex;
			++pdata;
		}
		Graph[3].TickRescaling = 1.0 / kB_;
		Graph[3].DrawSurface("sum[n_s T_s]/sum[n_s]",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			SEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false,
			GRAPH_TOTAL_T, &X);
		Graph[3].TickRescaling = 1.0;


		Graph[2].DrawSurface("Neutral n",
			DATA_HEIGHT, (real *)(&(X.pData[0].n_n)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].v_n)),
			false, // no inner mesh display
			GRAPH_NEUT_N, &X);
		Graph[4].DrawSurface("Neutral v",
			VELOCITY_HEIGHT, (real *)(&(X.pData[0].v_n)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].v_n)),
			false, // no inner mesh display
			GRAPH_NEUT_V, &X);


		Graph[5].TickRescaling = 1.0 / kB_;
		Graph[5].DrawSurface("Neutral T",
			DATA_HEIGHT, (real *)(&(X.pData[0].Tn)),
			SEGUE_COLOUR, (real *)(&(X.pData[0].Tn)),
			false, // no inner mesh display
			GRAPH_NEUT_T, &X);
		Graph[5].TickRescaling = 1.0;
		
		break;
	case SPECIES_ION:
		printf("\n\nRefreshGraphs: SPECIES_ION\n\n");

		Graph[3].TickRescaling = 1.0 / kB_;
		Graph[3].DrawSurface("Ion T",
			DATA_HEIGHT, (real *)(&(X.pData[0].Ti)),
			SEGUE_COLOUR, (real *)(&(X.pData[0].Ti)),
			false, // no inner mesh display
			GRAPH_ION_T, &X);
		Graph[3].TickRescaling = 1.0;

		// labels only appear on first 1 called.

		Graph[0].DrawSurface("Ion n",
			DATA_HEIGHT, (real *)(&(X.pData[0].n)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].vxy)),
			false, // no inner mesh display
			GRAPH_ION_N, &X);
		Graph[1].DrawSurface("Ion v",
			VELOCITY_HEIGHT, (real *)(&(X.pData[0].vxy)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].vxy)),
			false, // no inner mesh display
			GRAPH_ION_V, &X);

		// These are same so double up with elec.

		Graph[5].TickRescaling = 1.0 / kB_;
		Graph[5].DrawSurface("Elec T",
			DATA_HEIGHT, (real *)(&(X.pData[0].Te)),
			SEGUE_COLOUR, (real *)(&(X.pData[0].Te)),
			false, // no inner mesh display
			GRAPH_ELEC_T, &X);
		Graph[5].TickRescaling = 1.0;

		break;

/*	case SPECIES_ELEC:

		Graph[0].DrawSurface("Elec n",
			DATA_HEIGHT, (real *)(&(X.pData[0].n)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].vxy)),
			false, // no inner mesh display
			GRAPH_ELEC_T, &X);
		// colours == 0.0 ... because v = 0
		// First........... let's understand why surface normals come out unpredictable.
		// Then............ let's go and see what it does with y values (in Render and .fx)

		Graph[1].DrawSurface("Elec v",
			VELOCITY_HEIGHT, (real *)(&(X.pData[0].vxy)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].vxy)),
			false, // no inner mesh display
			GRAPH_ELEC_V, &X);
		break;

		// In other cases, (and even for the above),
		// here is a good place to call the 
		// setup routines for temp variables.
		*/

case OHMSLAW:
	printf("\n\nRefreshGraphs: OHMSLAW\n\n");

		// 0. q/ m_e nu_sum 
		// 1. qn / m_e nu_sum
		// 2. nu_sum
		// 3. prediction of Jz from uniform Ez
		// 4. prediction of Jz from actual Ez
		// 5. Actual Jz
		
		// Let temphost1 = nu_en + nu_ei_effective
		// Let temphost2 = nu_en/temphost1


	// Cannot explain why, that comes out black and this doesn't.
	// Oh because colourmax has been set to 1 or not?

	// Yet the following crashes it. Bizarre? Maybe dividing by 0?
	
	overc = 1.0 / c_;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		pVertex = X.X;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if (pVertex->flags == DOMAIN_VERTEX) {
				pdata->temp.x = q_ / (m_e_ * (1.0 + p_temphost1[iVertex + BEGINNING_OF_CENTRAL]));
				pdata->temp.y = p_temphost2[iVertex + BEGINNING_OF_CENTRAL]; // colour
			} else {
				pdata->temp.x = 0.0;
				pdata->temp.y = 0.0;
			}
			++pdata;
			++pVertex;
		};
		Graph[0].DrawSurface("q over m nu_effective",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			PPN_COLOUR, (real *)(&(X.pData[0].temp.y)),
			false, // no inner mesh display.
			GRAPH_VRESPONSEOHMS, &X);

		
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		pVertex = X.X;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if (pVertex->flags == DOMAIN_VERTEX) {
				pdata->temp.x = q_*X.pData[iVertex + BEGINNING_OF_CENTRAL].n /
					(m_e_ * (1.0 + p_temphost1[iVertex + BEGINNING_OF_CENTRAL]));
				pdata->temp.y = p_temphost2[iVertex + BEGINNING_OF_CENTRAL]; // colour
			} else {
				pdata->temp.x = 0.0;
				pdata->temp.y = 0.0;
			};
			++pdata;
			++pVertex;
		};

		Graph[1].DrawSurface("qn / m nu_effective",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			PPN_COLOUR, (real *)(&(X.pData[0].temp.y)),
			false, // no inner mesh display.
			GRAPH_CONDUCTIVITYOHMS, &X);
			

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		pVertex = X.X;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if (pVertex->flags == DOMAIN_VERTEX) {
				pdata->temp.x = p_temphost1[iVertex + BEGINNING_OF_CENTRAL];
				pdata->temp.y = p_temphost2[iVertex + BEGINNING_OF_CENTRAL]; // colour
			};
			++pVertex;
			++pdata;
		};
		Graph[2].DrawSurface("nu_effective (blue=neut dominates)",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			PPN_COLOUR, (real *)(&(X.pData[0].temp.y)),
			false, // no inner mesh display.
			GRAPH_NU_EFFECTIVE, &X);
			
		
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		pVertex = X.X;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if (pVertex->flags == DOMAIN_VERTEX) {
				pdata->temp.x = EzStrength_*q_*q_*X.pData[iVertex + BEGINNING_OF_CENTRAL].n /
					(m_e_ * (1.0 + p_temphost1[iVertex + BEGINNING_OF_CENTRAL]));
			};
			++pdata;
		};
		Graph[3].DrawSurface("predict Jz (uniform Ez)",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),			
			false, // no inner mesh display.
			GRAPH_JZ, &X);
					
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		pVertex = X.X;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if (pVertex->flags == DOMAIN_VERTEX) {
				pdata->temp.x = (EzStrength_
					- X.pData[iVertex + BEGINNING_OF_CENTRAL].Azdot*overc
					)*q_*q_*X.pData[iVertex + BEGINNING_OF_CENTRAL].n /
					(m_e_ * (1.0 + p_temphost1[iVertex + BEGINNING_OF_CENTRAL]));
			};
			++pdata;
		};
		Graph[4].DrawSurface("predict Jz (Ez)",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false, // no inner mesh display.
			GRAPH_JZ, &X);
			
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		pVertex = X.X;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if (pVertex->flags == DOMAIN_VERTEX) {
				pdata->temp.x = q_*X.pData[iVertex + BEGINNING_OF_CENTRAL].n*
					(X.pData[iVertex + BEGINNING_OF_CENTRAL].viz - X.pData[iVertex + BEGINNING_OF_CENTRAL].vez);
			};
			++pdata;
		};
		Graph[5].DrawSurface("actual Jz",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false, // no inner mesh display.
			GRAPH_JZ, &X);

		break;

	case JZAZBXYEZ:
		printf("\n\nRefreshGraphs: JZAZBXYEZ\n\n");

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = q_ * pdata->n*(pdata->viz - pdata->vez);
			++pdata;
		};
		Graph[3].DrawSurface("Jz",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false, // no inner mesh display.
			GRAPH_JZ, &X);

		// create graph data for Ez : add Ez_strength*Ezshape to -Azdot/c
		overc = 1.0 / c_;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			X.pData[iVertex + BEGINNING_OF_CENTRAL].temp.y =
				-X.pData[iVertex + BEGINNING_OF_CENTRAL].Azdot*overc
				+ GetEzShape__(X.pData[iVertex + BEGINNING_OF_CENTRAL].pos.modulus())*EzStrength_;
		}
		Graph[2].DrawSurface("Ez",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.y)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)), // use Jz's colour
			false, 
			GRAPH_EZ, &X);

		Graph[0].DrawSurface("Az",
			DATA_HEIGHT, (real *)(&(X.pData[0].Az)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].Az)),
			true, GRAPH_AZ, &X);

//		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
//		{
//			X.pData[iVertex + BEGINNING_OF_CENTRAL].temp.x = p_B_host[iVertex + BEGINNING_OF_CENTRAL].x;
//			X.pData[iVertex + BEGINNING_OF_CENTRAL].temp.y = p_B_host[iVertex + BEGINNING_OF_CENTRAL].y;
//		}
		Graph[1].DrawSurface("Bxy",
		VELOCITY_HEIGHT, (real *)(&(X.pData[0].B.x)),
		VELOCITY_COLOUR, (real *)(&(X.pData[0].B.x)),
		false,
		GRAPH_BXY, &X);

		Graph[5].DrawSurface("vez",
			DATA_HEIGHT, (real *)(&(X.pData[0].vez)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)), // colour is for Jz?
			false, GRAPH_VEZ, &X);

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = temp_array_host[iVertex + BEGINNING_OF_CENTRAL];
			++pdata;
		};
		Graph[4].DrawSurface("Lap Az",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			true, GRAPH_LAPAZ, &X);
		break;

	case VIZVEZJZAZDOT:
		printf("\n\nRefreshGraphs: VIZVEZJZAZDOT\n\n");

		// Set Jz:
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = q_ * pdata->n*(pdata->viz - pdata->vez);
			++pdata;
		};

		Graph[0].DrawSurface("viz",
			DATA_HEIGHT, (real *)(&(X.pData[0].viz)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false, GRAPH_VIZ, &X);

		Graph[1].DrawSurface("vez",
			DATA_HEIGHT, (real *)(&(X.pData[0].vez)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false, GRAPH_VEZ, &X);

		Graph[2].DrawSurface("Azdot",
			DATA_HEIGHT, (real *)(&(X.pData[0].Azdot)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].Azdot)),
			true, GRAPH_AZDOT, &X);

		Graph[3].DrawSurface("Jz",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false, GRAPH_JZ, &X);

		break;
		/*
	case NEWSTUFF:

		// Too bad substep is not stated. We should divide by substep to give anything meaningful
		// in these graphs.

		// Let temphost3 = vez0
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
		pdata->temp.x = p_temphost3[iVertex + BEGINNING_OF_CENTRAL];
		++pdata;
		};
		Graph[0].DrawSurface("vez0 : vez = vez0 + sigma Ez",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
		false, // no inner mesh display.
		GRAPH_VEZ0, &X);

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
		pdata->temp.x = p_OhmsCoeffs_host[iVertex + BEGINNING_OF_CENTRAL].sigma_e_zz;
		++pdata;
		};
		Graph[1].DrawSurface("sigma : vez = vez0 + sigma Ez",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
		false, // no inner mesh display.
		GRAPH_RESPONSE, &X);

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
		pdata->temp.x = q_*X.pData[iVertex + BEGINNING_OF_CENTRAL].n*
		(p_OhmsCoeffs_host[iVertex + BEGINNING_OF_CENTRAL].sigma_i_zz
		- p_OhmsCoeffs_host[iVertex + BEGINNING_OF_CENTRAL].sigma_e_zz);

		// Will show something not very useful ---- in a brief instant there
		// isn't much time for second-order (frictional) effects.
		++pdata;
		};
		Graph[2].DrawSurface("Ez=0 v addition: vez0-vez",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
		false, // no inner mesh display.
		GRAPH_DECEL, &X);
		// Too bad substep is not stated. We should divide by substep to give anything meaningful
		// in these graphs.

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
		pdata->temp.x = q_*X.pData[iVertex + BEGINNING_OF_CENTRAL].n*
		(p_OhmsCoeffs_host[iVertex + BEGINNING_OF_CENTRAL].sigma_i_zz
		- p_OhmsCoeffs_host[iVertex + BEGINNING_OF_CENTRAL].sigma_e_zz);

		// Will show something not very useful ---- in a brief instant there
		// isn't much time for second-order (frictional) effects.
		++pdata;
		};
		Graph[3].DrawSurface("dynamic conductivity q n sigma : vez = vez0 + sigma Ez",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
		false, // no inner mesh display.
		GRAPH_DYNCONDUCTIVITY, &X);

		// create graph data for Ez : add Ez_strength*Ezshape to -Azdot/c
		overc = 1.0 / c_;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
		X.pData[iVertex + BEGINNING_OF_CENTRAL].temp.y =
		-X.pData[iVertex + BEGINNING_OF_CENTRAL].Azdot*overc
		+ GetEzShape__(X.pData[iVertex + BEGINNING_OF_CENTRAL].pos.modulus())*EzStrength_;
		}
		Graph[4].DrawSurface("Ez",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.y)),
		AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)), // use Jz's colour
		false,
		GRAPH_EZ, &X);

		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
		pdata->temp.x = q_ * pdata->n*(pdata->viz - pdata->vez);
		++pdata;
		};
		Graph[5].DrawSurface("Jz",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
		false, // no inner mesh display.
		GRAPH_JZ, &X);

		break;*/
	case LAPAZ_AZ:
		
		printf("\n\nRefreshGraphs: LAPAZ_AZ\n\n");
		// Assume temp.x contains Lap Az
		Graph[0].DrawSurface("Lap Az",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			true, GRAPH_LAPAZ, &X);
		Graph[1].DrawSurface("Az",
			DATA_HEIGHT, (real *)(&(X.pData[0].Az)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].Az)),
			true, GRAPH_AZ, &X);
		Graph[2].DrawSurface("Azdot",
			DATA_HEIGHT, (real *)(&(X.pData[0].Azdot)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].Azdot)),
			true, GRAPH_AZDOT, &X);
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.y = q_ * pdata->n*(pdata->viz - pdata->vez);
			++pdata;
		};
		Graph[3].DrawSurface("Jz",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.y)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.y)),
			false, GRAPH_JZ, &X);

		break;
	case EXYCOMPONENTS:
		/*
		X.Setup_J(); // the others can already exist.

		Graph[0].DrawSurface("Adotxy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].Adot.x)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].Adot.x)),
		true,
		GRAPH_ADOTXY, &X);
		Graph[1].DrawSurface("Grad phi",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].GradTe)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].GradTe)),
		true, // no inner mesh display: ??
		GRAPH_GRADPHI, &X);
		Graph[2].DrawSurface("Exy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].E)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].E)),
		true,
		GRAPH_EXY, &X);
		Graph[3].DrawSurface("Jxy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].Temp.x)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].Temp.x)),
		false, // no inner mesh display.
		GRAPH_JXY, &X);

		*/
		// Set GradTe to grad phi
		break;
	case JXYAXYBZEXY:
		/*
		X.Setup_J(); // the others can already exist.

		Graph[0].DrawSurface("Axy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].A.x)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].A.x)),
		true,
		GRAPH_AXY, &X);
		Graph[1].DrawSurface("Bz",
		DATA_HEIGHT, (real *)(&(X.X[0].B.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].B.z)),
		true, // no inner mesh display: ??
		GRAPH_BZ, &X);
		Graph[2].DrawSurface("Exy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].E)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].E)),
		true,
		GRAPH_EXY, &X);
		Graph[3].DrawSurface("Jxy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].Temp.x)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].Temp.x)),
		false, // no inner mesh display.
		GRAPH_JXY, &X);
		*/
		break;
	case EXY_RHO_PHI_PHIDOT:
		/*
		// For this one do n_i-n_e
		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		if (pVertex->flags == DOMAIN_VERTEX) {
		pVertex->temp2.x = (pVertex->Ion.mass - pVertex->Elec.mass) / pVertex->AreaCell;
		}
		else {
		pVertex->temp2.x = 0.0;
		};
		++pVertex;
		}

		Graph[0].DrawSurface("phi",
		DATA_HEIGHT, (real *)(&(X.X[0].phi)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].phi)),
		true,
		GRAPH_PHI, &X);
		Graph[1].DrawSurface("phidot",
		DATA_HEIGHT, (real *)(&(X.X[0].phidot)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].phidot)),
		true,
		GRAPH_PHIDOT, &X);
		Graph[2].DrawSurface("Exy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].E)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].E)),
		true,
		GRAPH_EXY, &X);
		Graph[3].DrawSurface("n_i-n_e",
		DATA_HEIGHT, (real *)(&(X.X[0].temp2.x)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].temp2.x)),
		false, // no inner mesh display.
		GRAPH_NINE, &X);
		*/
		break;
	case EXY_RHO_PHI_JXY:
		// create rho on pVertex->temp2.x ... 
		/*
		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		if (pVertex->flags == DOMAIN_VERTEX) {
		pVertex->temp2.x = q * (pVertex->Ion.mass - pVertex->Elec.mass) / pVertex->AreaCell;
		}
		else {
		pVertex->temp2.x = 0.0;
		};
		++pVertex;
		}

		X.Setup_J();

		Graph[0].DrawSurface("phi",
		DATA_HEIGHT, (real *)(&(X.X[0].phi)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].phi)),
		false,
		GRAPH_PHI, &X);
		Graph[1].DrawSurface("Jxy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].Temp)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].Temp)),
		false, // no inner mesh display: ??
		GRAPH_JXY, &X);
		Graph[2].DrawSurface("Exy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].E)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].E)),
		false,
		GRAPH_EXY, &X);
		Graph[3].DrawSurface("rho",
		DATA_HEIGHT, (real *)(&(X.X[0].temp2.x)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].temp2.x)),
		false, // no inner mesh display.
		GRAPH_RHO, &X);
		*/
		break;

	case EXY_RHO_BZ_JXY:
		/*
		// create rho on pVertex->temp2.x ...
		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		if (pVertex->flags == DOMAIN_VERTEX) {
		pVertex->temp2.x = q * (pVertex->Ion.mass - pVertex->Elec.mass) / pVertex->AreaCell;
		}
		else {
		pVertex->temp2.x = 0.0;
		};
		++pVertex;
		}

		X.Setup_J();

		Graph[0].DrawSurface("Bz",
		DATA_HEIGHT, (real *)(&(X.X[0].B.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].B.z)),
		true, // no inner mesh display: ??
		GRAPH_BZ, &X);
		Graph[1].DrawSurface("Jxy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].Temp)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].Temp)),
		false, // no inner mesh display: ??
		GRAPH_JXY, &X);
		Graph[2].DrawSurface("Exy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].E)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].E)),
		false,
		GRAPH_EXY, &X);
		Graph[3].DrawSurface("rho",
		DATA_HEIGHT, (real *)(&(X.X[0].temp2.x)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].temp2.x)),
		false, // no inner mesh display.
		GRAPH_RHO, &X);
		*/
		break;

	//case SIGMA_E_J:
		/*
		X.Setup_J(); // the others can already exist.

		Graph[0].DrawSurface("sigma_e_zz",
		DATA_HEIGHT, (real *)(&(X.X[0].sigma_e.zz)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].sigma_e.zz)),
		true,
		GRAPH_SIGMA_E, &X);
		//Graph[1].DrawSurface("v_e_0.z",
		//	DATA_HEIGHT,(real *)(&(X.X[0].v_e_0.z)),
		//	AZSEGUE_COLOUR,(real *)(&(X.X[0].v_e_0.z)),
		//false, // no inner mesh display: ??
		// GRAPH_VE0Z, &X);
		Graph[1].DrawSurface("nsigma",
		DATA_HEIGHT, (real *)(&(X.X[0].xdotdot.x)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].xdotdot.x)),
		true, GRAPH_SIGMATEMP, &X);
		Graph[2].DrawSurface("Ez",
		DATA_HEIGHT, (real *)(&(X.X[0].E.z)),
		FLAG_AZSEGUE_COLOUR, (real *)(&(X.X[0].E.z)), // how to make SEGUE_COLOUR work?
		false, // ??
		GRAPH_EZ, &X);
		Graph[3].DrawSurface("Jz",
		DATA_HEIGHT, (real *)(&(X.X[0].Temp.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].Temp.z)),
		false, // no inner mesh display.
		GRAPH_JZ, &X);
		*/
	//	break;


	case TOTAL:
		
		// In this case we have to create data,
		// as we go.
		
		// Best put it here so we can see where
		// data is being populated.

		/*long iVertex;
		Vertex * pVertex = X;
		for (iVertex = 0; iVertex < numVertices; iVertex++)
		{
		if ((pVertex->flags == DOMAIN_VERTEX) || (pVertex->flags == OUTERMOST))
		{
		pVertex->n = (pVertex->Neut.mass + pVertex->Ion.mass) / pVertex->AreaCell;
		pVertex->v = (m_n*pVertex->Neut.mom + m_ion * pVertex->Ion.mom + m_e * pVertex->Elec.mom) /
		(m_n*pVertex->Neut.mass + m_ion * pVertex->Ion.mass + m_e * pVertex->Elec.mass);
		pVertex->T = (pVertex->Neut.heat + pVertex->Ion.heat + pVertex->Elec.heat) /
		(pVertex->Neut.mass + pVertex->Ion.mass + pVertex->Elec.mass);
		pVertex->Temp.x = pVertex->Ion.mass / (pVertex->Neut.mass + pVertex->Ion.mass);
		};
		++pVertex;
		}*/
		//X.CalculateTotalGraphingData();

		printf("\n\nRefreshGraphs: TOTAL\n\n");
		// ought to change this to use variables n,v,T !
		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if ((pVertex->flags == DOMAIN_VERTEX) || (pVertex->flags == OUTERMOST))
			{
				pdata->temp.x = pdata->n + pdata->n_n;
				pdata->temp.y = pdata->n / pdata->temp.x;
			}
			++pVertex;
			++pdata;
		}
		Graph[0].DrawSurface("n_n + n_ion",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		IONISE_COLOUR, (real *)(&(X.pData[0].temp.y)),
		false,
		GRAPH_TOTAL_N, &X);


		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if ((pVertex->flags == DOMAIN_VERTEX) || (pVertex->flags == OUTERMOST))
			{
				pdata->temp.x = (m_neutral_*pdata->n_n*pdata->v_n.x
					+ (m_ion_ + m_e_) * pdata->n*pdata->vxy.x) /
					(m_neutral_*pdata->n_n + (m_ion_ + m_e_)*pdata->n);
				pdata->temp.y = (m_neutral_*pdata->n_n*pdata->v_n.y
					+ (m_ion_ + m_e_) * pdata->n*pdata->vxy.y) /
					(m_neutral_*pdata->n_n + (m_ion_ + m_e_)*pdata->n);
			}
			++pVertex;
			++pdata;
		}
		Graph[1].DrawSurface("sum[n_s v_s m_s]/sum[n_s m_s]",
		VELOCITY_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		VELOCITY_COLOUR, (real *)(&(X.pData[0].temp.x)),
		false, // no inner mesh display
		GRAPH_TOTAL_V, &X);
		
		
		//Graph[2].DrawSurface("n_n+n_ion",
		//DATA_HEIGHT, (real *)(&(X.X[0].n)),
		//VELOCITY_COLOUR, (real *)(&(X.X[0].v)),
		//false,
		//GRAPH_TOTAL_N_II, &X);   // ok what we did here? we thought we'd colour with velocity .. but we haven't given ourselves room for 3 temp vars so drop this for now.
		
		
		pVertex = X.X;
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			if ((pVertex->flags == DOMAIN_VERTEX) || (pVertex->flags == OUTERMOST))
			{
				pdata->temp.x = (pdata->n_n*pdata->Tn
							+  pdata->n*(pdata->Ti + pdata->Te)) /
								(pdata->n_n + pdata->n + pdata->n);
			}
			++pVertex;
			++pdata;
		}
		Graph[3].TickRescaling = 1.0 / kB_;
		Graph[3].DrawSurface("sum[n_s T_s]/sum[n_s]",
		DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
		SEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
		false,
		GRAPH_TOTAL_T, &X);
		Graph[3].TickRescaling = 1.0;
		break;
		
	};

	// Graph 2 and 4, in case of species graphs:

	switch (iGraphsFlag) {
	//case SPECIES_NEUTRAL:
	case SPECIES_ION:
	//case SPECIES_ELEC:
	//case TOTAL:

		int offset_v, offset_T;
		offset_v = (real *)(&(X.pData[0].vxy)) - (real *)(&(X.pData[0]));
		offset_T = (real *)(&(X.pData[0].Te)) - (real *)(&(X.pData[0]));

		Graph[2].SetEyePlan(GlobalPlanEye);
		Graph[2].boolDisplayMeshWireframe = true;
		Graph[2].boolClearZBufferBeforeWireframe = true;
		Graph[2].boolDisplayMainMesh = true;
		Graph[2].boolDisplayInnerMesh = false;
		Graph[2].boolDisplayScales = false;

		if (GlobalColoursPlanView == 0)
		{
			// nothing
			Graph[2].mhTech = Graph[2].mFX->GetTechniqueByName("MeshTech");
			Graph[2].SetDataWithColour(X, FLAG_COLOUR_MESH, FLAG_FLAT_MESH, 0, 0,
				GRAPH_FLAT_WIRE_MESH);
			Graph[2].Render(buff, GlobalRenderLabels, &X);

		} else {
			// Tell SDWC not to mess with colourmax if it's a flat mesh.

			if (GlobalColoursPlanView == 1)
			{
				// velocity
				Graph[2].mhTech = Graph[2].mFX->GetTechniqueByName("VelociTech");
				Graph[2].colourmax = Graph[0].colourmax; // match colours

				Graph[2].SetDataWithColour(X, FLAG_VELOCITY_COLOUR, FLAG_FLAT_MESH, offset_v, offset_v,
					GRAPH_FLAT_WIRE_MESH);
				Graph[2].Render(buff, GlobalRenderLabels, &X);
			};
			////else {
			////	// temperature
			////	Graph[2].mhTech = Graph[2].mFX->GetTechniqueByName("SegueTech");
			////	// SegueVS should take maximum as a parameter;
			////	// at least for colours we should prefer an absolute scale for T
			////	// Is it ever used for anything else? Not so far? eps?

			////	Graph[2].SetDataWithColour(X, FLAG_SEGUE_COLOUR, FLAG_FLAT_MESH, offset_T, offset_T,
			////		GRAPH_FLAT_WIRE_MESH);
			////	Graph[2].Render(buff, GlobalRenderLabels, &X);
			////};
		};

		// =================================================================================

		printf("\ngot to here; graph [4]:\n\n");

		Graph[4].boolDisplayKeyButton = false; // it's temperature
		Graph[4].SetEyePlan(GlobalPlanEye);
		Graph[4].boolDisplayMeshWireframe = true;
		Graph[4].boolClearZBufferBeforeWireframe = true;
		Graph[4].boolDisplayMainMesh = true;
		Graph[4].boolDisplayInnerMesh = false;
		Graph[4].boolDisplayScales = false;

		Graph[4].mhTech = Graph[4].mFX->GetTechniqueByName("SegueTech");
		
		Graph[4].SetDataWithColour(X, FLAG_SEGUE_COLOUR, FLAG_FLAT_MESH, offset_T, offset_T,
					GRAPH_FLAT_WIRE_MESH);
		Graph[4].Render(buff, GlobalRenderLabels, &X);
				
		break;
	}
	printf("End of Refreshgraphs\n");

}




__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

HRESULT hresult;

int main()
{
   
	HINSTANCE hInstance = GetModuleHandle(NULL);
	HWND hwndConsole = GetConsoleWindow();
	char szInitialFilenameAvi[512];
	MSG msg;
	HDC hdc;
	//	HACCEL hAccelTable;
	real x, y, temp;
	int i, j;
	float a1, a2, a3, a4; 
	//HWND hwndConsole;
	FILE * fp;
	extern char Functionalfilename[1024];

	int nDevices, iWhich;
	hipDeviceProp_t prop;
	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Memory Clock Rate (KHz): %d\n",
			prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
			prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
			2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);

		if (prop.memoryBusWidth == 384) iWhich = i;
	}
	printf("Picked %d \n", iWhich);
	getch(); 

	hipSetDevice(iWhich); // K40?
	hipDeviceReset();

	size_t uFree, uTotal;
	hipMemGetInfo(&uFree, &uTotal);
	printf("Memory on device: uFree %d uTotal %d\n", uFree, uTotal);


	h = TIMESTEP;
	evaltime = 0.0; // gets updated before advance

	memset(Historic_powermax, 0, 200 * sizeof(int));
	memset(Historic_powermin, 0, 200 * sizeof(int));

	ZeroMemory(Historic_max, 512 * HISTORY * sizeof(float));
	ZeroMemory(Historic_min, 512 * HISTORY * sizeof(float));
	GlobalStepsCounter = 0; steps_remaining = 0; steps_remaining_CPU = 0;

	SetConsoleTitle("2D 1/16 annulus DPF simulation");
	Sleep(40);
	//hwndConsole = FindWindow(NULL, "2D 1/16 annulus DPF simulation");
	MoveWindow(hwndConsole, 0, 0, SCREEN_WIDTH - VIDEO_WIDTH - 10, SCREEN_HEIGHT - 30, TRUE);

	report_time(0);

	int filetag = 0;
	do { 
		filetag++;
		sprintf(Functionalfilename, FUNCTIONALFILE_START "%03d.txt", filetag);
	} while ((_access(Functionalfilename, 0)) != -1);
	printf("\n\nopening %s \n", Functionalfilename);
	fp = fopen(Functionalfilename, "w");
	if (fp == 0) {
		printf("error with %s \n", Functionalfilename);
		getch();
	}
	else {
		printf("opened %s \n", Functionalfilename);
	}; 
	fprintf(fp, "GSC evaltime Area neut.N ion.N elec.N neut.r ion.r elec.r SDneut.r SDion.r SDelec.r "
		" neut.vr neut.vth neut.vz  ion.vr ion.vth ion.vz elec.vr elec.vth elec.vz neut.heat ion.heat elec.heat neut.T ion.T elec.T "
		" neut.mnvv/3 ion.mnvv/3 elec.mnvv/3 elec.force(vxB)r within3.6 elec.Bth EE BB Heatings and dT changes - see code \n");
	fclose(fp);
	
	X1.Initialise(1); // Set evaltime first
	X2.Initialise(2);
	X3.Initialise(3);
	printf("Got to here 1\n");
	    
	{
		X4.Initialise(4);
		printf("Got to here 2\n");
		X4.CreateTilingAndResequence2(&X1);
		X4.CreateTilingAndResequence2(&X2);
		X4.CreateTilingAndResequence2(&X3);
		printf("Got to here 3\n");

		// 
		// Dropping it for now so we can pursue solving equations first.
		// 
	}
	X1.Recalculate_TriCentroids_VertexCellAreas_And_Centroids();
	X1.EnsureAnticlockwiseTriangleCornerSequences_SetupTriMinorNeighboursLists();
	X1.SetupMajorPBCTriArrays();
	X2.Recalculate_TriCentroids_VertexCellAreas_And_Centroids();
	X2.EnsureAnticlockwiseTriangleCornerSequences_SetupTriMinorNeighboursLists();
	X2.SetupMajorPBCTriArrays();
	X3.Recalculate_TriCentroids_VertexCellAreas_And_Centroids();
	X3.EnsureAnticlockwiseTriangleCornerSequences_SetupTriMinorNeighboursLists();
	X3.SetupMajorPBCTriArrays();
	printf("Got to here 4\n");
	X1.InitialPopulate();
	X2.InitialPopulate();
	X3.InitialPopulate();

	X1.Create4Volleys();
	X2.Create4Volleys();
	X3.Create4Volleys();

	pTriMesh = &X1;

	pX = &X1;
	pXnew = &X2;
	
	GlobalBothSystemsInUse = 0;

	printf(report_time(1));
	printf("\n");
	report_time(0);
	
	// Window setup
	LoadString(hInstance, IDS_APP_TITLE, szTitle, 1024);
	LoadString(hInstance, IDC_F2DVALS, szWindowClass, 1024);
	wcex.cbSize = sizeof(WNDCLASSEX);
	wcex.style = CS_HREDRAW | CS_VREDRAW;
	wcex.lpfnWndProc = WndProc;
	wcex.cbClsExtra = 0;
	wcex.cbWndExtra = 0;
	wcex.hInstance = hInstance;
	wcex.hIcon = LoadIcon(hInstance, MAKEINTRESOURCE(IDI_F2DVALS));
	wcex.hCursor = LoadCursor(NULL, IDC_ARROW);
	wcex.hbrBackground = (HBRUSH)(COLOR_WINDOW + 1);
	wcex.lpszMenuName = MAKEINTRESOURCE(IDR_MENU1);
	wcex.lpszClassName = szWindowClass;
	wcex.hIconSm = LoadIcon(wcex.hInstance, MAKEINTRESOURCE(IDI_SMALL));
	if (RegisterClassEx(&wcex) == 0) {
		char buff[128];
		MessageBox(NULL, "RegisterClassEx failed", itoa(GetLastError(), buff, 10), MB_OK);
	};

	printf("SCREEN_WIDTH %d VIDEO_WIDTH %d VIDEO_HEIGHT %d \n",
		SCREEN_WIDTH, VIDEO_WIDTH, VIDEO_HEIGHT);

	hWnd = CreateWindowEx(NULL, szWindowClass, szTitle, WS_BORDER | WS_POPUP,
		SCREEN_WIDTH - VIDEO_WIDTH - 5, 0, VIDEO_WIDTH + 5, VIDEO_HEIGHT + 20, NULL, NULL, hInstance, NULL);
	if (!hWnd) {
		DWORD dword = GetLastError();
		char buff[128];
		MessageBox(NULL, "CreateWindowEx failed", itoa(dword, buff, 10), MB_OK);
		return dword;
	}
	// This is sending a message to WndProc before any of the following happens.

	ShowWindow(hWnd, SW_SHOWNORMAL);
	UpdateWindow(hWnd);

	xzscale = 2.0 / 0.1; // very zoomed in. Now what?

	DXChk(Direct3D.Initialise(hWnd, hInstance, VIDEO_WIDTH, VIDEO_HEIGHT));

	// With Field Of View = PI/4 used this:
	/*
	GlobalEye.x = 0.0f;
	GlobalEye.y = 12.4f;  //7.2f;
	GlobalEye.z = -18.0f + 2.5*xzscale;//DEVICE_RADIUS_INSULATOR_OUTER*xzscale;//-17.8f+

	GlobalLookat.x = 0.4f;
	GlobalLookat.y = 3.0f;
	GlobalLookat.z = DEVICE_RADIUS_INITIAL_FILAMENT_CENTRE * xzscale;

	GlobalPlanEye.x = 0.0f;
	GlobalPlanEye.y = 35.0f;
	GlobalPlanEye.z = (3.44 + 4.1)*0.5*xzscale;

	GlobalPlanEye2.x = -0.1f;
	GlobalPlanEye2.y = 19.5f;
	GlobalPlanEye2.z = 2.8*xzscale;

	GlobalPlanLookat.x = GlobalPlanEye.x;
	GlobalPlanLookat.y = 0.0f;
	GlobalPlanLookat.z = GlobalPlanEye.z + 0.0001;

	GlobalPlanLookat2.x = GlobalPlanEye2.x;
	GlobalPlanLookat2.y = 0.0f;
	GlobalPlanLookat2.z = GlobalPlanEye2.z + 0.0001;*/

	GlobalEye.x = -10.4f;
	GlobalEye.y = 16.4f;  //7.2f;
	GlobalEye.z = 44.0f;

	GlobalLookat.x = 1.20f;
	GlobalLookat.y = 3.0f;
	GlobalLookat.z = 72.2f;

	GlobalPlanEye.x = 2.9f;
	GlobalPlanEye.y = 17.97f;
	GlobalPlanEye.z = 71.95f;

	GlobalPlanEye2.x = -0.1f;
	GlobalPlanEye2.y = 19.5f;
	GlobalPlanEye2.z = 2.8*xzscale;
	 
	GlobalPlanLookat.x = GlobalPlanEye.x;
	GlobalPlanLookat.y = 0.0f;
	GlobalPlanLookat.z = GlobalPlanEye.z + 0.0001;

	GlobalPlanLookat2.x = GlobalPlanEye2.x;
	GlobalPlanLookat2.y = 0.0f;
	GlobalPlanLookat2.z = GlobalPlanEye2.z + 0.0001;
	
	newEye.x = 0.0f;
	newEye.y = 0.1f;
	newEye.z = 40.0f;
	newLookat.x = 0.0f;
	newLookat.y = 0.0f;
	newLookat.z = 72.0f;

						 // Add vectors in parallel.
	hipError_t cudaStatus;

	if (DXChk(Graph[0].InitialiseWithoutBuffers(0, 0, GRAPH_WIDTH, GRAPH_HEIGHT, GlobalEye, GlobalLookat)) +
		DXChk(Graph[0].InitialiseBuffers(X1))
		)
	{
		PostQuitMessage(200);
	};
	if (DXChk(Graph[1].InitialiseWithoutBuffers(0, GRAPH_HEIGHT, GRAPH_WIDTH, GRAPH_HEIGHT, GlobalEye, GlobalLookat)) +
		DXChk(Graph[1].InitialiseBuffers(X1))
		)
	{
		PostQuitMessage(201);
	};
	if (DXChk(Graph[2].InitialiseWithoutBuffers(GRAPH_WIDTH, 0, GRAPH_WIDTH, GRAPH_HEIGHT, GlobalPlanEye, GlobalPlanLookat)) +
		DXChk(Graph[2].InitialiseBuffers(X1))
		)
	{
		PostQuitMessage(202);
	};
	if (DXChk(Graph[3].InitialiseWithoutBuffers(GRAPH_WIDTH, GRAPH_HEIGHT, GRAPH_WIDTH, GRAPH_HEIGHT, GlobalEye, GlobalLookat)) +
		DXChk(Graph[3].InitialiseBuffers(X1))
		)
	{
		PostQuitMessage(203);
	};
	   
	if (NUMGRAPHS > 4) {

		if (DXChk(Graph[4].InitialiseWithoutBuffers(GRAPH_WIDTH * 2, 0, GRAPH_WIDTH, GRAPH_HEIGHT, GlobalPlanEye, GlobalPlanLookat)) +
			DXChk(Graph[4].InitialiseBuffers(X1))
			)
		{
			PostQuitMessage(204);
		};

		if (DXChk(Graph[5].InitialiseWithoutBuffers(GRAPH_WIDTH * 2, GRAPH_HEIGHT, GRAPH_WIDTH, GRAPH_HEIGHT, GlobalEye, GlobalLookat)) +
			DXChk(Graph[5].InitialiseBuffers(X1))
			)
		{
			PostQuitMessage(204);
		};
		if (DXChk(Graph[6].InitialiseWithoutBuffers(0, 0, GRAPH_WIDTH*2, GRAPH_HEIGHT, newEye, GlobalLookat, true)) +
			DXChk(Graph[6].InitialiseBuffers(X1))
			)
		{
			PostQuitMessage(204);
		};
	};

	Graph[0].bDisplayTimestamp = false;
	Graph[1].bDisplayTimestamp = false;
	Graph[2].bDisplayTimestamp = false;
	Graph[3].bDisplayTimestamp = false;
	Graph[4].bDisplayTimestamp = true;
	Graph[5].bDisplayTimestamp = false;
	Graph[6].bDisplayTimestamp = true;

	Direct3D.pd3dDevice->GetBackBuffer(0, 0, D3DBACKBUFFER_TYPE_MONO, &p_backbuffer_surface);

	if (DXChk(p_backbuffer_surface->GetDC(&surfdc), 1000))
		MessageBox(NULL, "GetDC failed", "oh dear", MB_OK);

	surfbit = CreateCompatibleBitmap(surfdc, VIDEO_WIDTH, VIDEO_HEIGHT); // EXTRAHEIGHT = 90
	SelectObject(surfdc, surfbit);
	dibdc = CreateCompatibleDC(surfdc);

	long VideoWidth = VIDEO_WIDTH;
	long VideoHeight = VIDEO_HEIGHT;

	// pasted here just to set up format:
	bitmapinfo.bmiHeader.biSize = sizeof(BITMAPINFO);
	bitmapinfo.bmiHeader.biWidth = VideoWidth;
	bitmapinfo.bmiHeader.biHeight = VideoHeight;
	bitmapinfo.bmiHeader.biPlanes = 1;
	bitmapinfo.bmiHeader.biBitCount = 24;
	bitmapinfo.bmiHeader.biCompression = BI_RGB; // uncompressed  
	bitmapinfo.bmiHeader.biSizeImage = bitmapinfo.bmiHeader.biHeight;
	bitmapinfo.bmiHeader.biXPelsPerMeter = 3000;
	bitmapinfo.bmiHeader.biYPelsPerMeter = 3000;
	bitmapinfo.bmiHeader.biClrUsed = 0;
	bitmapinfo.bmiHeader.biClrImportant = 0;
	bitmapinfo.bmiColors->rgbBlue = 0;
	bitmapinfo.bmiColors->rgbRed = 0;
	bitmapinfo.bmiColors->rgbGreen = 0;
	bitmapinfo.bmiColors->rgbReserved = 0;
	// dimension DIB and set up pointer to bits
	dib = CreateDIBSection(dibdc, &bitmapinfo, DIB_RGB_COLORS, &lpvBits, NULL, 0);
	SelectObject(dibdc, dib);

	BitBlt(dibdc, 0, 0, VIDEO_WIDTH, VIDEO_HEIGHT, surfdc, 0, 0, SRCCOPY);
	
	for (i = 0; i < NUMAVI; i++)
	{
		sprintf(szInitialFilenameAvi, "%s%s_%s", FOLDER, szAvi[i], INITIALAVI);
		hAvi[i] = CreateAvi(szInitialFilenameAvi, AVIFRAMEPERIOD, NULL);
		if (hAvi[i] == 0) {
			printf("Failed to create avi file %d", i);
			getch(); getch(); getch();
		}
	};

	// 1000/25 = 40
	ZeroMemory(&opts, sizeof(opts));
	opts.fccHandler = mmioFOURCC('D', 'I', 'B', ' ');//('d','i','v','x');
	opts.dwFlags = 8;

	for (i = 0; i < NUMAVI; i++)
	{
		hresult = SetAviVideoCompression(hAvi[i], dib, &opts, false, hWnd); // always run this for every avi file but can
																  // call with false as long as we know opts contains valid information. 
		if (hresult != 0) {
			printf("error: i = %d, hresult = %d", i, (long)hresult);
			getch(); getch(); getch();
		};
	};

	counter = 0;
	//ReleaseDC(hWnd,surfdc);
	p_backbuffer_surface->ReleaseDC(surfdc);
	GlobalCutaway = true; // dies if true
	
	RefreshGraphs(*pX, GlobalSpeciesToGraph);
	
	Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);


	// Main message loop:
	memset(&msg, 0, sizeof(MSG));
	while (msg.message != WM_QUIT)
	{
		if (PeekMessage(&msg, NULL, 0U, 0U, PM_REMOVE))
		{
			TranslateMessage(&msg);
			DispatchMessage(&msg);
		} else {
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);
		};
	};

	UnregisterClass(szWindowClass, wcex.hInstance);
	
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

/* Auxiliary routine: printing a matrix */
void print_matrix(char* desc, lapack_int m, lapack_int n, double* a, lapack_int lda) {
	lapack_int i, j;
	printf("\n %s\n", desc);
	for (i = 0; i < m; i++) {
		for (j = 0; j < n; j++) printf(" %2.5E", a[i*lda + j]);
		printf("\n");
	}
}
 
/* Auxiliary routine: printing a vector of integers */
void print_int_vector(char* desc, lapack_int n, lapack_int* a) {
	lapack_int j;
	printf("\n %s\n", desc);
	for (j = 0; j < n; j++) printf(" %6i", a[j]);
	printf("\n");
}

LRESULT CALLBACK WndProc(HWND hWnd, UINT message, WPARAM wParam, LPARAM lParam)
{

	f64 lowest_vez;
	long iLow, iMinor;
	Triangle * pTri;
	Vertex * pVertex;
	
	long izTri[128];

	static bool bInvoked_cuSyst = false;
	static long GSCCPU = 0;
	int iAntiskips;
	int wmId, wmEvent;
	int i, j, ctr;
	PAINTSTRUCT ps;
	HDC hdc;
	real time_back_for_Adot;
	FILE * file, *fp;
	int maxeerr, count, iMin;
	char buf1000[1024];
	int attempts;
	real store_h;
	char ch, o;
	int failed;
	RECT rect;
	real TotalArea, TotalCharge;
	long iVertex;
	real mass_avg, mass_SD, mass_min, mass_max;
	OPENFILENAME ofn;       // common dialog box structure
	char szFile[260];       // buffer for file name
	char szFilter[1000]; // buffer for file filter
	char szfilter[256];
	char buffer[256];

	TriMesh * temp;

	static const real XCENTRE2 = DEVICE_RADIUS_INITIAL_FILAMENT_CENTRE * sin(PI / 32.0);
	static const real XCENTRE1 = -XCENTRE2;
	static const real YCENTRE = DEVICE_RADIUS_INITIAL_FILAMENT_CENTRE * cos(PI / 32.0);
	 
	switch (message)
	{
	case WM_CREATE: 

		// Don't ever try doing initialisation here;
		// That should be done manually from the menus.		

		break;
		 
	case WM_COMMAND:
		wmId = LOWORD(wParam);
		wmEvent = HIWORD(wParam);

		printf("\nWM_COMMAND: wmId %d\n\n", wmId);
	
		// Ensure that display menu items are consecutive IDs.
		// Parse the menu selections:
		switch (wmId)
		{
			
		case ID_DISPLAY_ONE_D:
			// printf("\a\n");
			// Don't know why resource.h is not working;
			// Maybe some #define overwrites it with 40024.
			//wmId += 50007 - 40024;
			GlobalSpeciesToGraph = ONE_D;
			printf("\nGlobalSpeciesToGraph = %d \n", GlobalSpeciesToGraph);
			RefreshGraphs(*pX, GlobalSpeciesToGraph);
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);
			break;
		// int const GraphFlags[NUMAVI] = { SPECIES_ION, OVERALL, JZAZBXYEZ, OHMSLAW, ONE_D, IONIZEGRAPH };
		case ID_DISPLAY_ION:
			GlobalSpeciesToGraph = SPECIES_ION;
			printf("\nGlobalSpeciesToGraph = %d \n", GlobalSpeciesToGraph);
			RefreshGraphs(*pX, GlobalSpeciesToGraph);
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);
			break;

		case ID_DISPLAY_TOTAL:
			GlobalSpeciesToGraph = OVERALL;
			printf("\nGlobalSpeciesToGraph = %d \n", GlobalSpeciesToGraph);
			RefreshGraphs(*pX, GlobalSpeciesToGraph);
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);
			break;

		case ID_DISPLAY_JZAZBXYEZ:
			GlobalSpeciesToGraph = JZAZBXYEZ;
			printf("\nGlobalSpeciesToGraph = %d \n", GlobalSpeciesToGraph);
			RefreshGraphs(*pX, GlobalSpeciesToGraph);
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);
			break;
		case ID_DISPLAY_IONIZEGRAPHS:
			GlobalSpeciesToGraph = IONIZEGRAPH;
			printf("\nGlobalSpeciesToGraph = %d \n", GlobalSpeciesToGraph);
			RefreshGraphs(*pX, GlobalSpeciesToGraph);
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);
			break;
		case ID_DISPLAY_OHMS:
			GlobalSpeciesToGraph = OHMSLAW;
			printf("\nGlobalSpeciesToGraph = %d \n", GlobalSpeciesToGraph);
			RefreshGraphs(*pX, GlobalSpeciesToGraph);
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);
			break;
		case ID_DISPLAY_DTGRAPH:

			GlobalSpeciesToGraph = DTGRAPH;
			printf("\nGlobalSpeciesToGraph = %d \n", GlobalSpeciesToGraph);
			RefreshGraphs(*pX, GlobalSpeciesToGraph);
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);

			break;
		case ID_DISPLAY_ACCELGRAPH:

			GlobalSpeciesToGraph = ACCELGRAPHS;
			printf("\nGlobalSpeciesToGraph = %d \n", GlobalSpeciesToGraph);
			RefreshGraphs(*pX, GlobalSpeciesToGraph);
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);

			break;

		case ID_DISPLAY_SIGMAEJ:

			i = wmId - ID_DISPLAY_NEUT;
			GlobalSpeciesToGraph = i;
			printf("\nGlobalSpeciesToGraph = %d \n", GlobalSpeciesToGraph);
			RefreshGraphs(*pX, GlobalSpeciesToGraph);
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);
			break;

		case ID_HELP_ABOUT:
			DialogBox(hInst, MAKEINTRESOURCE(IDD_ABOUTBOX), hWnd, About);
			break;
		case ID_FILE_EXIT:
			DestroyWindow(hWnd);
			break;

		case ID_FILE_SAVECAMERA:
			ZeroMemory(&ofn, sizeof(ofn));
			ofn.lStructSize = sizeof(ofn);
			ofn.hwndOwner = hWnd;
			ofn.lpstrFile = szFile;
			// Set lpstrFile[0] to '\0' so that GetOpenFileName does not 
			// use the contents of szFile to initialize itself.
			ofn.lpstrFile[0] = '\0';
			ofn.nMaxFile = sizeof(szFile);
			memcpy(szfilter, "All\0*.*\0cam\0*.CAM\0\0", 19); // strcpy stops at first null !!
			ofn.lpstrFilter = szfilter; //"All\0*.*\0Dat\0*.DAT\0\0";	// summat weird about that example code
			ofn.nFilterIndex = 1;
			ofn.lpstrFileTitle = NULL;
			ofn.nMaxFileTitle = 0;
			ofn.lpstrInitialDir = NULL;
			ofn.Flags = OFN_OVERWRITEPROMPT;
			ofn.lpstrTitle = NULL;

			if (GetSaveFileName(&ofn) == TRUE)
			{
				printf("\nsaving camera...");
				fp = fopen(ofn.lpstrFile, "wt");
				if (fp == 0) {
					printf("save failed.\n");
				}
				else {
					fprintf(fp, "%f %f %f ", GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
					fprintf(fp, "%f %f %f ", GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);
					fprintf(fp, "%f %f %f ", GlobalEye.x, GlobalEye.y, GlobalEye.z);
					fprintf(fp, "%f %f %f ", GlobalPlanLookat.x, GlobalPlanLookat.y, GlobalPlanLookat.z);
					fclose(fp);
					printf("done\n");
				};
			}
			else {
				printf("there was an issue\n");
			};
			break;
		case ID_FILE_LOADCAMERA:
			// Initialize OPENFILENAME
			ZeroMemory(&ofn, sizeof(ofn));
			ofn.lStructSize = sizeof(ofn);
			ofn.hwndOwner = hWnd;
			ofn.lpstrFile = szFile;
			// Set lpstrFile[0] to '\0' so that GetOpenFileName does not 
			// use the contents of szFile to initialize itself.
			ofn.lpstrFile[0] = '\0';
			ofn.nMaxFile = sizeof(szFile);
			memcpy(szfilter, "All\0*.*\0*.cam\0*.Cam\0\0", 21); // strcpy stops at first null !!
			ofn.lpstrFilter = szfilter; //"All\0*.*\0*.Dat\0*.DAT\0\0";	// summat weird about that example code
			ofn.nFilterIndex = 1;
			ofn.lpstrFileTitle = NULL;
			ofn.nMaxFileTitle = 0;
			ofn.lpstrInitialDir = NULL;
			ofn.Flags = OFN_PATHMUSTEXIST | OFN_FILEMUSTEXIST;
			// Display the Open dialog box. 
			if (GetOpenFileName(&ofn) == TRUE)
			{
				printf("\nloading camera...");
				fp = fopen(ofn.lpstrFile, "rt");
				if (fp == 0) {
					printf("failed.\n");
				}
				else {
					rewind(fp);
					fscanf(fp, "%f %f %f ", &(GlobalPlanEye.x), &(GlobalPlanEye.y), &(GlobalPlanEye.z));
					fscanf(fp, "%f %f %f ", &(GlobalLookat.x), &(GlobalLookat.y), &(GlobalLookat.z));
					fscanf(fp, "%f %f %f ", &(GlobalEye.x), &(GlobalEye.y), &(GlobalEye.z));
					fscanf(fp, "%f %f %f ", &(GlobalPlanLookat.x), &(GlobalPlanLookat.y), &(GlobalPlanLookat.z));
					fclose(fp);
				};
				RefreshGraphs(*pX, GlobalSpeciesToGraph); // sends data to graphs AND renders them
				Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);

			}
			else {
				printf("file error camera\n");
			};
			break; 
		case ID_FILE_LOADGPU:

			// Initialize OPENFILENAME:
			ZeroMemory(&ofn, sizeof(ofn));
			ofn.lStructSize = sizeof(ofn);
			ofn.hwndOwner = hWnd;
			ofn.lpstrFile = szFile;
			//
			// Set lpstrFile[0] to '\0' so that GetOpenFileName does not 
			// use the contents of szFile to initialize itself.
			ofn.lpstrFile[0] = '\0';
			ofn.nMaxFile = sizeof(szFile);
			//strcpy(szFilter, "All\0*.*\0Dat\0*.DAT\0\0");
			memcpy(szfilter, "All\0*.*\0Dat\0*.DAT\0\0", 19); // strcpy stops at first null !!
			ofn.lpstrFilter = szfilter; //"All\0*.*\0Dat\0*.DAT\0\0";	// summat weird about that example code
			ofn.nFilterIndex = 1;
			ofn.lpstrFileTitle = NULL;
			ofn.nMaxFileTitle = 0;
			ofn.lpstrInitialDir = NULL;
			ofn.Flags = OFN_PATHMUSTEXIST | OFN_FILEMUSTEXIST;

			// Display the Open dialog box. 
			if (GetOpenFileName(&ofn) == TRUE)
			{
				if (bInvoked_cuSyst == false) {
					bInvoked_cuSyst = true;

					pX->EnsureAnticlockwiseTriangleCornerSequences_SetupTriMinorNeighboursLists();
					pX->Average_n_T_to_tris_and_calc_centroids_and_minorpos();

					pX->Create4Volleys(); // THIS SHOULD NOT ALWAYS BE HERE !!
					printf("Called Create4Volleys! This should be removed in favour of loaded iVolley.\n");

					cuSyst_host.InvokeHost();
					cuSyst_host.PopulateFromTriMesh(pX);
					cuSyst_host2.InvokeHost();
					cuSyst_host2.PopulateFromTriMesh(pX);
					// transfer information.

					PerformCUDA_Invoke_Populate(
						&cuSyst_host,
						NUMVERTICES,
						pX->InnermostFrillCentroidRadius,
						pX->OutermostFrillCentroidRadius,
						pX->numStartZCurrentTriangles,
						pX->numEndZCurrentTriangles);
				};

				cuSyst_host.Load(ofn.lpstrFile);
			};
			printf("Populate *pX\n");
			cuSyst_host.PopulateTriMesh(pX);
			printf("send to device\n");
			cuSyst_host.SendToDevice(cuSyst1);
			printf("done\n");

			// Debug: redelaun on load:
			pX->RefreshVertexNeighboursOfVerticesOrdered();
			pX->Redelaunerize(true, true);

			// This isn't actually helpful?

			// pX->RefreshVertexNeighboursOfVerticesOrdered();
			// pX->X[89450-BEGINNING_OF_CENTRAL].GetTriIndexArray(izTri);
//			printf("89450 : %d %d %d %d %d %d \n",
//				izTri[0], izTri[1], izTri[2], izTri[3], izTri[4], izTri[5]);
//
			pX->EnsureAnticlockwiseTriangleCornerSequences_SetupTriMinorNeighboursLists();			 
			//	pX->Average_n_T_to_tris_and_calc_centroids_and_minorpos(); // Obviates some of our flip calcs to replace tri n,T 
			// not sure if needed .. just for calc centroid .. they do soon get wiped out anyway.
			cuSyst_host.PopulateFromTriMesh(pX);
			cuSyst_host.SendToDevice(cuSyst1); // check this is right
			cuSyst2.CopyStructuralDetailsFrom(cuSyst1);
			cuSyst3.CopyStructuralDetailsFrom(cuSyst1);
				// Let's assume these always carry through during GPU runs.
				// It certainly does not work as it stands if you don't populate them all the same, put it that way!!
			printf("sent back re-delaunerized system\n");

			break;
		case ID_FILE_SAVEBINARY:
			// Initialize OPENFILENAME
			ZeroMemory(&ofn, sizeof(ofn));
			ofn.lStructSize = sizeof(ofn);
			ofn.hwndOwner = hWnd;
			ofn.lpstrFile = szFile;
			// Set lpstrFile[0] to '\0' so that GetOpenFileName does not 
			// use the contents of szFile to initialize itself.
			ofn.lpstrFile[0] = '\0';
			ofn.nMaxFile = sizeof(szFile);
			memcpy(szfilter, "All\0*.*\0*.dat\0*.Dat\0\0", 21); // strcpy stops at first null !!
			ofn.lpstrFilter = szfilter; //"All\0*.*\0Dat\0*.DAT\0\0";	// summat weird about that example code
			ofn.nFilterIndex = 1;
			ofn.lpstrFileTitle = NULL;
			ofn.nMaxFileTitle = 0;
			ofn.lpstrInitialDir = NULL;
			ofn.Flags = OFN_OVERWRITEPROMPT;
			ofn.lpstrTitle = NULL;
			// Display the Open dialog box. 
			if (GetSaveFileName(&ofn) == TRUE)
			{
				printf("\nsaving system...");
				pX->Save(ofn.lpstrFile);
				printf("done\n");
			}
			else {
				printf("there was an issue\n");
			};
			break;
		case ID_FILE_SAVETEXT:
			// Initialize OPENFILENAME
			ZeroMemory(&ofn, sizeof(ofn));
			ofn.lStructSize = sizeof(ofn);
			ofn.hwndOwner = hWnd;
			ofn.lpstrFile = szFile;
			//
			// Set lpstrFile[0] to '\0' so that GetOpenFileName does not 
			// use the contents of szFile to initialize itself.
			ofn.lpstrFile[0] = '\0';
			ofn.nMaxFile = sizeof(szFile);
			//strcpy(szFilter,"All\0*.*\0Text\0*.TXT\0");
			memcpy(szfilter, "All\0*.*\0Dat\0*.DAT\0\0", 19); // strcpy stops at first null !!
			ofn.lpstrFilter = szfilter; //"All\0*.*\0Dat\0*.DAT\0\0";	// summat weird about that example code
			ofn.nFilterIndex = 1;
			ofn.lpstrFileTitle = NULL;
			ofn.nMaxFileTitle = 0;
			ofn.lpstrInitialDir = NULL;
			ofn.Flags = OFN_OVERWRITEPROMPT;
			ofn.lpstrTitle = NULL;
			// Display the Open dialog box. 
			if (GetSaveFileName(&ofn) == TRUE)
			{
				printf("\nsaving system...");
				pX->SaveText(ofn.lpstrFile);
				printf("done\n");
			}
			else {
				printf("there was an issue\n");
			};
			break;
		case ID_FILE_LOAD:

			// Initialize OPENFILENAME:
			ZeroMemory(&ofn, sizeof(ofn));
			ofn.lStructSize = sizeof(ofn);
			ofn.hwndOwner = hWnd;
			ofn.lpstrFile = szFile;
			//
			// Set lpstrFile[0] to '\0' so that GetOpenFileName does not 
			// use the contents of szFile to initialize itself.
			ofn.lpstrFile[0] = '\0';
			ofn.nMaxFile = sizeof(szFile);
			//strcpy(szFilter, "All\0*.*\0Dat\0*.DAT\0\0");
			memcpy(szfilter, "All\0*.*\0Dat\0*.DAT\0\0", 19); // strcpy stops at first null !!
			ofn.lpstrFilter = szfilter; //"All\0*.*\0Dat\0*.DAT\0\0";	// summat weird about that example code
			ofn.nFilterIndex = 1;
			ofn.lpstrFileTitle = NULL;
			ofn.nMaxFileTitle = 0;
			ofn.lpstrInitialDir = NULL;
			ofn.Flags = OFN_PATHMUSTEXIST | OFN_FILEMUSTEXIST;

			// Display the Open dialog box. 
			if (GetOpenFileName(&ofn) == TRUE)
			{
				pX->Load(ofn.lpstrFile);
				printf("\ndoin nothing...");
			};
		break;

		case ID_RUN_SIMULATIONSTEPS:

			GlobalSwitchBox = 0;
			DialogBox(hInst, MAKEINTRESOURCE(IDD_DIALOG1), hWnd, SetupBox);
			// that will not return with steps_remaining unset.

			if (steps_remaining > 0)
				SetTimer(hWnd, 1, 1, NULL); // 1 millisecond delay

			break;

		case ID_RUN_SIMULATIONSTEPS_CPU:

			GlobalSwitchBox = 0;
			steps_remaining_CPU = 1;
			// that will not return with steps_remaining unset.

			if (steps_remaining_CPU > 0)
				SetTimer(hWnd, 2, 1, NULL); // 1 millisecond delay

			break;

		case ID_RUN_STOP:

			steps_remaining = 0;
			steps_remaining_CPU = 0;
			break;
		case ID_INITIALISE_IONISATIONSTEPS:
			break;

		default:
			return DefWindowProc(hWnd, message, wParam, lParam);
		}
		break;

	case WM_TIMER:
		
		KillTimer(hWnd, wParam);
		report_time(0);
		if (wParam == 1)
		{
			if (bInvoked_cuSyst == false) {
				bInvoked_cuSyst = true;

				pX->EnsureAnticlockwiseTriangleCornerSequences_SetupTriMinorNeighboursLists();
				pX->Average_n_T_to_tris_and_calc_centroids_and_minorpos();
//
//				printf("tri 340: %d %d %d \n%1.14E %1.14E \n%1.14E %1.14E \n%1.14E %1.14E\n",
//					pX->T[340].cornerptr[0] - pX->X, pX->T[340].cornerptr[1] - pX->X, pX->T[340].cornerptr[2] - pX->X,
//					pX->T[340].cornerptr[0]->pos.x, pX->T[340].cornerptr[0]->pos.y,
//					pX->T[340].cornerptr[1]->pos.x, pX->T[340].cornerptr[1]->pos.y,
//					pX->T[340].cornerptr[2]->pos.x, pX->T[340].cornerptr[2]->pos.y);
//				printf("tri 340 periodic %d \n", pX->T[340].periodic);
//				getch();

				cuSyst_host.InvokeHost();
				cuSyst_host.PopulateFromTriMesh(pX);
				cuSyst_host2.InvokeHost();
				cuSyst_host2.PopulateFromTriMesh(pX);

				//		cuSyst_host.Output("n0.txt");

				PerformCUDA_Invoke_Populate(
					&cuSyst_host,
					NUMVERTICES,
					pX->InnermostFrillCentroidRadius,
					pX->OutermostFrillCentroidRadius,
					pX->numStartZCurrentTriangles,
					pX->numEndZCurrentTriangles);
			}

			// Run 1 step:
			printf("evaltime %1.9E\n", evaltime);

			//	PerformCUDA_RunStepsAndReturnSystem_Debug(&cuSyst_host, &cuSyst_host2, pX, &X3, pXnew);

			PerformCUDA_RunStepsAndReturnSystem(&cuSyst_host);

			//	printf("Stamp GPU over CPU y/n:");
			//	do {
			//		o = getch();
			//	} while ((o != 'y') && (o != 'n'));
			//	printf("%c\n\n", o);
			//	if (o == 'y') 

				// Auto-save system:
			if (GlobalStepsCounter % DATA_SAVE_FREQUENCY == 0)
			{
				sprintf(szFile, "auto%d.dat", GlobalStepsCounter);
				// SAVE cuSyst:
				cuSyst_host.Save(szFile);
			}

			// even number of steps should lead us back to pX having it
			steps_remaining--;
			GlobalStepsCounter++;

			printf("Done steps: %d   ||   Remaining this run: %d\n\n", GlobalStepsCounter, steps_remaining);

			if ((GlobalStepsCounter % GRAPHICS_FREQUENCY == 0) ||
				(GlobalStepsCounter % REDELAUN_FREQUENCY == 0) ||
				(steps_remaining == 0))
			{
				cuSyst_host.PopulateTriMesh(pX); // vertex n is populated into the minor array available on CPU
				printf("pulled back to host\n");
			}
		}
		else {
			pX->Advance(pXnew, &X3);
			temp = pX;
			pX = pXnew;
			pXnew = temp;

			steps_remaining_CPU--;
			GSCCPU++;
			printf("Done steps CPU: %d   ||   Remaining this run: %d\n\n", GSCCPU, steps_remaining_CPU);
			  
			sprintf(buf1000, "autosaveCPU%d.dat", GSCCPU);
			pX->Save(buf1000);
			printf("saved as %s\n", buf1000);
		};
		printf("%s\n", report_time(1));
		  
		if (GlobalStepsCounter % GRAPHICS_FREQUENCY == 0)
		{
			// make video frames:
			for (i = 0; i < NUMAVI; i++)
			{
				printf("i = %d \n", i);
				RefreshGraphs(*pX, GraphFlags[i]); // sends data to graphs AND renders them
															   //	::PlanViewGraphs1(*pX);
				printf(".DISHMOPS.\n");
				Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);
				printf("got to here 7\n");

				if (DXChk(p_backbuffer_surface->GetDC(&surfdc), 100))
					MessageBox(NULL, "GetDC failed", "oh dear", MB_OK);
				//SelectObject(surfdc,surfbit);
				BitBlt(dibdc, 0, 0, VIDEO_WIDTH, VIDEO_HEIGHT, surfdc, 0, 0, SRCCOPY);
				p_backbuffer_surface->ReleaseDC(surfdc);

				printf("got to here 7a\n");

				if (hAvi[i] == NULL) {
					printf("nojxycu.cu agrees that hAvi[i] == NULL\n");
				} else {
					printf("hAvi[i] not NULL here\n");
					// Debug
				}

				// getting hAvi[i] == 0 for the last one.
				// But on debug? No such thing? Same.

				hresult = AddAviFrame(hAvi[i], dib);
				if (hresult != 0) printf("\n******************************************************* \n"
					"hresult = %d\n********************************************** \n", hresult);
				
				printf("got to here 7b\n");

			};
			// sprintf(szFile, "System_%d", GlobalStepsCounter);
			// pX->SaveText(szFile);
		};
		printf("got to here 8");
		
		if (GlobalStepsCounter % (AVI_FILE_PINCHOFF_FREQUENCY * GRAPHICS_FREQUENCY) == 0)
		{
			for (i = 0; i < NUMAVI; i++)
			{
				// now have to pinch out avi file and make a new one
				CloseAvi(hAvi[i]);
				sprintf(buf1000, "%s%s_%d.avi", FOLDER, szAvi[i], GlobalStepsCounter);
				hAvi[i] = CreateAvi(buf1000, AVIFRAMEPERIOD, NULL);
				hresult = SetAviVideoCompression(hAvi[i], dib, &opts, false, hWnd);
				if (hresult != 0) printf("\n******************************************************* \n"
					"SetAviVideoCompression: hresult = %d\n********************************************** \n", hresult);

			};
		};
		 
		RefreshGraphs(*pX,GlobalSpeciesToGraph); // sends data to graphs AND renders them
		Direct3D.pd3dDevice->Present( NULL, NULL, NULL, NULL );
		 
		printf("got to here 9\n");


		if (GlobalStepsCounter % REDELAUN_FREQUENCY == 0)
		{
			pX->RefreshVertexNeighboursOfVerticesOrdered();
			long iFlips = pX->Redelaunerize(true, true);
			// Send back to GPU:
			pX->EnsureAnticlockwiseTriangleCornerSequences_SetupTriMinorNeighboursLists();

			//	pX->Average_n_T_to_tris_and_calc_centroids_and_minorpos(); // Obviates some of our flip calcs to replace tri n,T 
			// not sure if needed .. just for calc centroid .. they do soon get wiped out anyway.
			
			cuSyst_host.PopulateFromTriMesh(pX);// 1. Does it update lists? --- some had to be updated on CPU first.

			// Seems to copy structural information as well as data. n is copied from n_minor on CPU.
			
			//cuSyst1.SendToHost(cuSyst_host2);
			//cuSyst_host.ReportDifferencesHost(cuSyst_host2);
			cuSyst_host.SendToDevice(cuSyst1); 
			cuSyst2.CopyStructuralDetailsFrom(cuSyst1);
			cuSyst3.CopyStructuralDetailsFrom(cuSyst1);
			// Let's assume these always carry through during GPU runs.
			// It certainly does not work as it stands if you don't populate them all the same, put it that way!!

			printf("sent back re-delaunerized system\n");

			// Now reset A values more carefully in the sent-back system:

			f64 LapAz, viz, vez, n, coeffself, Az;
			int iRepeat;

			if (iFlips > 0) {
				kernelGetLapCoeffs_and_min << <numTriTiles, threadsPerTileMinor >> >(
					cuSyst1.p_info,
					cuSyst1.p_izTri_vert,
					cuSyst1.p_izNeigh_TriMinor,
					cuSyst1.p_szPBCtri_vert,
					cuSyst1.p_szPBC_triminor,
					p_LapCoeffself,
					p_temp1, // collect min
					p_longtemp
					);
				Call(hipDeviceSynchronize(), "cudaTS GetLapCoeffs x");

				for (iRepeat = 0; iRepeat < 3; iRepeat++) {
					// 1. Calculate Lap Az and coeffself Lap Az; including at our few points.

					kernelPullAzFromSyst << <numTilesMinor, threadsPerTileMinor >> > (
						cuSyst1.p_AAdot,
						p_Az
						);
					Call(hipDeviceSynchronize(), "cudaTS PullAz");

					kernelGetLap_minor << <numTriTiles, threadsPerTileMinor >> > (
						cuSyst1.p_info, // populated position... not neigh_len apparently
						p_Az,
						cuSyst1.p_izTri_vert,
						cuSyst1.p_izNeigh_TriMinor,
						cuSyst1.p_szPBCtri_vert,
						cuSyst1.p_szPBC_triminor,
						p_LapAz,
						cuSyst1.p_AreaMinor // OUTPUT
						);
					Call(hipDeviceSynchronize(), "cudaTS GetLapMinor addaa2");
					
					// 2. For each of our points bring Lap Az, Jz and coeffself to CPU
					for (i = 0; i < BEGINNING_OF_CENTRAL; i++)
					{
						if (flaglist[i]) {
							hipMemcpy(&LapAz, &(p_LapAz[i]), sizeof(f64), hipMemcpyDeviceToHost);
							hipMemcpy(&coeffself, &(p_LapCoeffself[i]), sizeof(f64), hipMemcpyDeviceToHost);
							hipMemcpy(&viz, &(cuSyst1.p_vie[i].viz), sizeof(f64), hipMemcpyDeviceToHost);
							hipMemcpy(&vez, &(cuSyst1.p_vie[i].vez), sizeof(f64), hipMemcpyDeviceToHost);
							hipMemcpy(&n, &(cuSyst1.p_n_minor[i].n), sizeof(f64), hipMemcpyDeviceToHost);
							hipMemcpy(&Az, &(cuSyst1.p_AAdot[i].Az), sizeof(f64), hipMemcpyDeviceToHost);
							// 3. For each of our points, adjust Az per Jacobi:
							printf("%d Az %1.11E ", i, Az);
							Az += 0.7* (-FOUR_PI_Q_OVER_C_*n*(viz - vez) - LapAz) / coeffself; // correct signs?
							// Aiming Lap Az = - 4pi/c Jz.
							// Therefore adjust LapAz by (-4pi/c Jz - LapAz).
							// underrelaxation & repeat 3 times
							printf("newAz %1.11E\n", Az);

							hipMemcpy(&(cuSyst1.p_AAdot[i].Az), &Az, sizeof(f64), hipMemcpyHostToDevice);
						};
					};
				};
			};
			
		};
		
		if (steps_remaining > 0) {
			SetTimer(hWnd, 1, DELAY_MILLISECS, NULL);
			printf("Waiting %d milliseconds to allow user input.\n", DELAY_MILLISECS);
		};
		if (steps_remaining_CPU > 0) {
			SetTimer(hWnd, 2, DELAY_MILLISECS, NULL);
			printf("Waiting %d milliseconds to allow user input.\n", DELAY_MILLISECS);
		};

		/*
		if (wParam == 1) {
			sprintf(buf1000, "autosaveGPU%d.dat", GlobalStepsCounter);
		} else {
			sprintf(buf1000, "autosaveCPU%d.dat", GSCCPU);
		}
		pX->Save(buf1000);
		printf("saved as %s\n", buf1000);
		
		lowest_vez = 0.0;
		iLow = 0;
		pTri = pX->T;
		for (iMinor = 0; iMinor < BEGINNING_OF_CENTRAL; iMinor++)
		{
			if ((pTri->u8domain_flag == DOMAIN_TRIANGLE) && (pX->pData[iMinor].vez < lowest_vez)) {
				lowest_vez = pX->pData[iMinor].vez;
				iLow = iMinor;
			}
			++pTri;
		}
		printf("Tris: lowest_vez %1.14E iLow %d \n", lowest_vez, iLow);
		iLow = 0;
		lowest_vez = 0.0;
		pVertex = pX->X;
		for (; iMinor < NMINOR; iMinor++)
		{
			if ((pVertex->flags == DOMAIN_VERTEX) && (pX->pData[iMinor].vez < lowest_vez)) {
				lowest_vez = pX->pData[iMinor].vez;
				iLow = iMinor;
			}
			++pVertex;
		}
		printf("Vertices: lowest_vez %1.14E iLow %d \n\n", lowest_vez, iLow);


		printf("save ascii?");
		do {
			o = getch();
		} while ((o != 'y') && (o != 'n'));
		printf("%c\n", o);
		if (o == 'y') {
			sprintf(buf1000, "SaveGPUtext1_trackedAA");
			pX->SaveText(buf1000);
			printf("Ascii file saved %s.\n",buf1000);
		}
		*/

		printf("steps_remaining GPU: %d  CPU: %d\n",steps_remaining, steps_remaining_CPU);
		
		
		break;

	case WM_KEYDOWN:

		switch (wParam)
		{
		case 'W':
			GlobalEye.z += 1.0f;
			printf("GlobalEye %f %f %f  \n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z);
			break;
		case 'S':
			GlobalEye.z -= 1.0f;
			printf("GlobalEye %f %f %f  \n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z);
			break;
		case 'A':
			GlobalEye.x -= 0.8f;
			printf("GlobalEye %f %f %f  \n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z);
			break;
		case 'D':
			GlobalEye.x += 0.8f;
			printf("GlobalEye %f %f %f  \n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z);
			break;
		case 'E':
			GlobalEye.y += 0.8f;
			printf("GlobalEye %f %f %f  \n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z);
			break;
		case 'C':
			GlobalEye.y -= 0.8f;
			printf("GlobalEye %f %f %f  \n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z);
			break;

		case 'V':
			GlobalLookat.z -= 0.4f;
			printf("GlobalEye %f %f %f  GlobalLookat %f %f %f\n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z, GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);
			break;
		case 'R':
			GlobalLookat.z += 0.4f;
			printf("GlobalEye %f %f %f  GlobalLookat %f %f %f\n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z, GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);
			break;
		case 'F':
			GlobalLookat.x -= 0.4f;
			printf("GlobalEye %f %f %f  GlobalLookat %f %f %f\n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z, GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);
			break;
	//	case 'G':
	//		GlobalLookat.x += 0.4f;
	//		printf("GlobalEye %f %f %f  GlobalLookat %f %f %f\n",
	//			GlobalEye.x, GlobalEye.y, GlobalEye.z, GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);
	//		break;
	//	case 'T':
	//		GlobalLookat.y += 0.4f;
	//		printf("GlobalLookat %f %f %f\n",
	//			GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);
	//		break;
	//	case 'B':
	//		GlobalLookat.y -= 0.4f;
	//		printf("GlobalLookat %f %f %f\n",
	//			GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);
	//		break;
		case '+':
			GlobalCutaway = !GlobalCutaway;
			break;
		case 'Y':
		case '<':
			GlobalEye.x = -10.4; GlobalEye.y = 16.4; GlobalEye.z = 44.0;
			GlobalLookat.x = -3.6; GlobalLookat.y = 3.0; GlobalLookat.z = 72.2;
			printf("GlobalEye %f %f %f  GlobalLookat %f %f %f\n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z, GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);

			GlobalPlanEye.x = 7.1; GlobalPlanEye.y = 11.5; GlobalPlanEye.z = 71.35;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);

			break;
		case '_':
		case '-':
		case '>':
			GlobalPlanEye.x = 7.0; GlobalPlanEye.y = 14.0; GlobalPlanEye.z = 71.0;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
			break;

		case 'U':
			GlobalPlanEye.z += 0.6f;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
			break;
		case 'J':
			GlobalPlanEye.z -= 0.6f;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
			break;
		case 'H':
			GlobalPlanEye.x -= 0.6f;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
			break;
		case 'K':
			GlobalPlanEye.x += 0.6f;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
			break;
		case 'I':
			GlobalPlanEye.y *= 1.25f;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
			break;
		case 'M':
			GlobalPlanEye.y *= 0.8f;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
			break;
		case 'N':
			GlobalboolDisplayMeshWireframe = !GlobalboolDisplayMeshWireframe;
			//Graph1.boolDisplayMeshWireframe = (!(Graph1.boolDisplayMeshWireframe));
			break;
		case '9':
			GlobalRenderLabels = false;
			break;
		case '5':
			GlobalRenderLabels = true;
			GlobalWhichLabels = 0;// iTri
			break;
		case '8':
			GlobalRenderLabels = true;
			GlobalWhichLabels = 1;//T
			break;
		case '7':
			GlobalRenderLabels = true;
			GlobalWhichLabels = 2;//v
			break;
		case '6':
			GlobalRenderLabels = true;
			GlobalWhichLabels = 3;	//n
			break;
		case '1':
			GlobalColoursPlanView = 1;//v
			break;
		case '4':
			GlobalColoursPlanView = 0;//nothing
			break;
		case '2':
			GlobalColoursPlanView = 2;//T
			break;
		case '0':
			steps_remaining = 0;
			break;



		case 'Q':
			newEye.z += 5.0f;
			printf("newEye.z %1.9E\n", newEye.z);
			break;
		case 'P':
			newEye.z -= 5.0f;
			printf("newEye.z %1.9E\n", newEye.z);
			break;

		case 'T':
			newEye.y += 5.0f;			
			printf("newEye.y %1.9E\n", newEye.y);
			break;
		case 'Z':
			newEye.y -= 5.0f;
			printf("newEye.y %1.9E\n", newEye.y);
			break;
		case 'B':
			newLookat.z -= 3.0f;
			printf("newLookat.z %1.9E\n", newLookat.z);
			break;
		case 'G':
			newLookat.z += 3.0f;
			printf("newLookat.z %1.9E\n", newLookat.z);
			break;

		default:
			return DefWindowProc(hWnd, message, wParam, lParam);

		};

		//PlanViewGraphs1(*pX);

		RefreshGraphs(*pX, GlobalSpeciesToGraph); // sends data to graphs AND renders them
		Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);

		break;
	case WM_PAINT:
		// Not sure, do we want to do this?
		//	RefreshGraphs(*pX,); // sends data to graphs AND renders them
		GetUpdateRect(hWnd, &rect, FALSE);
		if (Direct3D.pd3dDevice != NULL)
			Direct3D.pd3dDevice->Present(&rect, &rect, NULL, NULL);

		ValidateRect(hWnd, NULL);
		break;
	case WM_DESTROY:
		DeleteObject(dib);
		DeleteDC(dibdc);
		for (i = 0; i < NUMAVI; i++)
			CloseAvi(hAvi[i]);

		//  _controlfp_s(0, cw, _MCW_EM); // Line A
		PerformCUDA_Revoke();


		PostQuitMessage(0);
		break;
	default:
		return DefWindowProc(hWnd, message, wParam, lParam);
	}
	return 0;
}

// Message handler for about box.
INT_PTR CALLBACK About(HWND hDlg, UINT message, WPARAM wParam, LPARAM lParam)
{
	UNREFERENCED_PARAMETER(lParam);
	switch (message)
	{
	case WM_INITDIALOG:
		return (INT_PTR)TRUE;

	case WM_COMMAND:
		if (LOWORD(wParam) == IDOK || LOWORD(wParam) == IDCANCEL)
		{
			EndDialog(hDlg, LOWORD(wParam));
			return (INT_PTR)TRUE;
		}
		break;
	}
	return (INT_PTR)FALSE;
}


INT_PTR CALLBACK SetupBox(HWND hDlg, UINT message, WPARAM wParam, LPARAM lParam)
{
	UNREFERENCED_PARAMETER(lParam);
	char buffer[2048];
	char string[1024];
	real newh;

	switch (message)
	{
	case WM_INITDIALOG:
		sprintf(buffer, "New h? (present = %1.10E)", h);
		if (GlobalSwitchBox)
			SetDlgItemText(hDlg, IDC_STATIC, buffer);
		return (INT_PTR)TRUE;

	case WM_COMMAND:
		if (LOWORD(wParam) == IDOK)
		{
			// try to read data from edit control:
			GetDlgItemText(hDlg, IDC_EDIT1, buffer, 2048);
			if (GlobalSwitchBox == 0)
			{
				// 
				steps_remaining = atoi(buffer);
				if (steps_remaining >= 0)
				{
					EndDialog(hDlg, LOWORD(wParam));
				}
				else {
					MessageBox(NULL, "incorrect value", "Enter a nonnegative integer.", MB_OK);
				};
			}
			else {
				newh = atof(buffer);
				if (newh > 0.0)
				{
					EndDialog(hDlg, LOWORD(wParam));
					sprintf(string, "h = %1.10E\n", newh);
					h = newh;
					MessageBox(NULL, string, "New value of h", MB_OK);
				}
				else {
					MessageBox(NULL, "no good", "Negative h entered", MB_OK);
				};
			};
			return (INT_PTR)TRUE;
		}
		break;
	}
	return (INT_PTR)FALSE;
}

