#include "hip/hip_runtime.h"
#define real double
#define f64 double

#define HISTORY										4

#include "headers.h"
#include "hip/hip_runtime.h"
#include ""

#include <time.h>
#include <stdio.h>
#include <windows.h>
#include "resource.h"
#include "flags.h"
#include "FFxtubes.h"
//#include "cppconst.h"
#include "cuda_struct.h"
#include "constant.h"

#include "d3d.h"    
#include <d3dx9.h> 
#include <dxerr.h>

#include <commdlg.h>    // probably used by avi_utils

#include "surfacegraph_tri.h"

#include "avi_utils.cpp"     // for making .avi


//=======================================================
// Declarations of functions:

void RefreshGraphs(TriMesh & X, const int iGraphsFlag);
LRESULT CALLBACK	WndProc(HWND, UINT, WPARAM, LPARAM);
INT_PTR CALLBACK	About(HWND, UINT, WPARAM, LPARAM);
INT_PTR CALLBACK	SetupBox(HWND, UINT, WPARAM, LPARAM);
extern f64 GetEzShape__(f64 r);
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);



// Global variables:
// =================

extern D3D Direct3D;

float xzscale;

bool bCullNone = false;
bool bGlobalsave = false;
int GlobalSwitchBox = 0;
int iGlobalScratch;

real GlobalHeightScale;

int GlobalSpeciesToGraph = SPECIES_ELECTRON;
int GlobalWhichLabels = 0;
bool GlobalRenderLabels = false;
int GlobalColoursPlanView = 0;
bool GlobalBothSystemsInUse;

bool GlobalCutaway = true;

unsigned int cw; // control word for floating point hardware exception hiding

TriMesh * pX, *pXnew;
TriMesh X1, X2, X3;
cuSyst cuSyst_host, cuSyst_host2;

D3DXVECTOR3 GlobalEye, GlobalLookat, GlobalPlanEye, GlobalPlanEye2, GlobalPlanLookat,
GlobalPlanLookat2, GlobalEye2, GlobalLookat2;

IDirect3DSurface9* p_backbuffer_surface;

long steps_remaining, GlobalStepsCounter, steps_remaining_CPU;
real evaltime, h;

extern real GlobalIzElasticity;
FILE * massfile, *maxfile;

// Global Variables:
HINSTANCE hInst;   // current instance
				   // window vars:
HWND hWnd;
WNDCLASSEX wcex;
TCHAR szTitle[1024];					// The title bar text
TCHAR szWindowClass[1024];			// the main window class name

char Functionalfilename[1024];
int GlobalGraphSetting[7];
surfacegraph Graph[7]; // why was it 5? // 5th one can be whole thing.


float Historic_max[100][HISTORY]; // if max is falling, use historic maximum for graph.
float Historic_min[100][HISTORY];
int Historic_powermax[200];
int Historic_powermin[200]; // just store previous value only.

bool boolGlobalHistory, GlobalboolDisplayMeshWireframe;

// avi file -oriented variables
int const NUMAVI = 1;
HAVI hAvi[NUMAVI + 1];
int const GraphFlags[5] = { LAPAZ_AZ, VIZVEZJZAZDOT,
							SPECIES_ELECTRON, JZAZBXYEZ, SPECIES_NEUTRAL};
char szAvi[5][128] = { "LapAz_Az","VIZVEZJZAZDOT","Elec","JzAzBxyEz","Neut" };

AVICOMPRESSOPTIONS opts;
int counter;
HBITMAP surfbit, dib;
HDC surfdc, dibdc;
LPVOID lpvBits;
BITMAPINFO bitmapinfo;



char * report_time(int action)
{
	/* action = 0: reset ; action = 1: report */
	/* requires timebuffy to be defined as char[255] globally */

	static char timebuffer[255];
	static clock_t start;
	double timenow;
	long ops;

	if (action == 0)
	{
		start = clock();
	}
	else
	{
		timenow = ((double)(clock() - start) / (double)CLOCKS_PER_SEC);
		ops = (long)(clock() - start);
		/* create a null-terminated string */
		sprintf(timebuffer, "%6.4f sec.", timenow);
	};
	return &(timebuffer[0]);
};


f64 GetTriangleArea(f64_vec2 pos0, f64_vec2 pos1, f64_vec2 pos2)
{
	f64 area = 0.5*((pos0.x + pos1.x)*(pos1.y - pos0.y) + (pos1.x + pos2.x)*(pos2.y - pos1.y)
		+ (pos0.x + pos2.x)*(pos0.y - pos2.y));
	return fabs(area);
}


void TriMesh::CalculateTotalGraphingData()
{
	/*long iVertex;
	Vertex * pVertex = X;
	for (iVertex = 0; iVertex < numVertices; iVertex++)
	{
	if ((pVertex->flags == DOMAIN_VERTEX) || (pVertex->flags == OUTERMOST))
	{
	pVertex->n = (pVertex->Neut.mass + pVertex->Ion.mass) / pVertex->AreaCell;
	pVertex->v = (m_n*pVertex->Neut.mom + m_ion * pVertex->Ion.mom + m_e * pVertex->Elec.mom) /
	(m_n*pVertex->Neut.mass + m_ion * pVertex->Ion.mass + m_e * pVertex->Elec.mass);
	pVertex->T = (pVertex->Neut.heat + pVertex->Ion.heat + pVertex->Elec.heat) /
	(pVertex->Neut.mass + pVertex->Ion.mass + pVertex->Elec.mass);
	pVertex->Temp.x = pVertex->Ion.mass / (pVertex->Neut.mass + pVertex->Ion.mass);
	};
	++pVertex;
	}*/
}

void TriMesh::Setup_J()
{
	/*long iVertex;
	Vertex * pVertex = X;
	for (iVertex = 0; iVertex < numVertices; iVertex++)
	{
	if ((pVertex->flags == DOMAIN_VERTEX) || (pVertex->flags == OUTERMOST))
	{
	pVertex->Temp = q * (pVertex->Ion.mom - pVertex->Elec.mom) / pVertex->AreaCell;
	}
	else {
	memset(&(pVertex->Temp), 0, sizeof(Vector3));
	}
	++pVertex;
	}*/
}

void surfacegraph::DrawSurface(const char * szname,
	const int heightflag,
	const real * var_ptr_0,
	const int colourflag,
	const real * var_ptr_c,
	const bool bDisplayInner,
	const int code, // graph code, to pass to called routines - sometimes useful
	const TriMesh * pX // for passing to SetDataWithColour and Render
					   // and for working out offsets
)
{
	// replaced CreateSurfaceGraphs.
	// I think this is about the right balance.

	char buff[256];
	real * temprealptr = (real *)(pX->pData);
	long offset = var_ptr_0 - temprealptr;
	long offset_c = var_ptr_c - temprealptr;

	// Does shader always go with colour type?? yes I think.
	switch (colourflag) {
	case VELOCITY_COLOUR:
		this->mhTech = mFX->GetTechniqueByName("VelociTech");
		break;
	case SEGUE_COLOUR:
		this->mhTech = mFX->GetTechniqueByName("SegueTech");
		break;
	case CURRENT_COLOUR:
		this->mhTech = mFX->GetTechniqueByName("XYZTech");
		break;
	case AZSEGUE_COLOUR:
		mhTech = mFX->GetTechniqueByName("AzSegueTech");
		break;
	case IONISE_COLOUR:
		mhTech = mFX->GetTechniqueByName("IoniseTech");
		break;
	};

	// Usual settings:
	//if (GlobalGraphSetting[i] != GRAPH_NONE) {

	this->boolDisplayShadow = true;
	this->boolDisplayMainMesh = true;
	this->boolDisplayMeshWireframe = GlobalboolDisplayMeshWireframe;
	this->boolClearZBufferBeforeWireframe = false;
	// Or try setting true and CULL_CCW to see if this stops it showing "the back of the wireframe"
	this->SetEyeAndLookat(GlobalEye, GlobalLookat);
	this->boolDisplayScales = true;
	this->boolDisplayInnerMesh = bDisplayInner;

	// work out whether to display key button:
	if (((colourflag == FLAG_VELOCITY_COLOUR) || (colourflag == FLAG_CURRENT_COLOUR))
		&& (bDisplayInner == 0))
	{
		this->boolDisplayKeyButton = true;
	}
	else {
		this->boolDisplayKeyButton = false;
	};
	//int const FLAG_COLOUR_MESH = 0;
	//int const FLAG_SEGUE_COLOUR = 1;
	//int const FLAG_VELOCITY_COLOUR = 2;
	//int const FLAG_CURRENT_COLOUR = 3;
	//int const FLAG_AZSEGUE_COLOUR = 4;
	//int const FLAG_IONISE_COLOUR = 5;

	this->SetDataWithColour(*pX,
		colourflag, heightflag, // apparently it's that way round
		offset, offset_c,
		code);

	if (this->bDisplayTimestamp) {
		sprintf(buff, "%6.2f ps", evaltime*1.0e12);
		this->Render(szname, false, pX, buff);
	}
	else {
		this->Render(szname, false, pX);
	};
}

// Here we make a function that we can call to tidy up graph calling code:

// How we are going to do graphs:
// Use plasma_data objects from BEGINNING_OF_CENTRAL.

void RefreshGraphs(TriMesh & X, // only not const because of such as Reset_vertex_nvT
	const int iGraphsFlag)
{
	Vertex * pVertex;
	long iVertex;
	plasma_data * pdata;
	int offset_v, offset_T;
	char buff[256];
	sprintf(buff, "%5.2f ps", evaltime*1.0e12);

	switch (iGraphsFlag) {
		/*
		case JXY_RHO_EXY_GRADPHI_AXYDOTOC_AXY:

		X.Setup_J(); // the others can already exist.

		Graph[4].bDisplayTimestamp = true;

		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		if (pVertex->flags == DOMAIN_VERTEX) {
		pVertex->temp2.x = q * (pVertex->Ion.mass - pVertex->Elec.mass) / pVertex->AreaCell;
		}
		else {
		pVertex->temp2.x = 0.0;
		};
		pVertex->Adot /= c;
		++pVertex;
		}

		Graph[0].DrawSurface("Exy[statV/cm]",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].E)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].E)),
		true,
		GRAPH_EXY, &X);

		Graph[1].DrawSurface("Adotxy/c[statV/cm]",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].Adot.x)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].Adot.x)),
		true,
		GRAPH_ADOTXY, &X);

		Graph[2].DrawSurface("Jxy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].Temp.x)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].Temp.x)),
		false, // no inner mesh display.
		GRAPH_JXY, &X);

		Graph[3].DrawSurface("phidot[statV/s]",
		DATA_HEIGHT, (real *)(&(X.X[0].phidot)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].phidot)),
		true,
		GRAPH_PHIDOT, &X);

		Graph[4].DrawSurface("rho",
		DATA_HEIGHT, (real *)(&(X.X[0].temp2.x)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].temp2.x)),
		false, // no inner mesh display.
		GRAPH_RHO, &X);

		Graph[5].DrawSurface("phi[statV]",
		DATA_HEIGHT, (real *)(&(X.X[0].phi)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].phi)),
		true,
		GRAPH_PHI, &X);

		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		pVertex->Adot *= c;
		++pVertex;
		}
		break;
		case JZ_AZ_BXY_EZ_ADOTZOC_NVZ:

		X.Reset_vertex_nvT(SPECIES_ELEC);

		X.Setup_J(); // the others can already exist.

		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		pVertex->Adot /= c;
		++pVertex;
		}
		Graph[4].bDisplayTimestamp = true;

		Graph[0].DrawSurface("Ez[statV/cm]",
		DATA_HEIGHT, (real *)(&(X.X[0].E.z)),
		FLAG_SEGUE_COLOUR, (real *)(&(X.X[0].E.z)),
		false, // ??
		GRAPH_EZ, &X);

		Graph[1].DrawSurface("Az",
		DATA_HEIGHT, (real *)(&(X.X[0].A.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].A.z)),
		true,
		GRAPH_AZ, &X);
		Graph[2].DrawSurface("Jz",
		DATA_HEIGHT, (real *)(&(X.X[0].Temp.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].Temp.z)),
		false, // no inner mesh display.
		GRAPH_JZ, &X);
		Graph[3].DrawSurface("Bxy[Gs]",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].B)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].B)),
		true, // no inner mesh display: ??
		GRAPH_BXY, &X);

		Graph[4].DrawSurface("Adotz/c [statV/cm]",
		DATA_HEIGHT, (real *)(&(X.X[0].Adot.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].Adot.z)),
		true,
		GRAPH_AZ, &X);

		Graph[5].colourmax = Graph[2].colourmax;
		Graph[5].DrawSurface("Elec n",
		DATA_HEIGHT, (real *)(&(X.X[0].n)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].Temp.z)),
		false, // no inner mesh display
		GRAPH_ELEC_N, &X);

		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		pVertex->Adot *= c;
		++pVertex;
		}

		break;
		case SPECIES_ELECTRON2:

		X.Reset_vertex_nvT(SPECIES_ELEC);

		Graph[0].DrawSurface("Elec n [/cc]",
		DATA_HEIGHT, (real *)(&(X.X[0].n)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].v)),
		false, // no inner mesh display
		GRAPH_ELEC_N, &X);

		Graph[1].DrawSurface("v_e_xy[cm/s]",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].v)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].v)),
		false, // no inner mesh display
		GRAPH_ELEC_V, &X);

		Graph[3].DrawSurface("v_e_z[cm/s]",
		DATA_HEIGHT, (real *)(&(X.X[0].v.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].v.z)),
		false, // no inner mesh display.
		GRAPH_VEZ, &X);

		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		if (pVertex->flags == DOMAIN_VERTEX) {
		pVertex->temp2.x = (pVertex->Ion.mass - pVertex->Elec.mass) / pVertex->AreaCell;
		}
		else {
		pVertex->temp2.x = 0.0;
		};
		++pVertex;
		}
		Graph[2].bDisplayTimestamp = false;
		Graph[2].DrawSurface("n_i-n_e",
		DATA_HEIGHT, (real *)(&(X.X[0].temp2.x)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].temp2.x)),
		false, // no inner mesh display.
		GRAPH_NINE, &X);

		Graph[5].TickRescaling = 1.0 / kB;
		Graph[5].DrawSurface("Elec T [eV]",
		DATA_HEIGHT, (real *)(&(X.X[0].T)),
		SEGUE_COLOUR, (real *)(&(X.X[0].T)),
		false, // no inner mesh display
		GRAPH_ELEC_T, &X);
		Graph[5].TickRescaling = 1.0;

		offset_v = (real *)(&(X.X[0].v)) - (real *)(&(X.X[0]));
		offset_T = (real *)(&(X.X[0].T)) - (real *)(&(X.X[0]));

		Graph[4].SetEyePlan(GlobalPlanEye);
		Graph[4].boolDisplayMeshWireframe = true;
		Graph[4].boolClearZBufferBeforeWireframe = true;
		Graph[4].boolDisplayMainMesh = true;
		Graph[4].boolDisplayInnerMesh = false;
		Graph[4].boolDisplayScales = false;
		Graph[4].boolDisplayShadow = false;
		Graph[4].mhTech = Graph[4].mFX->GetTechniqueByName("VelociTech");
		Graph[4].colourmax = Graph[0].colourmax; // match colours
		Graph[4].SetDataWithColour(X, FLAG_VELOCITY_COLOUR, FLAG_FLAT_MESH, offset_v, offset_v,
		GRAPH_FLAT_WIRE_MESH);
		Graph[4].Render(buff, GlobalRenderLabels, &X);

		break;
		*/
	case SPECIES_NEUTRAL:


		Graph[0].DrawSurface("Neutral n",
			DATA_HEIGHT, (real *)(&(X.pData[0].n_n)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].v_n)),
			false, // no inner mesh display
			GRAPH_NEUT_N, &X);
		Graph[1].DrawSurface("Neutral v",
			VELOCITY_HEIGHT, (real *)(&(X.pData[0].v_n)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].v_n)),
			false, // no inner mesh display
			GRAPH_NEUT_V, &X);
		Graph[3].TickRescaling = 1.0 / kB_;
		Graph[3].DrawSurface("Neutral T",
			DATA_HEIGHT, (real *)(&(X.pData[0].Tn)),
			SEGUE_COLOUR, (real *)(&(X.pData[0].Tn)),
			false, // no inner mesh display
			GRAPH_NEUT_T, &X);
		Graph[3].TickRescaling = 1.0;
		// How to handle Graph[2] ?

		break;
	case SPECIES_ION:


		Graph[3].TickRescaling = 1.0 / kB_;
		Graph[3].DrawSurface("Ion T",
			DATA_HEIGHT, (real *)(&(X.pData[0].Ti)),
			SEGUE_COLOUR, (real *)(&(X.pData[0].Ti)),
			false, // no inner mesh display
			GRAPH_ION_T, &X);
		Graph[3].TickRescaling = 1.0;

		// labels only appear on first 1 called.

		Graph[0].DrawSurface("Ion n",
			DATA_HEIGHT, (real *)(&(X.pData[0].n)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].vxy)),
			false, // no inner mesh display
			GRAPH_ION_T, &X);
		Graph[1].DrawSurface("Ion v",
			VELOCITY_HEIGHT, (real *)(&(X.pData[0].vxy)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].vxy)),
			false, // no inner mesh display
			GRAPH_ION_V, &X);

		break;

	case SPECIES_ELEC:

		Graph[0].DrawSurface("Elec n",
			DATA_HEIGHT, (real *)(&(X.pData[0].n)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].vxy)),
			false, // no inner mesh display
			GRAPH_ELEC_T, &X);
		// colours == 0.0 ... because v = 0
		// First........... let's understand why surface normals come out unpredictable.
		// Then............ let's go and see what it does with y values (in Render and .fx)

		Graph[1].DrawSurface("Elec v",
			VELOCITY_HEIGHT, (real *)(&(X.pData[0].vxy)),
			VELOCITY_COLOUR, (real *)(&(X.pData[0].vxy)),
			false, // no inner mesh display
			GRAPH_ELEC_V, &X);
		Graph[3].TickRescaling = 1.0 / kB_;
		Graph[3].DrawSurface("Elec T",
			DATA_HEIGHT, (real *)(&(X.pData[0].Te)),
			SEGUE_COLOUR, (real *)(&(X.pData[0].Te)),
			false, // no inner mesh display
			GRAPH_ELEC_T, &X);
		Graph[3].TickRescaling = 1.0;
		break;

		// In other cases, (and even for the above),
		// here is a good place to call the 
		// setup routines for temp variables.

	case OVERALL:
		break;

	case JZAZBXYEZ:

		/*
		X.Setup_J(); // the others can already exist.

		printf("About to do Az\n");
		Graph[0].DrawSurface("Az",
		DATA_HEIGHT, (real *)(&(X.X[0].A.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].A.z)),
		true,
		GRAPH_AZ, &X);
		printf("About to do Bxy\n");
		Graph[1].DrawSurface("Bxy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].B)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].B)),
		true, // no inner mesh display: ??
		GRAPH_BXY, &X);
		printf("About to do Ez\n");
		Graph[2].DrawSurface("Ez",
		DATA_HEIGHT, (real *)(&(X.X[0].E.z)),
		FLAG_SEGUE_COLOUR, (real *)(&(X.X[0].E.z)),
		false, // ??
		GRAPH_EZ, &X);
		printf("About to do Jz\n");
		Graph[3].DrawSurface("Jz",
		DATA_HEIGHT, (real *)(&(X.X[0].Temp.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].Temp.z)),
		false, // no inner mesh display.
		GRAPH_JZ, &X);
		break;
		*/
		break;
	case VIZVEZJZAZDOT:

		// Set Jz:
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.x = q_ * pdata->n*(pdata->viz - pdata->vez);
			++pdata;
		};

		Graph[0].DrawSurface("viz",
			DATA_HEIGHT, (real *)(&(X.pData[0].viz)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false, GRAPH_VIZ, &X);

		Graph[1].DrawSurface("vez",
			DATA_HEIGHT, (real *)(&(X.pData[0].vez)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false, GRAPH_VEZ, &X);

		Graph[2].DrawSurface("Azdot",
			DATA_HEIGHT, (real *)(&(X.pData[0].Azdot)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].Azdot)),
			true, GRAPH_AZDOT, &X);

		Graph[3].DrawSurface("Jz",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			false, GRAPH_JZ, &X);

		break;
	case LAPAZ_AZ:
		// Assume temp.x contains Lap Az
		Graph[0].DrawSurface("Lap Az",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.x)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.x)),
			true, GRAPH_LAPAZ, &X);
		Graph[1].DrawSurface("Az",
			DATA_HEIGHT, (real *)(&(X.pData[0].Az)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].Az)),
			true, GRAPH_AZ, &X);
		Graph[2].DrawSurface("Azdot",
			DATA_HEIGHT, (real *)(&(X.pData[0].Azdot)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].Azdot)),
			true, GRAPH_AZDOT, &X);
		pdata = X.pData + BEGINNING_OF_CENTRAL;
		for (iVertex = 0; iVertex < NUMVERTICES; iVertex++)
		{
			pdata->temp.y = q_ * pdata->n*(pdata->viz - pdata->vez);
			++pdata;
		};
		Graph[3].DrawSurface("Jz",
			DATA_HEIGHT, (real *)(&(X.pData[0].temp.y)),
			AZSEGUE_COLOUR, (real *)(&(X.pData[0].temp.y)),
			false, GRAPH_JZ, &X);

		break;
	case EXYCOMPONENTS:
		/*
		X.Setup_J(); // the others can already exist.

		Graph[0].DrawSurface("Adotxy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].Adot.x)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].Adot.x)),
		true,
		GRAPH_ADOTXY, &X);
		Graph[1].DrawSurface("Grad phi",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].GradTe)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].GradTe)),
		true, // no inner mesh display: ??
		GRAPH_GRADPHI, &X);
		Graph[2].DrawSurface("Exy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].E)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].E)),
		true,
		GRAPH_EXY, &X);
		Graph[3].DrawSurface("Jxy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].Temp.x)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].Temp.x)),
		false, // no inner mesh display.
		GRAPH_JXY, &X);

		*/
		// Set GradTe to grad phi
		break;
	case JXYAXYBZEXY:
		/*
		X.Setup_J(); // the others can already exist.

		Graph[0].DrawSurface("Axy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].A.x)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].A.x)),
		true,
		GRAPH_AXY, &X);
		Graph[1].DrawSurface("Bz",
		DATA_HEIGHT, (real *)(&(X.X[0].B.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].B.z)),
		true, // no inner mesh display: ??
		GRAPH_BZ, &X);
		Graph[2].DrawSurface("Exy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].E)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].E)),
		true,
		GRAPH_EXY, &X);
		Graph[3].DrawSurface("Jxy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].Temp.x)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].Temp.x)),
		false, // no inner mesh display.
		GRAPH_JXY, &X);
		*/
		break;
	case EXY_RHO_PHI_PHIDOT:
		/*
		// For this one do n_i-n_e
		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		if (pVertex->flags == DOMAIN_VERTEX) {
		pVertex->temp2.x = (pVertex->Ion.mass - pVertex->Elec.mass) / pVertex->AreaCell;
		}
		else {
		pVertex->temp2.x = 0.0;
		};
		++pVertex;
		}

		Graph[0].DrawSurface("phi",
		DATA_HEIGHT, (real *)(&(X.X[0].phi)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].phi)),
		true,
		GRAPH_PHI, &X);
		Graph[1].DrawSurface("phidot",
		DATA_HEIGHT, (real *)(&(X.X[0].phidot)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].phidot)),
		true,
		GRAPH_PHIDOT, &X);
		Graph[2].DrawSurface("Exy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].E)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].E)),
		true,
		GRAPH_EXY, &X);
		Graph[3].DrawSurface("n_i-n_e",
		DATA_HEIGHT, (real *)(&(X.X[0].temp2.x)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].temp2.x)),
		false, // no inner mesh display.
		GRAPH_NINE, &X);
		*/
		break;
	case EXY_RHO_PHI_JXY:
		// create rho on pVertex->temp2.x ... 
		/*
		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		if (pVertex->flags == DOMAIN_VERTEX) {
		pVertex->temp2.x = q * (pVertex->Ion.mass - pVertex->Elec.mass) / pVertex->AreaCell;
		}
		else {
		pVertex->temp2.x = 0.0;
		};
		++pVertex;
		}

		X.Setup_J();

		Graph[0].DrawSurface("phi",
		DATA_HEIGHT, (real *)(&(X.X[0].phi)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].phi)),
		false,
		GRAPH_PHI, &X);
		Graph[1].DrawSurface("Jxy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].Temp)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].Temp)),
		false, // no inner mesh display: ??
		GRAPH_JXY, &X);
		Graph[2].DrawSurface("Exy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].E)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].E)),
		false,
		GRAPH_EXY, &X);
		Graph[3].DrawSurface("rho",
		DATA_HEIGHT, (real *)(&(X.X[0].temp2.x)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].temp2.x)),
		false, // no inner mesh display.
		GRAPH_RHO, &X);
		*/
		break;

	case EXY_RHO_BZ_JXY:
		/*
		// create rho on pVertex->temp2.x ...
		pVertex = pX->X;
		for (iVertex = 0; iVertex < pX->numVertices; iVertex++)
		{
		if (pVertex->flags == DOMAIN_VERTEX) {
		pVertex->temp2.x = q * (pVertex->Ion.mass - pVertex->Elec.mass) / pVertex->AreaCell;
		}
		else {
		pVertex->temp2.x = 0.0;
		};
		++pVertex;
		}

		X.Setup_J();

		Graph[0].DrawSurface("Bz",
		DATA_HEIGHT, (real *)(&(X.X[0].B.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].B.z)),
		true, // no inner mesh display: ??
		GRAPH_BZ, &X);
		Graph[1].DrawSurface("Jxy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].Temp)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].Temp)),
		false, // no inner mesh display: ??
		GRAPH_JXY, &X);
		Graph[2].DrawSurface("Exy",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].E)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].E)),
		false,
		GRAPH_EXY, &X);
		Graph[3].DrawSurface("rho",
		DATA_HEIGHT, (real *)(&(X.X[0].temp2.x)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].temp2.x)),
		false, // no inner mesh display.
		GRAPH_RHO, &X);
		*/
		break;

	case SIGMA_E_J:
		/*
		X.Setup_J(); // the others can already exist.

		Graph[0].DrawSurface("sigma_e_zz",
		DATA_HEIGHT, (real *)(&(X.X[0].sigma_e.zz)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].sigma_e.zz)),
		true,
		GRAPH_SIGMA_E, &X);
		//Graph[1].DrawSurface("v_e_0.z",
		//	DATA_HEIGHT,(real *)(&(X.X[0].v_e_0.z)),
		//	AZSEGUE_COLOUR,(real *)(&(X.X[0].v_e_0.z)),
		//false, // no inner mesh display: ??
		// GRAPH_VE0Z, &X);
		Graph[1].DrawSurface("nsigma",
		DATA_HEIGHT, (real *)(&(X.X[0].xdotdot.x)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].xdotdot.x)),
		true, GRAPH_SIGMATEMP, &X);
		Graph[2].DrawSurface("Ez",
		DATA_HEIGHT, (real *)(&(X.X[0].E.z)),
		FLAG_AZSEGUE_COLOUR, (real *)(&(X.X[0].E.z)), // how to make SEGUE_COLOUR work?
		false, // ??
		GRAPH_EZ, &X);
		Graph[3].DrawSurface("Jz",
		DATA_HEIGHT, (real *)(&(X.X[0].Temp.z)),
		AZSEGUE_COLOUR, (real *)(&(X.X[0].Temp.z)),
		false, // no inner mesh display.
		GRAPH_JZ, &X);
		*/
		break;

	case TOTAL:
		break;
		// In this case we have to create data,
		// as we go.
		/*
		// Best put it here so we can see where
		// data is being populated.

		X.CalculateTotalGraphingData();

		// ought to change this to use variables n,v,T !

		Graph[0].DrawSurface("n_n+n_ion",
		DATA_HEIGHT, (real *)(&(X.X[0].n)),
		IONISE_COLOUR, (real *)(&(X.X[0].Temp.y)),
		false,
		GRAPH_TOTAL_N, &X);
		Graph[1].DrawSurface("[n_s v_s m_s]/[n_s m_s]",
		VELOCITY_HEIGHT, (real *)(&(X.X[0].v)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].v)),
		false, // no inner mesh display
		GRAPH_TOTAL_V, &X);
		Graph[2].DrawSurface("n_n+n_ion",
		DATA_HEIGHT, (real *)(&(X.X[0].n)),
		VELOCITY_COLOUR, (real *)(&(X.X[0].v)),
		false,
		GRAPH_TOTAL_N_II, &X);
		Graph[3].TickRescaling = 1.0 / kB;
		Graph[3].DrawSurface("[n_s T_s]/[n_s]",
		DATA_HEIGHT, (real *)(&(X.X[0].T)),
		SEGUE_COLOUR, (real *)(&(X.X[0].T)),
		false,
		GRAPH_TOTAL_T, &X);
		Graph[3].TickRescaling = 1.0;
		break;
		*/
	};

	// Graph 2, in case of species graphs:

	switch (iGraphsFlag) {
	case SPECIES_NEUTRAL:
	case SPECIES_ION:
	case SPECIES_ELEC:

		int offset_v, offset_T;
		offset_v = (real *)(&(X.pData[0].vxy)) - (real *)(&(X.pData[0]));
		offset_T = (real *)(&(X.pData[0].Te)) - (real *)(&(X.pData[0]));

		Graph[2].SetEyePlan(GlobalPlanEye);
		Graph[2].boolDisplayMeshWireframe = true;
		Graph[2].boolClearZBufferBeforeWireframe = true;
		Graph[2].boolDisplayMainMesh = true;
		Graph[2].boolDisplayInnerMesh = false;
		Graph[2].boolDisplayScales = false;

		if (GlobalColoursPlanView == 0)
		{
			// nothing
			Graph[2].mhTech = Graph[2].mFX->GetTechniqueByName("MeshTech");
			Graph[2].SetDataWithColour(X, FLAG_COLOUR_MESH, FLAG_FLAT_MESH, 0, 0,
				GRAPH_FLAT_WIRE_MESH);
			Graph[2].Render(buff, GlobalRenderLabels, &X);

		}
		else {
			// Tell SDWC not to mess with colourmax if it's a flat mesh.

			if (GlobalColoursPlanView == 1)
			{
				// velocity
				Graph[2].mhTech = Graph[2].mFX->GetTechniqueByName("VelociTech");
				Graph[2].colourmax = Graph[0].colourmax; // match colours

				Graph[2].SetDataWithColour(X, FLAG_VELOCITY_COLOUR, FLAG_FLAT_MESH, offset_v, offset_v,
					GRAPH_FLAT_WIRE_MESH);
				Graph[2].Render(buff, GlobalRenderLabels, &X);
			}
			else {
				// temperature
				Graph[2].mhTech = Graph[2].mFX->GetTechniqueByName("SegueTech");
				// SegueVS should take maximum as a parameter;
				// at least for colours we should prefer an absolute scale for T
				// Is it ever used for anything else? Not so far? eps?

				Graph[2].SetDataWithColour(X, FLAG_SEGUE_COLOUR, FLAG_FLAT_MESH, offset_T, offset_T,
					GRAPH_FLAT_WIRE_MESH);
				Graph[2].Render(buff, GlobalRenderLabels, &X);
			};
		};
		break;
	}
}




__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };
	
	HINSTANCE hInstance = GetModuleHandle(NULL);
	HWND hwndConsole = GetConsoleWindow();
	char szInitialFilenameAvi[512];
	MSG msg;
	HDC hdc;
	//	HACCEL hAccelTable;
	real x, y, temp;
	int i, j;
	float a1, a2, a3, a4;
	//HWND hwndConsole;
	FILE * fp;
	extern char Functionalfilename[1024];
	
	h = TIMESTEP;
	evaltime = 0.0; // gets updated before advance

	memset(Historic_powermax, 0, 200 * sizeof(int));
	memset(Historic_powermin, 0, 200 * sizeof(int));

	ZeroMemory(Historic_max, 100 * HISTORY * sizeof(float));
	ZeroMemory(Historic_min, 100 * HISTORY * sizeof(float));
	GlobalStepsCounter = 0; steps_remaining = 0; steps_remaining_CPU = 0;

	SetConsoleTitle("2D 1/16 annulus DPF simulation");
	Sleep(40);
	//hwndConsole = FindWindow(NULL, "2D 1/16 annulus DPF simulation");
	MoveWindow(hwndConsole, 0, 0, SCREEN_WIDTH - VIDEO_WIDTH - 10, SCREEN_HEIGHT - 30, TRUE);

	report_time(0);

	int filetag = 0;
	do {
		filetag++;
		sprintf(Functionalfilename, FUNCTIONALFILE_START "%03d.txt", filetag);
	} while ((_access(Functionalfilename, 0)) != -1);
	printf("\n\nopening %s \n", Functionalfilename);
	fp = fopen(Functionalfilename, "w");
	if (fp == 0) {
		printf("error with %s \n", Functionalfilename);
		getch();
	}
	else {
		printf("opened %s \n", Functionalfilename);
	};
	fprintf(fp, "GSC evaltime Area neut.N ion.N elec.N neut.r ion.r elec.r SDneut.r SDion.r SDelec.r "
		" neut.vr neut.vth neut.vz  ion.vr ion.vth ion.vz elec.vr elec.vth elec.vz neut.heat ion.heat elec.heat neut.T ion.T elec.T "
		" neut.mnvv/3 ion.mnvv/3 elec.mnvv/3 elec.force(vxB)r within3.6 elec.Bth EE BB Heatings and dT changes - see code \n");
	fclose(fp);

	X1.Initialise(1); // Set evaltime first
	X2.Initialise(2);
	X3.Initialise(3);
	pX = &X1;
	pXnew = &X2;

	GlobalBothSystemsInUse = 0;

	printf(report_time(1));
	printf("\n");
	report_time(0);



	// Window setup
	LoadString(hInstance, IDS_APP_TITLE, szTitle, 1024);
	LoadString(hInstance, IDC_F2DVALS, szWindowClass, 1024);
	wcex.cbSize = sizeof(WNDCLASSEX);
	wcex.style = CS_HREDRAW | CS_VREDRAW;
	wcex.lpfnWndProc = WndProc;
	wcex.cbClsExtra = 0;
	wcex.cbWndExtra = 0;
	wcex.hInstance = hInstance;
	wcex.hIcon = LoadIcon(hInstance, MAKEINTRESOURCE(IDI_F2DVALS));
	wcex.hCursor = LoadCursor(NULL, IDC_ARROW);
	wcex.hbrBackground = (HBRUSH)(COLOR_WINDOW + 1);
	wcex.lpszMenuName = MAKEINTRESOURCE(IDR_MENU1);
	wcex.lpszClassName = szWindowClass;
	wcex.hIconSm = LoadIcon(wcex.hInstance, MAKEINTRESOURCE(IDI_SMALL));
	if (RegisterClassEx(&wcex) == 0) {
		char buff[128];
		MessageBox(NULL, "RegisterClassEx failed", itoa(GetLastError(), buff, 10), MB_OK);
	};


	printf("SCREEN_WIDTH %d VIDEO_WIDTH %d VIDEO_HEIGHT %d \n",
		SCREEN_WIDTH, VIDEO_WIDTH, VIDEO_HEIGHT);

	hWnd = CreateWindowEx(NULL, szWindowClass, szTitle, WS_BORDER | WS_POPUP,
		SCREEN_WIDTH - VIDEO_WIDTH - 5, 0, VIDEO_WIDTH + 5, VIDEO_HEIGHT + 20, NULL, NULL, hInstance, NULL);
	if (!hWnd) {
		DWORD dword = GetLastError();
		char buff[128];
		MessageBox(NULL, "CreateWindowEx failed", itoa(dword, buff, 10), MB_OK);
		return dword;
	}
	// This is sending a message to WndProc before any of the following happens.

	ShowWindow(hWnd, SW_SHOWNORMAL);
	UpdateWindow(hWnd);

	xzscale = 2.0 / 0.1; // very zoomed in. Now what?

	DXChk(Direct3D.Initialise(hWnd, hInstance, VIDEO_WIDTH, VIDEO_HEIGHT));

	// With Field Of View = PI/4 used this:

	GlobalEye.x = 0.0f;
	GlobalEye.y = 12.4f;  //7.2f;
	GlobalEye.z = -18.0f + 2.5*xzscale;//DEVICE_RADIUS_INSULATOR_OUTER*xzscale;//-17.8f+

	GlobalLookat.x = 0.4f;
	GlobalLookat.y = 3.0f;
	GlobalLookat.z = DEVICE_RADIUS_INITIAL_FILAMENT_CENTRE * xzscale;

	GlobalPlanEye.x = 0.0f;
	GlobalPlanEye.y = 35.0f;
	GlobalPlanEye.z = (3.44 + 4.1)*0.5*xzscale;

	GlobalPlanEye2.x = -0.1f;
	GlobalPlanEye2.y = 19.5f;
	GlobalPlanEye2.z = 2.8*xzscale;

	GlobalPlanLookat.x = GlobalPlanEye.x;
	GlobalPlanLookat.y = 0.0f;
	GlobalPlanLookat.z = GlobalPlanEye.z + 0.0001;

	GlobalPlanLookat2.x = GlobalPlanEye2.x;
	GlobalPlanLookat2.y = 0.0f;
	GlobalPlanLookat2.z = GlobalPlanEye2.z + 0.0001;

						 // Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	if (DXChk(Graph[0].InitialiseWithoutBuffers(0, 0, GRAPH_WIDTH, GRAPH_HEIGHT, GlobalEye, GlobalLookat)) +
		DXChk(Graph[0].InitialiseBuffers(X1))
		)
	{
		PostQuitMessage(200);
	};
	if (DXChk(Graph[1].InitialiseWithoutBuffers(0, GRAPH_HEIGHT, GRAPH_WIDTH, GRAPH_HEIGHT, GlobalEye, GlobalLookat)) +
		DXChk(Graph[1].InitialiseBuffers(X1))
		)
	{
		PostQuitMessage(201);
	};
	if (DXChk(Graph[2].InitialiseWithoutBuffers(GRAPH_WIDTH, 0, GRAPH_WIDTH, GRAPH_HEIGHT, GlobalPlanEye, GlobalPlanLookat)) +
		DXChk(Graph[2].InitialiseBuffers(X1))
		)
	{
		PostQuitMessage(202);
	};
	if (DXChk(Graph[3].InitialiseWithoutBuffers(GRAPH_WIDTH, GRAPH_HEIGHT, GRAPH_WIDTH, GRAPH_HEIGHT, GlobalEye, GlobalLookat)) +
		DXChk(Graph[3].InitialiseBuffers(X1))
		)
	{
		PostQuitMessage(203);
	};

	if (NUMGRAPHS > 4) {

		if (DXChk(Graph[4].InitialiseWithoutBuffers(GRAPH_WIDTH * 2, 0, GRAPH_WIDTH, GRAPH_HEIGHT, GlobalPlanEye, GlobalPlanLookat)) +
			DXChk(Graph[4].InitialiseBuffers(X1))
			)
		{
			PostQuitMessage(204);
		};

		if (DXChk(Graph[5].InitialiseWithoutBuffers(GRAPH_WIDTH * 2, GRAPH_HEIGHT, GRAPH_WIDTH, GRAPH_HEIGHT, GlobalPlanEye, GlobalPlanLookat)) +
			DXChk(Graph[5].InitialiseBuffers(X1))
			)
		{
			PostQuitMessage(204);
		};

	};

	Graph[0].bDisplayTimestamp = false;
	Graph[1].bDisplayTimestamp = false;
	Graph[2].bDisplayTimestamp = true;
	Graph[3].bDisplayTimestamp = false;
	Graph[4].bDisplayTimestamp = false;
	Graph[5].bDisplayTimestamp = false;

	Direct3D.pd3dDevice->GetBackBuffer(0, 0, D3DBACKBUFFER_TYPE_MONO, &p_backbuffer_surface);

	if (DXChk(p_backbuffer_surface->GetDC(&surfdc), 1000))
		MessageBox(NULL, "GetDC failed", "oh dear", MB_OK);

	surfbit = CreateCompatibleBitmap(surfdc, VIDEO_WIDTH, VIDEO_HEIGHT); // EXTRAHEIGHT = 90
	SelectObject(surfdc, surfbit);
	dibdc = CreateCompatibleDC(surfdc);

	long VideoWidth = VIDEO_WIDTH;
	long VideoHeight = VIDEO_HEIGHT;

	// pasted here just to set up format:
	bitmapinfo.bmiHeader.biSize = sizeof(BITMAPINFO);
	bitmapinfo.bmiHeader.biWidth = VideoWidth;
	bitmapinfo.bmiHeader.biHeight = VideoHeight;
	bitmapinfo.bmiHeader.biPlanes = 1;
	bitmapinfo.bmiHeader.biBitCount = 24;
	bitmapinfo.bmiHeader.biCompression = BI_RGB; // uncompressed  
	bitmapinfo.bmiHeader.biSizeImage = bitmapinfo.bmiHeader.biHeight;
	bitmapinfo.bmiHeader.biXPelsPerMeter = 3000;
	bitmapinfo.bmiHeader.biYPelsPerMeter = 3000;
	bitmapinfo.bmiHeader.biClrUsed = 0;
	bitmapinfo.bmiHeader.biClrImportant = 0;
	bitmapinfo.bmiColors->rgbBlue = 0;
	bitmapinfo.bmiColors->rgbRed = 0;
	bitmapinfo.bmiColors->rgbGreen = 0;
	bitmapinfo.bmiColors->rgbReserved = 0;
	// dimension DIB and set up pointer to bits
	dib = CreateDIBSection(dibdc, &bitmapinfo, DIB_RGB_COLORS, &lpvBits, NULL, 0);
	SelectObject(dibdc, dib);

	BitBlt(dibdc, 0, 0, VIDEO_WIDTH, VIDEO_HEIGHT, surfdc, 0, 0, SRCCOPY);


	for (i = 0; i < NUMAVI; i++)
	{
		sprintf(szInitialFilenameAvi, "%s%s_%s", FOLDER, szAvi[i], INITIALAVI);
		hAvi[i] = CreateAvi(szInitialFilenameAvi, AVIFRAMEPERIOD, NULL);
	};

	// 1000/25 = 40
	ZeroMemory(&opts, sizeof(opts));
	opts.fccHandler = mmioFOURCC('D', 'I', 'B', ' ');//('d','i','v','x');
	opts.dwFlags = 8;

	for (i = 0; i < NUMAVI; i++)
		SetAviVideoCompression(hAvi[i], dib, &opts, false, hWnd); // always run this for every avi file but can
																  // call with false as long as we know opts contains valid information. 
	counter = 0;
	//ReleaseDC(hWnd,surfdc);
	p_backbuffer_surface->ReleaseDC(surfdc);

	GlobalCutaway = true;
	
	RefreshGraphs(*pX, GlobalSpeciesToGraph); 

	//printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
	//	c[0], c[1], c[2], c[3], c[4]);
	Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);

						 // Main message loop:
	memset(&msg, 0, sizeof(MSG));
	while (msg.message != WM_QUIT)
	{
		if (PeekMessage(&msg, NULL, 0U, 0U, PM_REMOVE))
		{
			TranslateMessage(&msg);
			DispatchMessage(&msg);
		}
		else {
		
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);

		};
	};

	UnregisterClass(szWindowClass, wcex.hInstance);


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}


LRESULT CALLBACK WndProc(HWND hWnd, UINT message, WPARAM wParam, LPARAM lParam)
{
	static bool bInvoked_cuSyst = false;
	static long GSCCPU = 0;
	int iAntiskips;
	int wmId, wmEvent;
	int i, j, ctr;
	PAINTSTRUCT ps;
	HDC hdc;
	real time_back_for_Adot;
	FILE * file, *fp;
	int maxeerr, count, iMin;
	char buf1000[1024];
	int attempts;
	real store_h;
	char ch, o;
	int failed;
	RECT rect;
	real TotalArea, TotalCharge;
	long iVertex;
	real mass_avg, mass_SD, mass_min, mass_max;
	OPENFILENAME ofn;       // common dialog box structure
	char szFile[260];       // buffer for file name
	char szFilter[1000]; // buffer for file filter
	char szfilter[256];
	char buffer[256];

	TriMesh * temp;
	Vertex * pVertex;

	static const real XCENTRE2 = DEVICE_RADIUS_INITIAL_FILAMENT_CENTRE * sin(PI / 32.0);
	static const real XCENTRE1 = -XCENTRE2;
	static const real YCENTRE = DEVICE_RADIUS_INITIAL_FILAMENT_CENTRE * cos(PI / 32.0);

	switch (message)
	{
	case WM_CREATE:

		// Don't ever try doing initialisation here;
		// That should be done manually from the menus.		

		break;

	case WM_COMMAND:
		wmId = LOWORD(wParam);
		wmEvent = HIWORD(wParam);

		// Ensure that display menu items are consecutive IDs.
		// Parse the menu selections:
		switch (wmId)
		{
		case ID_DISPLAY_NEUT:
		case ID_DISPLAY_ELECTRON:
		case ID_DISPLAY_ION:
		case ID_DISPLAY_TOTAL:
		case ID_DISPLAY_JZAZBXYEZ:
		case ID_DISPLAY_VIZVEZJZAZDOT:

			GlobalSpeciesToGraph = wmId;

			i = wmId - ID_DISPLAY_NEUT;
			printf("\nGraph: %d %s", i, szAvi[i]);
			RefreshGraphs(*pX, i);
			Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);
			break;

		case ID_HELP_ABOUT:
			DialogBox(hInst, MAKEINTRESOURCE(IDD_ABOUTBOX), hWnd, About);
			break;
		case ID_FILE_EXIT:
			DestroyWindow(hWnd);
			break;

		case ID_FILE_SAVECAMERA:
			ZeroMemory(&ofn, sizeof(ofn));
			ofn.lStructSize = sizeof(ofn);
			ofn.hwndOwner = hWnd;
			ofn.lpstrFile = szFile;
			// Set lpstrFile[0] to '\0' so that GetOpenFileName does not 
			// use the contents of szFile to initialize itself.
			ofn.lpstrFile[0] = '\0';
			ofn.nMaxFile = sizeof(szFile);
			memcpy(szfilter, "All\0*.*\0cam\0*.CAM\0\0", 19); // strcpy stops at first null !!
			ofn.lpstrFilter = szfilter; //"All\0*.*\0Dat\0*.DAT\0\0";	// summat weird about that example code
			ofn.nFilterIndex = 1;
			ofn.lpstrFileTitle = NULL;
			ofn.nMaxFileTitle = 0;
			ofn.lpstrInitialDir = NULL;
			ofn.Flags = OFN_OVERWRITEPROMPT;
			ofn.lpstrTitle = NULL;

			if (GetSaveFileName(&ofn) == TRUE)
			{
				printf("\nsaving camera...");
				fp = fopen(ofn.lpstrFile, "wt");
				if (fp == 0) {
					printf("save failed.\n");
				}
				else {
					fprintf(fp, "%f %f %f ", GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
					fprintf(fp, "%f %f %f ", GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);
					fprintf(fp, "%f %f %f ", GlobalEye.x, GlobalEye.y, GlobalEye.z);
					fprintf(fp, "%f %f %f ", GlobalPlanLookat.x, GlobalPlanLookat.y, GlobalPlanLookat.z);
					fclose(fp);
					printf("done\n");
				};
			}
			else {
				printf("there was an issue\n");
			};
			break;
		case ID_FILE_LOADCAMERA:
			// Initialize OPENFILENAME
			ZeroMemory(&ofn, sizeof(ofn));
			ofn.lStructSize = sizeof(ofn);
			ofn.hwndOwner = hWnd;
			ofn.lpstrFile = szFile;
			// Set lpstrFile[0] to '\0' so that GetOpenFileName does not 
			// use the contents of szFile to initialize itself.
			ofn.lpstrFile[0] = '\0';
			ofn.nMaxFile = sizeof(szFile);
			memcpy(szfilter, "All\0*.*\0*.cam\0*.Cam\0\0", 21); // strcpy stops at first null !!
			ofn.lpstrFilter = szfilter; //"All\0*.*\0*.Dat\0*.DAT\0\0";	// summat weird about that example code
			ofn.nFilterIndex = 1;
			ofn.lpstrFileTitle = NULL;
			ofn.nMaxFileTitle = 0;
			ofn.lpstrInitialDir = NULL;
			ofn.Flags = OFN_PATHMUSTEXIST | OFN_FILEMUSTEXIST;
			// Display the Open dialog box. 
			if (GetOpenFileName(&ofn) == TRUE)
			{
				printf("\nloading camera...");
				fp = fopen(ofn.lpstrFile, "rt");
				if (fp == 0) {
					printf("failed.\n");
				}
				else {
					rewind(fp);
					fscanf(fp, "%f %f %f ", &(GlobalPlanEye.x), &(GlobalPlanEye.y), &(GlobalPlanEye.z));
					fscanf(fp, "%f %f %f ", &(GlobalLookat.x), &(GlobalLookat.y), &(GlobalLookat.z));
					fscanf(fp, "%f %f %f ", &(GlobalEye.x), &(GlobalEye.y), &(GlobalEye.z));
					fscanf(fp, "%f %f %f ", &(GlobalPlanLookat.x), &(GlobalPlanLookat.y), &(GlobalPlanLookat.z));
					fclose(fp);
				};
				RefreshGraphs(*pX, GlobalSpeciesToGraph); // sends data to graphs AND renders them
				Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);

			}
			else {
				printf("file error camera\n");
			};
			break; 
		case ID_FILE_SAVEBINARY:
			// Initialize OPENFILENAME
			ZeroMemory(&ofn, sizeof(ofn));
			ofn.lStructSize = sizeof(ofn);
			ofn.hwndOwner = hWnd;
			ofn.lpstrFile = szFile;
			// Set lpstrFile[0] to '\0' so that GetOpenFileName does not 
			// use the contents of szFile to initialize itself.
			ofn.lpstrFile[0] = '\0';
			ofn.nMaxFile = sizeof(szFile);
			memcpy(szfilter, "All\0*.*\0*.dat\0*.Dat\0\0", 21); // strcpy stops at first null !!
			ofn.lpstrFilter = szfilter; //"All\0*.*\0Dat\0*.DAT\0\0";	// summat weird about that example code
			ofn.nFilterIndex = 1;
			ofn.lpstrFileTitle = NULL;
			ofn.nMaxFileTitle = 0;
			ofn.lpstrInitialDir = NULL;
			ofn.Flags = OFN_OVERWRITEPROMPT;
			ofn.lpstrTitle = NULL;
			// Display the Open dialog box. 
			if (GetSaveFileName(&ofn) == TRUE)
			{
				printf("\nsaving system...");
				pX->Save(ofn.lpstrFile);
				printf("done\n");
			}
			else {
				printf("there was an issue\n");
			};
			break;
		case ID_FILE_SAVETEXT:
			// Initialize OPENFILENAME
			ZeroMemory(&ofn, sizeof(ofn));
			ofn.lStructSize = sizeof(ofn);
			ofn.hwndOwner = hWnd;
			ofn.lpstrFile = szFile;
			//
			// Set lpstrFile[0] to '\0' so that GetOpenFileName does not 
			// use the contents of szFile to initialize itself.
			ofn.lpstrFile[0] = '\0';
			ofn.nMaxFile = sizeof(szFile);
			//strcpy(szFilter,"All\0*.*\0Text\0*.TXT\0");
			memcpy(szfilter, "All\0*.*\0Dat\0*.DAT\0\0", 19); // strcpy stops at first null !!
			ofn.lpstrFilter = szfilter; //"All\0*.*\0Dat\0*.DAT\0\0";	// summat weird about that example code
			ofn.nFilterIndex = 1;
			ofn.lpstrFileTitle = NULL;
			ofn.nMaxFileTitle = 0;
			ofn.lpstrInitialDir = NULL;
			ofn.Flags = OFN_OVERWRITEPROMPT;
			ofn.lpstrTitle = NULL;
			// Display the Open dialog box. 
			if (GetSaveFileName(&ofn) == TRUE)
			{
				printf("\nsaving system...");
				pX->SaveText(ofn.lpstrFile);
				printf("done\n");
			}
			else {
				printf("there was an issue\n");
			};
			break;
		case ID_FILE_LOAD:

			// Initialize OPENFILENAME:
			ZeroMemory(&ofn, sizeof(ofn));
			ofn.lStructSize = sizeof(ofn);
			ofn.hwndOwner = hWnd;
			ofn.lpstrFile = szFile;
			//
			// Set lpstrFile[0] to '\0' so that GetOpenFileName does not 
			// use the contents of szFile to initialize itself.
			ofn.lpstrFile[0] = '\0';
			ofn.nMaxFile = sizeof(szFile);
			//strcpy(szFilter, "All\0*.*\0Dat\0*.DAT\0\0");
			memcpy(szfilter, "All\0*.*\0Dat\0*.DAT\0\0", 19); // strcpy stops at first null !!
			ofn.lpstrFilter = szfilter; //"All\0*.*\0Dat\0*.DAT\0\0";	// summat weird about that example code
			ofn.nFilterIndex = 1;
			ofn.lpstrFileTitle = NULL;
			ofn.nMaxFileTitle = 0;
			ofn.lpstrInitialDir = NULL;
			ofn.Flags = OFN_PATHMUSTEXIST | OFN_FILEMUSTEXIST;

			// Display the Open dialog box. 
			if (GetOpenFileName(&ofn) == TRUE)
			{
				pX->Load(ofn.lpstrFile);
				printf("\ndoin nothing...");
			};
		break;

		case ID_RUN_SIMULATIONSTEPS:

			GlobalSwitchBox = 0;
			DialogBox(hInst, MAKEINTRESOURCE(IDD_DIALOG1), hWnd, SetupBox);
			// that will not return with steps_remaining unset.

			if (steps_remaining > 0)
				SetTimer(hWnd, 1, 1, NULL); // 1 millisecond delay

			break;

		case ID_RUN_SIMULATIONSTEPS_CPU:

			GlobalSwitchBox = 0;
			steps_remaining_CPU = 1;
			// that will not return with steps_remaining unset.

			if (steps_remaining_CPU > 0)
				SetTimer(hWnd, 2, 1, NULL); // 1 millisecond delay

			break;

		case ID_RUN_STOP:

			steps_remaining = 0;
			steps_remaining_CPU = 0;
			break;
		case ID_INITIALISE_IONISATIONSTEPS:
			break;

		default:
			return DefWindowProc(hWnd, message, wParam, lParam);
		}
		break;

	case WM_TIMER:
		
		KillTimer(hWnd, wParam);
		report_time(0);
		if (wParam == 1)
		{
			if (bInvoked_cuSyst == false) {
				bInvoked_cuSyst = true;
				cuSyst_host.InvokeHost();

				pX->EnsureAnticlockwiseTriangleCornerSequences_SetupTriMinorNeighboursLists();
				pX->Average_n_T_to_tris_and_calc_centroids_and_minorpos();
				cuSyst_host.PopulateFromTriMesh(pX);
				cuSyst_host2.InvokeHost();
				cuSyst_host2.PopulateFromTriMesh(pX);
				
				cuSyst_host.Output("n0.txt");

				PerformCUDA_Invoke_Populate(
					&cuSyst_host,
					NUMVERTICES,
					pX->InnermostFrillCentroidRadius,
					pX->OutermostFrillCentroidRadius,
					pX->numStartZCurrentTriangles,
					pX->numEndZCurrentTriangles);
			}

			// Run 1 step:
			printf("evaltime %1.8E\n", evaltime);
			
			PerformCUDA_RunStepsAndReturnSystem_Debug(&cuSyst_host, &cuSyst_host2, pX, &X3);
			cuSyst_host.PopulateTriMesh(pX);

			steps_remaining--;
			GlobalStepsCounter++;

			printf("Done steps: %d   ||   Remaining this run: %d\n\n", GlobalStepsCounter, steps_remaining);
		}
		else {
			pX->Advance(pXnew, &X3);
			temp = pX;
			pX = pXnew;
			pXnew = temp;

			steps_remaining_CPU--;
			GSCCPU++;
			printf("Done steps CPU: %d   ||   Remaining this run: %d\n\n", GSCCPU, steps_remaining_CPU);

			sprintf(buf1000, "autosaveCPU%d.dat", GSCCPU);
			pX->Save(buf1000);
			printf("saved as %s\n", buf1000);
		};
		printf("%s\n", report_time(1));

		/*	
		if (GlobalStepsCounter % GRAPHICS_FREQUENCY == 0)
		{
			// make video frames:
			for (i = 0; i < NUMAVI; i++)
			{
				RefreshGraphs(*pX, GraphFlags[i]); // sends data to graphs AND renders them
															   //	::PlanViewGraphs1(*pX);

															   //RefreshGraphs(*pX,GraphFlags[i]); // sends data to graphs AND renders them
				Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);

				if (DXChk(p_backbuffer_surface->GetDC(&surfdc), 100))
					MessageBox(NULL, "GetDC failed", "oh dear", MB_OK);

							//SelectObject(surfdc,surfbit);
				BitBlt(dibdc, 0, 0, VIDEO_WIDTH, VIDEO_HEIGHT, surfdc, 0, 0, SRCCOPY);
					p_backbuffer_surface->ReleaseDC(surfdc);
					AddAviFrame(hAvi[i], dib);

			};
						sprintf(szFile, "System_%d", GlobalStepsCounter);
						//			pX->SaveText(szFile);
		};*/

		/*
		if (GlobalStepsCounter % (AVI_FILE_PINCHOFF_FREQUENCY * GRAPHICS_FREQUENCY) == 0)
		{
		for (i = 0; i < NUMAVI; i++)
		{
		// now have to pinch out avi file and make a new one
		CloseAvi(hAvi[i]);
		sprintf(buf1000, "%s%s_%d.avi", FOLDER, szAvi[i], GlobalStepsCounter);
		hAvi[i] = CreateAvi(buf1000, AVIFRAMEPERIOD, NULL);
		SetAviVideoCompression(hAvi[i], dib, &opts, false, hWnd);
		};
		};
		*/
		RefreshGraphs(*pX,GlobalSpeciesToGraph); // sends data to graphs AND renders them
		Direct3D.pd3dDevice->Present( NULL, NULL, NULL, NULL );
		
		if (steps_remaining > 0) {
			SetTimer(hWnd, 1, DELAY_MILLISECS, NULL);
			printf("Waiting %d milliseconds to allow user input.\n", DELAY_MILLISECS);
		};
		if (steps_remaining_CPU > 0) {
			SetTimer(hWnd, 2, DELAY_MILLISECS, NULL);
			printf("Waiting %d milliseconds to allow user input.\n", DELAY_MILLISECS);
		};
		if (wParam == 1) {
			sprintf(buf1000, "autosaveGPU%d.dat", GlobalStepsCounter);
		} else {
			sprintf(buf1000, "autosaveCPU%d.dat", GSCCPU);
		}
		pX->Save(buf1000);
		printf("saved as %s\n", buf1000);
		printf("save ascii?");
		
		do {
			o = getch();
		} while ((o != 'y') && (o != 'n'));
		printf("%c\n", o);
		if (o == 'y') {
			pX->SaveText(buf1000);
			printf("Ascii file saved.\n");
		}
		printf("steps_remaining GPU: %d  CPU: %d\n",steps_remaining, steps_remaining_CPU);
		
			// Auto-save system:
		//if (GlobalStepsCounter % DATA_SAVE_FREQUENCY == 0)
		
		break;

	case WM_KEYDOWN:

		switch (wParam)
		{
		case 'W':
			GlobalEye.z += 1.0f;
			printf("GlobalEye %f %f %f  \n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z);
			break;
		case 'S':
			GlobalEye.z -= 1.0f;
			printf("GlobalEye %f %f %f  \n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z);
			break;
		case 'A':
			GlobalEye.x -= 0.8f;
			printf("GlobalEye %f %f %f  \n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z);
			break;
		case 'D':
			GlobalEye.x += 0.8f;
			printf("GlobalEye %f %f %f  \n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z);
			break;
		case 'E':
			GlobalEye.y += 0.8f;
			printf("GlobalEye %f %f %f  \n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z);
			break;
		case 'C':
			GlobalEye.y -= 0.8f;
			printf("GlobalEye %f %f %f  \n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z);
			break;

		case 'V':
			GlobalLookat.z -= 0.4f;
			printf("GlobalEye %f %f %f  GlobalLookat %f %f %f\n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z, GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);
			break;
		case 'R':
			GlobalLookat.z += 0.4f;
			printf("GlobalEye %f %f %f  GlobalLookat %f %f %f\n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z, GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);
			break;
		case 'F':
			GlobalLookat.x -= 0.4f;
			printf("GlobalEye %f %f %f  GlobalLookat %f %f %f\n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z, GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);
			break;
		case 'G':
			GlobalLookat.x += 0.4f;
			printf("GlobalEye %f %f %f  GlobalLookat %f %f %f\n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z, GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);
			break;
		case 'T':
			GlobalLookat.y += 0.4f;
			printf("GlobalLookat %f %f %f\n",
				GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);
			break;
		case 'B':
			GlobalLookat.y -= 0.4f;
			printf("GlobalLookat %f %f %f\n",
				GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);
			break;
		case 'Q':
			GlobalCutaway = !GlobalCutaway;
			break;
		case 'Y':
		case '<':
			GlobalEye.x = -10.4; GlobalEye.y = 16.4; GlobalEye.z = 44.0;
			GlobalLookat.x = -3.6; GlobalLookat.y = 3.0; GlobalLookat.z = 72.2;
			printf("GlobalEye %f %f %f  GlobalLookat %f %f %f\n",
				GlobalEye.x, GlobalEye.y, GlobalEye.z, GlobalLookat.x, GlobalLookat.y, GlobalLookat.z);

			GlobalPlanEye.x = 7.1; GlobalPlanEye.y = 11.5; GlobalPlanEye.z = 71.35;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);

			break;
		case '_':
		case '-':
		case '>':
			GlobalPlanEye.x = 7.0; GlobalPlanEye.y = 14.0; GlobalPlanEye.z = 71.0;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
			break;

		case 'U':
			GlobalPlanEye.z += 0.6f;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
			break;
		case 'J':
			GlobalPlanEye.z -= 0.6f;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
			break;
		case 'H':
			GlobalPlanEye.x -= 0.6f;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
			break;
		case 'K':
			GlobalPlanEye.x += 0.6f;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
			break;
		case 'I':
			GlobalPlanEye.y *= 1.25f;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
			break;
		case 'M':
			GlobalPlanEye.y *= 0.8f;
			printf("GlobalPlanEye %f %f %f\n",
				GlobalPlanEye.x, GlobalPlanEye.y, GlobalPlanEye.z);
			break;
		case 'N':
			GlobalboolDisplayMeshWireframe = !GlobalboolDisplayMeshWireframe;
			//Graph1.boolDisplayMeshWireframe = (!(Graph1.boolDisplayMeshWireframe));
			break;
		case '9':
			GlobalRenderLabels = false;
			break;
		case '5':
			GlobalRenderLabels = true;
			GlobalWhichLabels = 0;// iTri
			break;
		case '8':
			GlobalRenderLabels = true;
			GlobalWhichLabels = 1;//T
			break;
		case '7':
			GlobalRenderLabels = true;
			GlobalWhichLabels = 2;//v
			break;
		case '6':
			GlobalRenderLabels = true;
			GlobalWhichLabels = 3;	//n
			break;
		case '1':
			GlobalColoursPlanView = 1;//v
			break;
		case '4':
			GlobalColoursPlanView = 0;//nothing
			break;
		case '2':
			GlobalColoursPlanView = 2;//T
			break;
		case '0':
			steps_remaining = 0;
			break;

		default:
			return DefWindowProc(hWnd, message, wParam, lParam);

		};

		//PlanViewGraphs1(*pX);

		RefreshGraphs(*pX, GlobalSpeciesToGraph); // sends data to graphs AND renders them
		Direct3D.pd3dDevice->Present(NULL, NULL, NULL, NULL);

		break;
	case WM_PAINT:
		// Not sure, do we want to do this?
		//	RefreshGraphs(*pX,); // sends data to graphs AND renders them
		GetUpdateRect(hWnd, &rect, FALSE);
		if (Direct3D.pd3dDevice != NULL)
			Direct3D.pd3dDevice->Present(&rect, &rect, NULL, NULL);

		ValidateRect(hWnd, NULL);
		break;
	case WM_DESTROY:
		DeleteObject(dib);
		DeleteDC(dibdc);
		for (i = 0; i < NUMAVI; i++)
			CloseAvi(hAvi[i]);

		//  _controlfp_s(0, cw, _MCW_EM); // Line A
		PerformCUDA_Revoke();


		PostQuitMessage(0);
		break;
	default:
		return DefWindowProc(hWnd, message, wParam, lParam);
	}
	return 0;
}

// Message handler for about box.
INT_PTR CALLBACK About(HWND hDlg, UINT message, WPARAM wParam, LPARAM lParam)
{
	UNREFERENCED_PARAMETER(lParam);
	switch (message)
	{
	case WM_INITDIALOG:
		return (INT_PTR)TRUE;

	case WM_COMMAND:
		if (LOWORD(wParam) == IDOK || LOWORD(wParam) == IDCANCEL)
		{
			EndDialog(hDlg, LOWORD(wParam));
			return (INT_PTR)TRUE;
		}
		break;
	}
	return (INT_PTR)FALSE;
}


INT_PTR CALLBACK SetupBox(HWND hDlg, UINT message, WPARAM wParam, LPARAM lParam)
{
	UNREFERENCED_PARAMETER(lParam);
	char buffer[2048];
	char string[1024];
	real newh;

	switch (message)
	{
	case WM_INITDIALOG:
		sprintf(buffer, "New h? (present = %1.10E)", h);
		if (GlobalSwitchBox)
			SetDlgItemText(hDlg, IDC_STATIC, buffer);
		return (INT_PTR)TRUE;

	case WM_COMMAND:
		if (LOWORD(wParam) == IDOK)
		{
			// try to read data from edit control:
			GetDlgItemText(hDlg, IDC_EDIT1, buffer, 2048);
			if (GlobalSwitchBox == 0)
			{
				// 
				steps_remaining = atoi(buffer);
				if (steps_remaining >= 0)
				{
					EndDialog(hDlg, LOWORD(wParam));
				}
				else {
					MessageBox(NULL, "incorrect value", "Enter a nonnegative integer.", MB_OK);
				};
			}
			else {
				newh = atof(buffer);
				if (newh > 0.0)
				{
					EndDialog(hDlg, LOWORD(wParam));
					sprintf(string, "h = %1.10E\n", newh);
					h = newh;
					MessageBox(NULL, string, "New value of h", MB_OK);
				}
				else {
					MessageBox(NULL, "no good", "Negative h entered", MB_OK);
				};
			};
			return (INT_PTR)TRUE;
		}
		break;
	}
	return (INT_PTR)FALSE;
}

